#include "hip/hip_runtime.h"
/*#############################################################################
 ******************************************************************************
 * <name> hydro_calcOperator2d_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This file provides CUDA kernels to compute the operator for the low-order
 * scheme in 2D using different types if artificial viscosities.
 * </purpose>
 *
 *#############################################################################/
 */

#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <coproc_core.h>
#include <coproc_storage_cuda.h>
#include "../../cudaGatherScatter.h"
#ifdef HAS_INLINE_PTX
#include "../../cudaDMA.h"
#endif

#define LANGUAGE LANGUAGE_C
#include "../../flagship.h"
#include "../../cudaMacros.h"

#define HYDRO_NDIM 2
#include "hydro.h"

// Defines for baseline implementation
#define BASELINE_THREADS_PER_CTA  32*2
#define BASELINE_NEQ_PER_THREAD   1
#define BASELINE_NEDGE_PER_THREAD 1

// Defines for empty cudaDMA implementation
#ifndef CUDADMA_KERNEL
#define CUDADMA_COMPUTE_THREADS_PER_CTA 0
#define CUDADMA_THREADS_PER_LD          0
#define CUDADMA_NEQ_PER_THREAD          0
#define CUDADMA_NEDGE_PER_THREAD        0
#define CUDADMA_DMA_LDS_IND             0
#define CUDADMA_DMA_LDS_SRC             0
#define CUDADMA_DMA_LDS_DEST            0
#define CUDADMA_DMA_LDS_COEFF           0
#define CUDADMA_DMA_LDS                 0
#endif

using namespace std;

namespace hydro2d_cuda
{
  /*****************************************************************************
   * CUDA kernels for hydrodynamic model in 2D
   ****************************************************************************/
  
  // Memory pool in constant device memory
  __device__ __constant__ __SIZET constMemPool[NVAR2D*NVAR2D];
  
  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase (basic functionality and specialisations)
   ****************************************************************************/

  template <int isystemcoupling>
  struct InviscidFluxJacobiMatrixBase
  {
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase: Specialization for block-diagonal matrix
   ****************************************************************************/

  template<>
  struct InviscidFluxJacobiMatrixBase<SYSTEM_SEGREGATED>
  {
    /***************************************************************************
     * Compute flux Jacobian matrix for neqsim nodes
     **************************************************************************/
    template <int neqsim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
							 Tc *CoeffsAtDiag,
							 Td scale,
							 Td ui,
							 Td vi,
							 Ti ipos,
							 Ti ieq,
							 Ti neq,
							 Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ii = diag(A_i)*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
#else
      // Compute Galerkin coefficient $K_ii = -diag(A_i)*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
#endif
    }

    /***************************************************************************
     * Compute flux Jacobian matrix for nedgesim edges
     **************************************************************************/
    template <int nedgesim, bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti ipos,
							 Ti iedge,
							 Ti nedge,
							 Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ij = diag(A_j)*C_{ji}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);

      // Compute Galerkin coefficient $K_ji = diag(A_i)*C_{ij}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
#else
      // Compute Galerkin coefficient $K_ij = -diag(A_j)*C_{ij}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      
      // Compute Galerkin coefficient $K_ji = -diag(A_i)*C_{ji}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
#endif
    }
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase: Specialization for full matrix
   ****************************************************************************/

  template <>
  struct InviscidFluxJacobiMatrixBase<SYSTEM_ALLCOUPLED>
  {
    /***************************************************************************
     * Compute flux Jacobian matrix for neqsim nodes
     **************************************************************************/
    template <int neqsim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
							 Tc *CoeffsAtDiag,
							 Td scale,
							 Td ui,
							 Td vi,
							 Td Ei,
							 Ti ipos,
							 Ti ieq,
							 Ti neq,
							 Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ii = A_i*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX21(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX31(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX41(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,5,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX12(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,6,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,7,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX32(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,8,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX42(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,9,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX13(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,10,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX23(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,11,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,12,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX43(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,13,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX14(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,14,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX24(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,15,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX34(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,16,ipos,NVAR2D*NVAR2D,neqsim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
#else
      // Compute Galerkin coefficient $K_ii = A_i*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX21(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX31(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX41(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,5,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX12(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,6,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,7,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX32(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,8,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX42(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,9,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX13(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,10,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX23(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,11,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,12,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX43(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,13,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX14(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,14,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX24(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,15,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX34(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,16,ipos,NVAR2D*NVAR2D,neqsim) = -
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
								   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
#endif
    }

    /***************************************************************************
     * Compute flux Jacobian matrix for nedgesim edges
     **************************************************************************/
    template <int nedgesim, bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ij = A_j*C_{ji}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX21(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX31(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX41(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,5,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX12(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,6,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,7,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX32(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,8,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX42(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,9,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX13(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,10,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX23(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,11,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,12,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX43(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,13,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX14(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,14,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX24(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,15,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX34(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,16,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);

      // Compute Galerkin coefficient $K_ji = A_i*C_{ij}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX21(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX31(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX41(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,5,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX12(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,6,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,7,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX32(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,8,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX42(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,9,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX13(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,10,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX23(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,11,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,12,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX43(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,13,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX14(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,14,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX24(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,15,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX34(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,16,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
#else
      // Compute Galerkin coefficient $K_ij = -A_j*C_{ij}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX21(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX31(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX41(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,5,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX12(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,6,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,7,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX32(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,8,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX42(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,9,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX13(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,10,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX23(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,11,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,12,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX43(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,13,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX14(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,14,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX24(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,15,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX34(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,16,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      
      // Compute Galerkin coefficient $K_ji = -A_i*C_{ji}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX11(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX21(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX31(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX41(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,5,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX12(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,6,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX22(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,7,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX32(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,8,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX42(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,9,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX13(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,10,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX23(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,11,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX33(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,12,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX43(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,13,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX14(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,14,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX24(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,15,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX34(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,16,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
		INVISCIDFLUXJACOBIMATRIX44(-scale,
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
								   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
#endif
    }
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrix
   ****************************************************************************/

  template <int isystemcoupling>
  struct InviscidFluxJacobiMatrix : public InviscidFluxJacobiMatrixBase<isystemcoupling>
  {
    // Enable use of inherited functions
    using InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData;
    using InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData;

    /**************************************************************************
     * Wrapper routine for processing a single node
     *************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
							 Tc *CoeffsAtDiag,
							 Td scale,
							 Td ui,
							 Td vi, 
							 Ti ieq,
							 Ti neq,
							 Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData<1>
		(MatrixAtDiag,CoeffsAtDiag,scale,ui,vi,1,ieq,neq,ncoeff);
    }

    /**************************************************************************
     * Wrapper routine for processing a single node
     *************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
							 Tc *CoeffsAtDiag,
							 Td scale,
							 Td ui,
							 Td vi,
							 Td Ei,
							 Ti ieq,
							 Ti neq,
							 Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData<1>
		(MatrixAtDiag,CoeffsAtDiag,scale,ui,vi,Ei,1,ieq,neq,ncoeff);
    }
    
    /**************************************************************************
     * Wrapper routine for processing a single edge
     *************************************************************************/
    template <bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti iedge,
							 Ti nedge,
							 Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData<1,bstabilise>
		(MatrixAtEdge,CoeffsAtEdge,scale,ui,uj,vi,vj,1,iedge,nedge,ncoeff);
    }

    /**************************************************************************
     * Wrapper routine for processing a single edge
     *************************************************************************/
    template <bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti iedge,
							 Ti nedge,
							 Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData<1,bstabilise>
		(MatrixAtEdge,CoeffsAtEdge,scale,ui,uj,vi,vj,Ei,Ej,1,iedge,nedge,ncoeff);
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase (basic functionality individual specialisations)
   ****************************************************************************/
  
  template <int isystemcoupling, int idissipationtype>
  struct InviscidFluxDissipationMatrixBase
  {
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing zero artificial dissipation, aka standard Galerkin
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_ZERO>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
#pragma unroll
      for (int i=1; i<=NVAR2D; i++)
		IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = 0.0;
    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing zero artificial dissipation, aka standard Galerkin
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_ZERO>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
#pragma unroll
      for (int i=1; i<=NVAR2D*NVAR2D; i++)
		IDX3(MatrixAtEdge,i,1,ipos,NVAR2D*NVAR2D,3,nedgesim) = 0.0;
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing scalar artificial dissipation proportional to the
   * spectral radius of the Roe matrix
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_SCALAR>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
						  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
						  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);
    
      // Compute pressures
      Td pi = PRESSURE3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td pj = PRESSURE3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);

      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
      
      // Compute scalar dissipation
      Td d_ij = abs(vel_ij) + anorm*c_ij;

#pragma unroll
      for (int i=1; i<=NVAR2D; i++)
		IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = d_ij;
    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing scalar artificial dissipation proportional to the
   * spectral radius of the Roe matrix
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_SCALAR>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
						  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
						  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);
    
      // Compute pressures
      Td pi = PRESSURE3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td pj = PRESSURE3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);

      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
      
      // Compute scalar dissipation
      Td d_ij = abs(vel_ij) + anorm*c_ij;

#pragma unroll
      for (int i=1; i<=NVAR2D*NVAR2D; i++)
		IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = 0.0;

      for (int i=1; i<=NVAR2D*NVAR2D; i+=(i-1)*NVAR2D)
		IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = d_ij;
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing tensorial artificial dissipation of Roe-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_ROE>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {

    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing tensorial artificial dissipation of Roe-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_ROE>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {

    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing scalar artificial dissipation of Rusanov-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_RUSANOV>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {

    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing scalar artificial dissipation of Rusanov-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_RUSANOV>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti ipos,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {

    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrix: Artificial dissipation
   ****************************************************************************/
  
  template <int isystemcoupling, int idissipationtype>
  struct InviscidFluxDissipationMatrix : public InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>
  {   
    // Enable use of inherited functions
    using InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData;
    
    /***************************************************************************
     * Wrapper routine for processing a single edge
     **************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
      InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData<1>
		(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,scale,ui,uj,vi,vj,1,iedge,nedge,ncoeff);
    }

    /***************************************************************************
     * Wrapper routine for processing a single edge
     **************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
							 Tc *CoeffsAtEdge,
							 Td *DataAtEdge,
							 Td scale,
							 Td ui,
							 Td uj,
							 Td vi,
							 Td vj,
							 Td Ei,
							 Td Ej,
							 Ti iedge, 
							 Ti nedge,
							 Ti ncoeff)
    {
      InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData<1>
		(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,scale,ui,uj,vi,vj,Ei,Ej,1,iedge,nedge,ncoeff);
    }
  }; 
  
  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * block-diagonal global operator (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int isystemformat,
			int threads_per_cta>
  __launch_bounds__(threads_per_cta)
  __global__ void hydro_calcMatDiagMatD2d_baseline(Tc *CoeffsAtDiag,
												   Ti *IdiagList,
												   Tv *vec,
												   Tm *mat,
												   Tm scale,
												   Ti neq,
												   Ti na,
												   Ti ncoeff,
												   Ti neq_last,
												   Ti neq_per_thread=1,
												   Ti neq_offset=0)
  {  
    // Loop over all items per thread
    for (int ipt=0; ipt<neq_per_thread; ++ipt) {
      
      // Global node ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+neq_offset+threadIdx.x;
      
      if (idx < neq_last) {
		// Get actual equation number
		Ti ieq = IDX2_DIAGLIST(IdiagList,1,idx+1,2,neq);
	
		// Local data at node from local memory
		Tm DataAtDiag[NVAR2D];
	
		// Get solution values at node
		Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
		  gatherNodeData<true>(DataAtDiag,vec,ieq,neq);
	
		// Compute velocities
		Tm ui = XVELOCITY2(DataAtDiag,IDX2,1,NVAR2D,1);
		Tm vi = YVELOCITY2(DataAtDiag,IDX2,1,NVAR2D,1);
	
		// Compute Galerkin coefficient $K_ii$
		InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
		  calcNodeData(DataAtDiag,CoeffsAtDiag,scale,ui,vi,ieq,neq,ncoeff);
	
		// Get diagonal position in the global matrix
		Ti ia  = IDX2_DIAGLIST(IdiagList,2,idx+1,2,neq);
	
		// Build coefficients into global operator
		Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
		  scatterNodeData<true>(mat,DataAtDiag,ia,na);
      }
    }
  };
  
  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * full global operator (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int isystemformat,
			int threads_per_cta>
  __launch_bounds__(threads_per_cta)
  __global__ void hydro_calcMatDiag2d_baseline(Tc *CoeffsAtDiag,
											   Ti *IdiagList,
											   Tv *vec,
											   Tm *mat,
											   Tm scale,
											   Ti neq,
											   Ti na,
											   Ti ncoeff,
											   Ti neq_last,
											   Ti neq_per_thread=1,
											   Ti neq_offset=0)
  {
    // Loop over all items per thread
    for (int ipt=0; ipt<neq_per_thread; ++ipt) {
      
      // Global node ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+neq_offset+threadIdx.x;
      
      if (idx < neq_last) {
		// Get actual equation
		Ti ieq = IDX2_DIAGLIST(IdiagList,1,idx+1,2,neq);
	
		// Local solution data at node from local memory
		Tm DataAtDiag[NVAR2D*NVAR2D];
	
		// Get solution values at node
		Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
		  gatherNodeData<true>(DataAtDiag,vec,ieq,neq);
	
		// Compute velocities and energy
		Tm ui = XVELOCITY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
		Tm vi = YVELOCITY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
		Tm Ei = SPECIFICTOTALENERGY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
	
		// Compute Galerkin coefficient $K_ii$
		InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
		  calcNodeData(DataAtDiag,CoeffsAtDiag,scale,ui,vi,Ei,ieq,neq,ncoeff);
	
		// Get diagonal position in the global matrix
		Ti ia  = IDX2_DIAGLIST(IdiagList,2,idx+1,2,neq);
	
		// Build coefficients into global operator
		Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
		  scatterNodeData<true>(mat,DataAtDiag,ia,na);
      }
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the
   * block-diagonal global operator and assembles the artificial
   * dissipation tensor if required (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int isystemformat,
			int idissipation,
			bool blumping,
			int threads_per_cta>
  __launch_bounds__(threads_per_cta)
  __global__ void hydro_calcMatrixMatD2d_baseline(Tc *CoeffsAtEdge,
												  Ti *IedgeList,
												  Tv *vec,
												  Tm *mat,
												  Tm scale,
												  Ti neq,
												  Ti na,
												  Ti nedge,
												  Ti ncoeff,
												  Ti nedge_last,
												  Ti nedge_per_thread=1,
												  Ti nedge_offset=0)
  {
    // Loop over all items per thread
    for (int ipt=0; ipt<nedge_per_thread; ++ipt) {
      
      // Global edge ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+nedge_offset+threadIdx.x;
      
      if (idx < nedge_last) {
		// Get positions of edge endpoints (idx starts at zero)
		Ti i = IDX2_EDGELIST(IedgeList,1,idx+1,6,nedge);
		Ti j = IDX2_EDGELIST(IedgeList,2,idx+1,6,nedge);
	
		// Local solution data at edge from local memory
		Tm DataAtEdge[2*NVAR2D];
	
		// Get solution values at edge endpoints
		Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
		  gatherEdgeData<true>(DataAtEdge,vec,i,j,neq);
	
		// Compute velocities
		Tm ui = XVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
		Tm vi = YVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	
		Tm uj = XVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
		Tm vj = YVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	
		if (idissipation == DISSIPATION_ZERO) {
	  
		  // Local matrix data at edge from local memory
		  Tm MatrixAtEdge[2*NVAR2D];
	  
		  // Compute Galerkin coefficient $K_ij$ and $K_ji$
		  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
			calcEdgeData<false>(MatrixAtEdge,CoeffsAtEdge,
								scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
		  // Build matrix coefficients into global operator
		  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
			scatterEdgeData<true,false,blumping>(mat,MatrixAtEdge,
												 IedgeList,idx+1,nedge,na);
	  
		} 
		else {
	  
		  // Local matrix data at edge from local memory
		  Tm MatrixAtEdge[3*NVAR2D];
	  
		  // Compute Galerkin coefficient $K_ij$ and $K_ji$
		  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
			calcEdgeData<true>(MatrixAtEdge,CoeffsAtEdge,
							   scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
		  // Compute contribution of artificial diffusion
		  InviscidFluxDissipationMatrix<SYSTEM_SEGREGATED,idissipation>::
			calcEdgeData(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,
						 scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
		  // Build matrix coefficients into global operator
		  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
			scatterEdgeData<true,true,blumping>(mat,MatrixAtEdge,
												IedgeList,idx+1,nedge,na);
		}
      }
    }
  };
  
  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the full
   * global operator and assembles the artificial dissipation tensor
   * if required (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int isystemformat,
			int idissipation,
			bool blumping,
			int threads_per_cta>
  __launch_bounds__(threads_per_cta)
  __global__ void hydro_calcMatrix2d_baseline(Tc *CoeffsAtEdge,
											  Ti *IedgeList,
											  Tv *vec,
											  Tm *mat,
											  Tm scale,
											  Ti neq,
											  Ti na,
											  Ti nedge,
											  Ti ncoeff,
											  Ti nedge_last,
											  Ti nedge_per_thread=1,
											  Ti nedge_offset=0)

  {
    // Loop over all items per thread
    for (int ipt=0; ipt<nedge_per_thread; ++ipt) {
      
      // Global edge ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+nedge_offset+threadIdx.x;

      if (idx < nedge_last)
		{
		  // Get positions of edge endpoints (idx starts at zero)
		  Ti i = IDX2_EDGELIST(IedgeList,1,idx+1,6,nedge);
		  Ti j = IDX2_EDGELIST(IedgeList,2,idx+1,6,nedge);
	  
		  // Local solution data at edge from local memory
		  Tm DataAtEdge[2*NVAR2D];
	  
		  // Get solution values at edge endpoints
		  Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
			gatherEdgeData<true>(DataAtEdge,vec,i,j,neq);
	  
		  // Compute velocities
		  Tm ui = XVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
		  Tm vi = YVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	  
		  Tm uj = XVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
		  Tm vj = YVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	  
		  // Compute specific energies
		  Tm Ei = SPECIFICTOTALENERGY2(DataAtEdge,IDX2,1,NVAR2D,2);
		  Tm Ej = SPECIFICTOTALENERGY2(DataAtEdge,IDX2,2,NVAR2D,2);
	  
		  if (idissipation == DISSIPATION_ZERO) {
	    
			// Local matrix data at edge from local memory
			Tm MatrixAtEdge[2*NVAR2D*NVAR2D];
	    
			// Compute Galerkin coefficient $K_ij$ and $K_ji$
			InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
			  calcEdgeData<false>(MatrixAtEdge,CoeffsAtEdge,
								  scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
			// // Build matrix coefficients into global operator
			Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
			  scatterEdgeData<true,false,blumping>(mat,MatrixAtEdge,
												   IedgeList,idx+1,nedge,na);
		  } else {
	    
			// Local matrix data at edge from local memory
			Tm MatrixAtEdge[3*NVAR2D*NVAR2D];
	    
			// Compute Galerkin coefficient $K_ij$ and $K_ji$
			InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
			  calcEdgeData<true>(MatrixAtEdge,CoeffsAtEdge,
								 scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
			// Compute contribution of artificial diffusion
			InviscidFluxDissipationMatrix<SYSTEM_ALLCOUPLED,idissipation>::
			  calcEdgeData(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,
						   scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
			// Build matrix coefficients into global operator
			Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
			  scatterEdgeData<true,true,blumping>(mat,MatrixAtEdge,
												  IedgeList,idx+1,nedge,na);
		  }
		}
    }
  };
  
  /*****************************************************************************
   * Internal C++ functions which invoke the CUDA kernels
   ****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti>
  inline
  int hydro_calcMatDiagMatD2d_cuda(__SIZET *d_CoeffsAtDiag,
								   __SIZET *d_IdiagList,
								   __SIZET *d_vec,
								   __SIZET *d_mat,
								   Tm scale,
								   Ti nblocks,
								   Ti neq,
								   Ti na,
								   Ti ncoeff,
								   hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();
    
    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta = CUDADMA_COMPUTE_THREADS_PER_CTA;
    const int dma_threads_per_ld      = CUDADMA_THREADS_PER_LD;
    const int dma_lds                 = CUDADMA_DMA_LDS;
    int neq_per_thread_cudaDMA        = CUDADMA_NEQ_PER_THREAD;

    const int threads_per_cta_baseline = BASELINE_THREADS_PER_CTA;
    int neq_per_thread_baseline        = BASELINE_NEQ_PER_THREAD;
    
    int blocks, threads, neq_cudaDMA, neq_baseline;
    prepare_cudaDMA(devProp, neq,
					&neq_per_thread_cudaDMA,
					compute_threads_per_cta, dma_threads_per_ld,
					dma_lds, &blocks, &threads, &neq_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, neq-neq_cudaDMA,
					 &neq_per_thread_baseline, threads_per_cta_baseline,
					 &blocks, &threads, &neq_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);
    
    Tv *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtDiag = (Tc*)(*d_CoeffsAtDiag);
    Ti *IdiagList = (Ti*)(*d_IdiagList);
        
    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);
    
#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
      	// CudaDMA implementation
      	hydro_calcMatDiagMatD2d_cudaDMA
      	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,compute_threads_per_cta,dma_threads_per_ld>
      	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
													   IdiagList,
													   vec, mat, scale,
													   neq, na, ncoeff,
													   neq_cudaDMA,
													   neq_per_thread_cudaDMA);
#endif

      if (grid_baseline.x>0)
      	// Baseline implementation
      	hydro_calcMatDiagMatD2d_baseline
      	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,threads_per_cta_baseline>
      	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
														 IdiagList,
														 vec, mat, scale,
														 neq, na, ncoeff,
														 neq, 
														 neq_per_thread_baseline,
														 neq_cudaDMA);
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      __SIZET cmemPool[NVAR2D];
#pragma unroll
      for (int i=0; i<NVAR2D; i++)
		cmemPool[i] = d_mat[i*(NVAR2D+1)];
      
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), cmemPool,
							  sizeof(__SIZET)*NVAR2D, 0,
							  hipMemcpyHostToDevice,
							  stream);
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
		// CudaDMA implementation
		hydro_calcMatDiagMatD2d_cudaDMA
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,compute_threads_per_cta,dma_threads_per_ld>
		  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
													   IdiagList,
													   vec, mat, scale,
													   neq, na, ncoeff,
													   neq_cudaDMA,
													   neq_per_thread_cudaDMA);
#endif

      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatDiagMatD2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
														 IdiagList,
														 vec, mat, scale,
														 neq, na, ncoeff,
														 neq,
														 neq_per_thread_baseline,
														 neq_cudaDMA);
    }
    
    coproc_checkError("hydro_calcMatDiagMatD2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti>
  inline
  int hydro_calcMatDiag2d_cuda(__SIZET *d_CoeffsAtDiag,
							   __SIZET *d_IdiagList,
							   __SIZET *d_vec,
							   __SIZET *d_mat,
							   Tm scale,
							   Ti nblocks,
							   Ti neq,
							   Ti na,
							   Ti ncoeff,
							   hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();
    
    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta = CUDADMA_COMPUTE_THREADS_PER_CTA;
    const int dma_threads_per_ld      = CUDADMA_THREADS_PER_LD;
    const int dma_lds                 = CUDADMA_DMA_LDS;
    int neq_per_thread_cudaDMA        = CUDADMA_NEQ_PER_THREAD;

    const int threads_per_cta_baseline = BASELINE_THREADS_PER_CTA;
    int neq_per_thread_baseline        = BASELINE_NEQ_PER_THREAD;
    
    int blocks, threads, neq_cudaDMA, neq_baseline;
    prepare_cudaDMA(devProp, neq,
					&neq_per_thread_cudaDMA,
					compute_threads_per_cta, dma_threads_per_ld,
					dma_lds, &blocks, &threads, &neq_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, neq-neq_cudaDMA,
					 &neq_per_thread_baseline, threads_per_cta_baseline,
					 &blocks, &threads, &neq_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);

    Tv *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtDiag = (Tc*)(*d_CoeffsAtDiag);
    Ti *IdiagList = (Ti*)(*d_IdiagList);
    
    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);
      
#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
		//CudaDMA implementation
		hydro_calcMatDiag2d_cudaDMA
		  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,compute_threads_per_cta,dma_threads_per_ld>
		  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
													   IdiagList,
													   vec, mat, scale,
													   neq, na, ncoeff,
													   neq_cudaDMA,
													   neq_per_thread_cudaDMA);
#endif

      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatDiag2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
														 IdiagList,
														 vec, mat, scale,
														 neq, na, ncoeff,
														 neq,
														 neq_per_thread_baseline,
														 neq_cudaDMA);
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), d_mat,
							  sizeof(__SIZET)*NVAR2D*NVAR2D, 0,
							  hipMemcpyHostToDevice,
							  stream);
      
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
		// CudaDMA implementation
		hydro_calcMatDiag2d_cudaDMA
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,compute_threads_per_cta,dma_threads_per_ld>
		  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
													   IdiagList,
													   vec, mat, scale,
													   neq, na, ncoeff,
													   neq_cudaDMA,
													   neq_per_thread_cudaDMA);
#endif

      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatDiag2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
														 IdiagList,
														 vec, mat, scale,
														 neq, na, ncoeff,
														 neq,
														 neq_per_thread_baseline,
														 neq_cudaDMA);
    }
    
    coproc_checkError("hydro_calcMatDiag2d_cuda");
    return 0;
  };

  /*****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int idissipationtype,
			bool blumping>
  inline
  int hydro_calcMatrixMatD2d_cuda(__SIZET *d_CoeffsAtEdge,
								  __SIZET *d_IedgeList,
								  __SIZET *d_vec,
								  __SIZET *d_mat,
								  Tm scale,
								  Ti nblocks,
								  Ti neq,
								  Ti na,
								  Ti nedge,
								  Ti ncoeff,
								  Ti nedgeset,
								  Ti iedgeset,
								  hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();

    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta  = CUDADMA_COMPUTE_THREADS_PER_CTA;
    const int dma_threads_per_ld       = CUDADMA_THREADS_PER_LD;
    const int dma_lds                  = CUDADMA_DMA_LDS;
    int nedge_per_thread_cudaDMA       = CUDADMA_NEDGE_PER_THREAD;

    const int threads_per_cta_baseline = BASELINE_THREADS_PER_CTA;
    int nedge_per_thread_baseline      = BASELINE_NEDGE_PER_THREAD;
    
    int blocks, threads, nedge_cudaDMA, nedge_baseline;
    prepare_cudaDMA(devProp, nedgeset,
					&nedge_per_thread_cudaDMA,
					compute_threads_per_cta, dma_threads_per_ld,
					dma_lds, &blocks, &threads, &nedge_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, nedgeset-nedge_cudaDMA,
					 &nedge_per_thread_baseline, threads_per_cta_baseline,
					 &blocks, &threads, &nedge_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);
    
    Tv  *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);

    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);
      
#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
      	// CudaDMA implementation
      	hydro_calcMatrixMatD2d_cudaDMA
      	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,
      	   compute_threads_per_cta,dma_threads_per_ld>
      	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
													   IedgeList,
													   vec, mat, scale,
													   neq, na, nedge, ncoeff,
													   nedge_cudaDMA+iedgeset-1, 
													   nedge_per_thread_cudaDMA,
													   iedgeset-1);
#endif

      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatrixMatD2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
														 IedgeList,
														 vec, mat, scale,
														 neq, na, nedge, ncoeff,
														 nedgeset+iedgeset-1, 
														 nedge_per_thread_baseline,
														 nedge_cudaDMA+iedgeset-1);
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      __SIZET cmemPool[NVAR2D];

#pragma unroll
      for (int i=0; i<NVAR2D; i++)
		cmemPool[i] = d_mat[i*(NVAR2D+1)];
      
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), cmemPool,
							  sizeof(__SIZET)*NVAR2D, 0,
							  hipMemcpyHostToDevice,
							  stream);
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

#ifdef CUDADMA_KERNEL
      if (grid_cudaDMA.x>0)
      	// CudaDMA implementation
      	hydro_calcMatrixMatD2d_cudaDMA
      	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,
      	   compute_threads_per_cta,dma_threads_per_ld>
      	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
													   IedgeList,
													   vec, mat, scale,
													   neq, na, nedge, ncoeff,
													   nedge_cudaDMA+iedgeset-1, 
													   nedge_per_thread_cudaDMA,
													   iedgeset-1);
#endif

      if (grid_baseline.x>0)
		hydro_calcMatrixMatD2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
														 IedgeList,
														 vec, mat, scale,
														 neq, na, nedge, ncoeff,
														 nedgeset+iedgeset-1,
														 nedge_per_thread_baseline,
														 nedge_cudaDMA+iedgeset-1);
    }
    
    coproc_checkError("hydro_calcMatrixMatD2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
			typename Tv,
			typename Tm,
			typename Ti,
			int idissipationtype,
			bool blumping>
  inline
  int hydro_calcMatrix2d_cuda(__SIZET *d_CoeffsAtEdge,
							  __SIZET *d_IedgeList,
							  __SIZET *d_vec,
							  __SIZET *d_mat,
							  Tm scale,
							  Ti nblocks,
							  Ti neq,
							  Ti na,
							  Ti nedge,
							  Ti ncoeff,
							  Ti nedgeset,
							  Ti iedgeset,
							  hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();

    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta  = CUDADMA_COMPUTE_THREADS_PER_CTA;
    const int dma_threads_per_ld       = CUDADMA_THREADS_PER_LD;
    const int dma_lds                  = CUDADMA_DMA_LDS;
    int nedge_per_thread_cudaDMA       = CUDADMA_NEDGE_PER_THREAD;

    const int threads_per_cta_baseline = BASELINE_THREADS_PER_CTA;
    int nedge_per_thread_baseline      = BASELINE_NEDGE_PER_THREAD;
    
    int blocks, threads, nedge_cudaDMA, nedge_baseline;
    prepare_cudaDMA(devProp, nedgeset,
					&nedge_per_thread_cudaDMA,
					compute_threads_per_cta, dma_threads_per_ld,
					dma_lds, &blocks, &threads, &nedge_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, nedgeset-nedge_cudaDMA,
					 &nedge_per_thread_baseline, threads_per_cta_baseline,
					 &blocks, &threads, &nedge_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);

    Tv  *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);

    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);

#ifdef CUDADMA_KERNEL    
      if (grid_cudaDMA.x>0)
		// CudaDMA implementation
		hydro_calcMatrix2d_cudaDMA
		  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,
		   compute_threads_per_cta,dma_threads_per_ld>
		  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
													   IedgeList,
													   vec, mat, scale,
													   neq, na, nedge, ncoeff,
													   nedge_cudaDMA+iedgeset-1, 
													   nedge_per_thread_cudaDMA,
													   iedgeset-1);
#endif

      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatrix2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
														 IedgeList,
														 vec, mat, scale,
														 neq, na, nedge, ncoeff,
														 nedgeset+iedgeset-1, 
														 nedge_per_thread_baseline,
														 nedge_cudaDMA+iedgeset-1);
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), d_mat,
							  sizeof(__SIZET)*NVAR2D*NVAR2D, 0,
							  hipMemcpyHostToDevice,
							  stream);
      
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

#ifdef CUDADMA_KERNEL    
      if (grid_cudaDMA.x>0)
		// CudaDMA implementation
		hydro_calcMatrix2d_cudaDMA
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,
		   compute_threads_per_cta,dma_threads_per_ld>
		  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
													   IedgeList,
													   vec, mat, scale,
													   neq, na, nedge, ncoeff,
													   nedge_cudaDMA+iedgeset-1, 
													   nedge_per_thread_cudaDMA,
													   iedgeset-1);
#endif
      
      if (grid_baseline.x>0)
		// Baseline implementation
		hydro_calcMatrix2d_baseline
		  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,threads_per_cta_baseline>
		  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
														 IedgeList,
														 vec, mat, scale,
														 neq, na, nedge, ncoeff,
														 nedgeset+iedgeset-1, 
														 nedge_per_thread_baseline,
														 nedge_cudaDMA+iedgeset-1);
    }
    
    coproc_checkError("hydro_calcMatrix2d_cuda");
    return 0;
  };
  
  /*****************************************************************************
   * External C functions which can be called from the Fortran code
   ****************************************************************************/

  extern "C" {
    __INT FNAME(hydro_calcmatdiagmatd2d_cuda)(__SIZET *d_CoeffsAtDiag,
											  __SIZET *d_IdiagList,
											  __SIZET *d_vec,
											  __SIZET *d_mat,
											  __DP *scale,
											  __INT *nblocks,
											  __INT *neq,
											  __INT *na,
											  __INT *ncoeff,
											  __I64 *stream)
    {
      return (__INT) hydro_calcMatDiagMatD2d_cuda
		<__DP,__DP,__DP,__INT>(d_CoeffsAtDiag, d_IdiagList, d_vec, d_mat,
							   *scale, *nblocks, *neq, *na, *ncoeff,
							   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatdiag2d_cuda)(__SIZET *d_CoeffsAtDiag,
										  __SIZET *d_IdiagList,
										  __SIZET *d_vec,
										  __SIZET *d_mat,
										  __DP *scale,
										  __INT *nblocks,
										  __INT *neq,
										  __INT *na,
										  __INT *ncoeff,
										  __I64 *stream)
    {
      return (__INT) hydro_calcMatDiag2d_cuda
		<__DP,__DP,__DP,__INT>(d_CoeffsAtDiag, d_IdiagList, d_vec, d_mat,
							   *scale, *nblocks, *neq, *na, *ncoeff,
							   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatgalmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
											 __SIZET *d_IedgeList,
											 __SIZET *d_vec,
											 __SIZET *d_mat,
											 __DP *scale,
											 __INT *nblocks,
											 __INT *neq,
											 __INT *na,
											 __INT *nedge,
											 __INT *ncoeff,
											 __INT *nedgeset,
											 __INT *iedgeset,
											 __INT *cconstrType,
											 __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }
    
    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatgalerkin2d_cuda)(__SIZET *d_CoeffsAtEdge,
											  __SIZET *d_IedgeList,
											  __SIZET *d_vec,
											  __SIZET *d_mat,
											  __DP *scale,
											  __INT *nblocks,
											  __INT *neq,
											  __INT *na,
											  __INT *nedge,
											  __INT *ncoeff,
											  __INT *nedgeset,
											  __INT *iedgeset,
											  __INT *cconstrType,
											  __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatscdissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
												__SIZET *d_IedgeList,
												__SIZET *d_vec,
												__SIZET *d_mat,
												__DP *scale,
												__INT *nblocks,
												__INT *neq,
												__INT *na,
												__INT *nedge,
												__INT *ncoeff,
												__INT *nedgeset,
												__INT *iedgeset,
												__INT *cconstrType,
												__I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatscdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
											__SIZET *d_IedgeList,
											__SIZET *d_vec,
											__SIZET *d_mat,
											__DP *scale,
											__INT *nblocks,
											__INT *neq,
											__INT *na,
											__INT *nedge,
											__INT *ncoeff,
											__INT *nedgeset,
											__INT *iedgeset,
											__INT *cconstrType,
											__I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatroedissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
												 __SIZET *d_IedgeList,
												 __SIZET *d_vec,
												 __SIZET *d_mat,
												 __DP *scale,
												 __INT *nblocks,
												 __INT *neq,
												 __INT *na,
												 __INT *nedge,
												 __INT *ncoeff,
												 __INT *nedgeset,
												 __INT *iedgeset,
												 __INT *cconstrType,
												 __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatroediss2d_cuda)(__SIZET *d_CoeffsAtEdge,
											 __SIZET *d_IedgeList,
											 __SIZET *d_vec,
											 __SIZET *d_mat,
											 __DP *scale,
											 __INT *nblocks,
											 __INT *neq,
											 __INT *na,
											 __INT *nedge,
											 __INT *ncoeff,
											 __INT *nedgeset,
											 __INT *iedgeset,
											 __INT *cconstrType,
											 __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatrusdissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
												 __SIZET *d_IedgeList,
												 __SIZET *d_vec,
												 __SIZET *d_mat,
												 __DP *scale,
												 __INT *nblocks,
												 __INT *neq,
												 __INT *na,
												 __INT *nedge,
												 __INT *ncoeff,
												 __INT *nedgeset,
												 __INT *iedgeset,
												 __INT *cconstrType,
												 __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrixMatD2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatrusdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
											 __SIZET *d_IedgeList,
											 __SIZET *d_vec,
											 __SIZET *d_mat,
											 __DP *scale,
											 __INT *nblocks,
											 __INT *neq,
											 __INT *na,
											 __INT *nedge,
											 __INT *ncoeff,
											 __INT *nedgeset,
											 __INT *iedgeset,
											 __INT *cconstrType,
											 __I64 *stream)
    {
      if (*cconstrType == 0)
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,false>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
      else
		return (__INT) hydro_calcMatrix2d_cuda
		  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,true>
		  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
		   *scale, *nblocks, *neq, *na, *nedge,
		   *ncoeff, *nedgeset, *iedgeset,
		   (hipStream_t)(*stream));
    }
  };
}
