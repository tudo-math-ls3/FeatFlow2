#include "hip/hip_runtime.h"
/*#############################################################################
******************************************************************************
* <name> hydro_calcOperator2d_cuda </name>
******************************************************************************
*
* <purpose>
* This file provides CUDA kernels to compute the operator for the low-order
* scheme in 2D using different types if artificial viscosities.
* </purpose>
*
*#############################################################################/
*/

#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <coproc_core.h>
#include <coproc_storage_cuda.h>
#include "../../cudaDMA.h"
#include "../../cudaGatherScatter.h"

#define LANGUAGE LANGUAGE_C
#include "../../flagship.h"
#include "../../cudaMacros.h"

#define HYDRO_NDIM 2
#include "hydro.h"

using namespace std;

namespace hydro2d_cuda
{
  /*****************************************************************************
   * CUDA kernels for hydrodynamic model in 2D
   ****************************************************************************/
  
  // Memory pool in constant device memory
  __device__ __constant__ __SIZET constMemPool[NVAR2D*NVAR2D];
  
  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase (basic functionality and specialisations)
   ****************************************************************************/

  template <int isystemcoupling>
  struct InviscidFluxJacobiMatrixBase
  {
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase: Specialization for block-diagonal matrix
   ****************************************************************************/

  template<>
  struct InviscidFluxJacobiMatrixBase<SYSTEM_SEGREGATED>
  {
    /***************************************************************************
     * Compute flux Jacobian matrix for neqsim nodes
     **************************************************************************/
    template <int neqsim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
			     Tc *CoeffsAtDiag,
			     Td scale,
			     Td ui,
			     Td vi,
			     Ti ipos,
			     Ti ieq,
			     Ti neq,
			     Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ii = diag(A_i)*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
#else
      // Compute Galerkin coefficient $K_ii = -diag(A_i)*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,_);
#endif
    }

    /***************************************************************************
     * Compute flux Jacobian matrix for nedgesim edges
     **************************************************************************/
    template <int nedgesim, bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti ipos,
			     Ti iedge,
			     Ti nedge,
			     Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ij = diag(A_j)*C_{ji}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);

      // Compute Galerkin coefficient $K_ji = diag(A_i)*C_{ij}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
#else
      // Compute Galerkin coefficient $K_ij = -diag(A_j)*C_{ij}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,_);
      
      // Compute Galerkin coefficient $K_ji = -diag(A_i)*C_{ji}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,_);
#endif
    }
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrixBase: Specialization for full matrix
   ****************************************************************************/

  template <>
  struct InviscidFluxJacobiMatrixBase<SYSTEM_ALLCOUPLED>
  {
    /***************************************************************************
     * Compute flux Jacobian matrix for neqsim nodes
     **************************************************************************/
    template <int neqsim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
			     Tc *CoeffsAtDiag,
			     Td scale,
			     Td ui,
			     Td vi,
			     Td Ei,
			     Ti ipos,
			     Ti ieq,
			     Ti neq,
			     Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ii = A_i*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX21(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX31(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX41(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,5,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX12(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,6,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,7,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX32(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,8,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX42(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,9,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX13(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,10,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX23(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,11,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,12,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX43(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,13,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX14(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,14,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX24(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,15,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX34(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,16,ipos,NVAR2D*NVAR2D,neqsim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
#else
      // Compute Galerkin coefficient $K_ii = A_i*C_{ii}$
      IDX2(MatrixAtDiag,1,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,2,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX21(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,3,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX31(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,4,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX41(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,5,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX12(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,6,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,7,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX32(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,8,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX42(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,9,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX13(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,10,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX23(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,11,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,12,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX43(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,13,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX14(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,14,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX24(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,15,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX34(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
      IDX2(MatrixAtDiag,16,ipos,NVAR2D*NVAR2D,neqsim) = -
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,1,ieq,ncoeff,neq),
				   IDX2_COEFFSATDIAG(CoeffsAtDiag,2,ieq,ncoeff,neq),ui,vi,Ei);
#endif
    }

    /***************************************************************************
     * Compute flux Jacobian matrix for nedgesim edges
     **************************************************************************/
    template <int nedgesim, bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
#ifdef HYDRO_USE_IBP
      // Compute Galerkin coefficient $K_ij = A_j*C_{ji}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX21(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX31(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX41(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,5,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX12(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,6,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,7,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX32(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,8,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX42(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,9,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX13(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,10,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX23(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,11,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,12,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX43(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,13,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX14(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,14,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX24(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,15,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX34(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,16,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);

      // Compute Galerkin coefficient $K_ji = A_i*C_{ij}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX21(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX31(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX41(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,5,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX12(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,6,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,7,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX32(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,8,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX42(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,9,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX13(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,10,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX23(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,11,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,12,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX43(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,13,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX14(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,14,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX24(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,15,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX34(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,16,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
#else
      // Compute Galerkin coefficient $K_ij = -A_j*C_{ij}$
      IDX3(MatrixAtEdge,1,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,2,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX21(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,3,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX31(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,4,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX41(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,5,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX12(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,6,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,7,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX32(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,8,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX42(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,9,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX13(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,10,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX23(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,11,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,12,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX43(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,13,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX14(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,14,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX24(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,15,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX34(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      IDX3(MatrixAtEdge,16,1,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),uj,vj,Ej);
      
      // Compute Galerkin coefficient $K_ji = -A_i*C_{ji}$
      IDX3(MatrixAtEdge,1,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX11(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,2,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX21(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,3,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX31(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,4,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX41(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,5,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX12(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,6,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX22(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,7,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX32(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,8,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX42(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,9,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX13(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,10,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX23(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,11,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX33(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,12,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX43(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,13,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX14(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,14,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX24(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,15,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX34(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
      IDX3(MatrixAtEdge,16,2,ipos,NVAR2D*NVAR2D,(bstabilise ? 3 : 2),nedgesim) =
	INVISCIDFLUXJACOBIMATRIX44(-scale,
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),
				   IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),ui,vi,Ei);
#endif
    }
  };

  /*****************************************************************************
   * InviscidFluxJacobiMatrix
   ****************************************************************************/

  template <int isystemcoupling>
  struct InviscidFluxJacobiMatrix : public InviscidFluxJacobiMatrixBase<isystemcoupling>
  {
    // Enable use of inherited functions
    using InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData;
    using InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData;

    /**************************************************************************
     * Wrapper routine for processing a single node
     *************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
			     Tc *CoeffsAtDiag,
			     Td scale,
			     Td ui,
			     Td vi, 
			     Ti ieq,
			     Ti neq,
			     Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData<1>
	(MatrixAtDiag,CoeffsAtDiag,scale,ui,vi,1,ieq,neq,ncoeff);
    }

    /**************************************************************************
     * Wrapper routine for processing a single node
     *************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcNodeData(Td *MatrixAtDiag,
			     Tc *CoeffsAtDiag,
			     Td scale,
			     Td ui,
			     Td vi,
			     Td Ei,
			     Ti ieq,
			     Ti neq,
			     Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcNodeData<1>
	(MatrixAtDiag,CoeffsAtDiag,scale,ui,vi,Ei,1,ieq,neq,ncoeff);
    }
    
    /**************************************************************************
     * Wrapper routine for processing a single edge
     *************************************************************************/
    template <bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti iedge,
			     Ti nedge,
			     Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData<1,bstabilise>
	(MatrixAtEdge,CoeffsAtEdge,scale,ui,uj,vi,vj,1,iedge,nedge,ncoeff);
    }

    /**************************************************************************
     * Wrapper routine for processing a single edge
     *************************************************************************/
    template <bool bstabilise, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti iedge,
			     Ti nedge,
			     Ti ncoeff)
    {
      InviscidFluxJacobiMatrixBase<isystemcoupling>::calcEdgeData<1,bstabilise>
	(MatrixAtEdge,CoeffsAtEdge,scale,ui,uj,vi,vj,Ei,Ej,1,iedge,nedge,ncoeff);
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase (basic functionality individual specialisations)
   ****************************************************************************/
  
  template <int isystemcoupling, int idissipationtype>
  struct InviscidFluxDissipationMatrixBase
  {
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing zero artificial dissipation, aka standard Galerkin
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_ZERO>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
#pragma unroll
      for (int i=1; i<=NVAR2D; i++)
	IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = 0.0;
    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing zero artificial dissipation, aka standard Galerkin
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_ZERO>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
#pragma unroll
      for (int i=1; i<=NVAR2D*NVAR2D; i++)
	IDX3(MatrixAtEdge,i,1,ipos,NVAR2D*NVAR2D,3,nedgesim) = 0.0;
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing scalar artificial dissipation proportional to the
   * spectral radius of the Roe matrix
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_SCALAR>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);
    
      // Compute pressures
      Td pi = PRESSURE3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td pj = PRESSURE3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);

      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
      
      // Compute scalar dissipation
      Td d_ij = abs(vel_ij) + anorm*c_ij;

#pragma unroll
      for (int i=1; i<=NVAR2D; i++)
	IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = d_ij;
    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing scalar artificial dissipation proportional to the
   * spectral radius of the Roe matrix
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_SCALAR>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);
    
      // Compute pressures
      Td pi = PRESSURE3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim);
      Td pj = PRESSURE3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim);

      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,ipos,NVAR2D,2,nedgesim)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,ipos,NVAR2D,2,nedgesim)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
      
      // Compute scalar dissipation
      Td d_ij = abs(vel_ij) + anorm*c_ij;

#pragma unroll
      for (int i=1; i<=NVAR2D*NVAR2D; i++)
	IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = 0.0;

      for (int i=1; i<=NVAR2D*NVAR2D; i+=(i-1)*NVAR2D)
	IDX3(MatrixAtEdge,i,1,ipos,NVAR2D,3,nedgesim) = d_ij;
    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing tensorial artificial dissipation of Roe-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_ROE>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing tensorial artificial dissipation of Roe-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_ROE>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {

    }
  };

  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for block-diagonal matrix
   * computing scalar artificial dissipation of Rusanov-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_SEGREGATED,DISSIPATION_RUSANOV>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {

    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrixBase: Specialization for full matrix
   * computing scalar artificial dissipation of Rusanov-type
   ****************************************************************************/

  template <>  
  struct InviscidFluxDissipationMatrixBase<SYSTEM_ALLCOUPLED,DISSIPATION_RUSANOV>
  {
    template <int nedgesim, typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti ipos,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {

    }
  };
  
  /*****************************************************************************
   * InviscidFluxDissipationMatrix: Artificial dissipation
   ****************************************************************************/
  
  template <int isystemcoupling, int idissipationtype>
  struct InviscidFluxDissipationMatrix : public InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>
  {   
    // Enable use of inherited functions
    using InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData;
    
    /***************************************************************************
     * Wrapper routine for processing a single edge
     **************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
      InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData<1>
	(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,scale,ui,uj,vi,vj,1,iedge,nedge,ncoeff);
    }

    /***************************************************************************
     * Wrapper routine for processing a single edge
     **************************************************************************/
    template <typename Tc, typename Td, typename Ti>
    __device__ __forceinline__
    static void calcEdgeData(Td *MatrixAtEdge,
			     Tc *CoeffsAtEdge,
			     Td *DataAtEdge,
			     Td scale,
			     Td ui,
			     Td uj,
			     Td vi,
			     Td vj,
			     Td Ei,
			     Td Ej,
			     Ti iedge, 
			     Ti nedge,
			     Ti ncoeff)
    {
      InviscidFluxDissipationMatrixBase<isystemcoupling,idissipationtype>::calcEdgeData<1>
	(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,scale,ui,uj,vi,vj,Ei,Ej,1,iedge,nedge,ncoeff);
    }
  }; 
  
  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * block-diagonal global operator (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat>
  __global__ void hydro_calcMatDiagMatD2d_baseline(Tc *CoeffsAtDiag,
						   Ti *IdiagList,
						   Tv *vec,
						   Tm *mat,
						   Tm scale,
						   Ti neq,
						   Ti na,
						   Ti ncoeff,
						   Ti neq_last,
						   Ti neq_per_thread=1,
						   Ti neq_offset=0)
  {  
    // Loop over all items per thread
    for (int ipt=0; ipt<neq_per_thread; ++ipt) {
      
      // Global node ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+neq_offset+threadIdx.x;
      
      if (idx < neq_last) {
	// Get actual equation number
	Ti ieq = IDX2_DIAGLIST(IdiagList,1,idx+1,2,neq);
	
	// Local data at node from local memory
	Tm DataAtDiag[NVAR2D];
	
	// Get solution values at node
	Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	  gatherNodeData<true>(DataAtDiag,vec,ieq,neq);
	
	// Compute velocities
	Tm ui = XVELOCITY2(DataAtDiag,IDX2,1,NVAR2D,1);
	Tm vi = YVELOCITY2(DataAtDiag,IDX2,1,NVAR2D,1);
	
	// Compute Galerkin coefficient $K_ii$
	InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
	  calcNodeData(DataAtDiag,CoeffsAtDiag,scale,ui,vi,ieq,neq,ncoeff);
	
	// Get diagonal position in the global matrix
	Ti ia  = IDX2_DIAGLIST(IdiagList,2,idx+1,2,neq);
	
	// Build coefficients into global operator
	Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	  scatterNodeData<true>(mat,DataAtDiag,ia,na);
      }
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * block-diagonal global operator (cudaDMA implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int compute_threads_per_cta,
	    int dma_threads_per_ld>
  __global__ void hydro_calcMatDiagMatD2d_cudaDMA(Tc *CoeffsAtDiag,
						  Ti *IdiagList,
						  Tv *vec,
						  Tm *mat,
						  Tm scale,
						  Ti neq,
						  Ti na,
						  Ti ncoeff,
						  Ti neq_last,
						  Ti neq_per_thread=1,
						  Ti neq_offset=0)
  {
    // Not implemented yet
    printf("Not implemented\n");
  };
  
  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * full global operator (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat>
  __global__ void hydro_calcMatDiag2d_baseline(Tc *CoeffsAtDiag,
					       Ti *IdiagList,
					       Tv *vec,
					       Tm *mat,
					       Tm scale,
					       Ti neq,
					       Ti na,
					       Ti ncoeff,
					       Ti neq_last,
					       Ti neq_per_thread=1,
					       Ti neq_offset=0)
  {
    // Loop over all items per thread
    for (int ipt=0; ipt<neq_per_thread; ++ipt) {
      
      // Global node ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+neq_offset+threadIdx.x;
      
      if (idx < neq_last) {
	// Get actual equation
	Ti ieq = IDX2_DIAGLIST(IdiagList,1,idx+1,2,neq);
	
	// Local solution data at node from local memory
	Tm DataAtDiag[NVAR2D*NVAR2D];
	
	// Get solution values at node
	Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	  gatherNodeData<true>(DataAtDiag,vec,ieq,neq);
	
	// Compute velocities and energy
	Tm ui = XVELOCITY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
	Tm vi = YVELOCITY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
	Tm Ei = SPECIFICTOTALENERGY2(DataAtDiag,IDX2,1,NVAR2D*NVAR2D,1);
	
	// Compute Galerkin coefficient $K_ii$
	InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
	  calcNodeData(DataAtDiag,CoeffsAtDiag,scale,ui,vi,Ei,ieq,neq,ncoeff);
	
	// Get diagonal position in the global matrix
	Ti ia  = IDX2_DIAGLIST(IdiagList,2,idx+1,2,neq);
	
	// Build coefficients into global operator
	Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
	  scatterNodeData<true>(mat,DataAtDiag,ia,na);
      }
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the diagonal entries of the
   * full global operator (cudaDMA implementation).
   ****************************************************************************/
  
  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int compute_threads_per_cta,
	    int dma_threads_per_ld>
  __global__ void hydro_calcMatDiag2d_cudaDMA(Tc *CoeffsAtDiag,
					      Ti *IdiagList,
					      Tv *vec,
					      Tm *mat,
					      Tm scale,
					      Ti neq,
					      Ti na,
					      Ti ncoeff,
					      Ti neq_last,
					      Ti neq_per_thread=1,
					      Ti neq_offset=0)
  {
    // Not implemented yet
    printf("Not implemented\n");
  };

  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the
   * block-diagonal global operator and assembles the artificial
   * dissipation tensor if required (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int idissipation,
	    bool blumping>
  __global__ void hydro_calcMatrixMatD2d_baseline(Tc *CoeffsAtEdge,
						  Ti *IedgeList,
						  Tv *vec,
						  Tm *mat,
						  Tm scale,
						  Ti neq,
						  Ti na,
						  Ti nedge,
						  Ti ncoeff,
						  Ti nedge_last,
						  Ti nedge_per_thread=1,
						  Ti nedge_offset=0)
  {
    // Loop over all items per thread
    for (int ipt=0; ipt<nedge_per_thread; ++ipt) {
      
      // Global edge ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+nedge_offset+threadIdx.x;
      
      if (idx < nedge_last) {
	// Get positions of edge endpoints (idx starts at zero)
	Ti i = IDX2_EDGELIST(IedgeList,1,idx+1,6,nedge);
	Ti j = IDX2_EDGELIST(IedgeList,2,idx+1,6,nedge);
	
	// Local solution data at edge from local memory
	Tv DataAtEdge[2*NVAR2D];
	
	// Get solution values at edge endpoints
	Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	  gatherEdgeData<true>(DataAtEdge,vec,i,j,neq);
	
	// Compute velocities
	Tv ui = XVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	Tv vi = YVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	
	Tv uj = XVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	Tv vj = YVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	
	if (idissipation == DISSIPATION_ZERO) {
	  
	  // Local matrix data at edge from local memory
	  Tm MatrixAtEdge[2*NVAR2D];
	  
	  // Compute Galerkin coefficient $K_ij$ and $K_ji$
	  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
	    calcEdgeData<false>(MatrixAtEdge,CoeffsAtEdge,
				scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
	  // Build matrix coefficients into global operator
	  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	    scatterEdgeData<true,false,blumping>(mat,MatrixAtEdge,
						 IedgeList,idx+1,nedge,na);
	  
	} 
	else {
	  
	  // Local matrix data at edge from local memory
	  Tm MatrixAtEdge[3*NVAR2D];
	  
	  // Compute Galerkin coefficient $K_ij$ and $K_ji$
	  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
	    calcEdgeData<true>(MatrixAtEdge,CoeffsAtEdge,
			       scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
	  // Compute contribution of artificial diffusion
	  InviscidFluxDissipationMatrix<SYSTEM_SEGREGATED,idissipation>::
	    calcEdgeData(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,
			 scale,ui,uj,vi,vj,idx+1,nedge,ncoeff);
	  
	  // Build matrix coefficients into global operator
	  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	    scatterEdgeData<true,true,blumping>(mat,MatrixAtEdge,
						IedgeList,idx+1,nedge,na);
	}
      }
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the
   * block-diagonal global operator and assembles the artificial
   * dissipation tensor if required (cudaDMA implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int idissipation,
	    bool blumping,
	    int compute_threads_per_cta,
	    int dma_threads_per_ld>
  __global__ void hydro_calcMatrixMatD2d_cudaDMA(Tc *CoeffsAtEdge,
						 Ti *IedgeList,
						 Tv *vec,
						 Tm *mat,
						 Tm scale,
						 Ti neq,
						 Ti na,
						 Ti nedge,
						 Ti ncoeff,
						 Ti nedge_last,
						 Ti nedge_per_thread=1,
						 Ti nedge_offset=0)
  {
    const int tid = threadIdx.x;
    const int total_threads_per_cta = compute_threads_per_cta+4*dma_threads_per_ld;
    
    // Local variables
    Tv ui,uj,vi,vj;

    // Shared memory
    __shared__ Ti s_IedgeList0[6*compute_threads_per_cta];
    __shared__ Ti s_IedgeList1[6*compute_threads_per_cta];
    __shared__ Tv s_DataAtEdge0[2*NVAR2D*compute_threads_per_cta];
    __shared__ Tv s_DataAtEdge1[2*NVAR2D*compute_threads_per_cta];
    __shared__ Tc s_CoeffsAtEdge0[2*HYDRO_NDIM*compute_threads_per_cta];
    __shared__ Tc s_CoeffsAtEdge1[2*HYDRO_NDIM*compute_threads_per_cta];

    //--------------------------------------------------------------------------

#if EDGELIST_DEVICE == SOA
    // List of edges is stored as structure of arrays, that is, we
    // have 6 integer subarrays of length nedge which store:
    //
    // 0-subarray: first end point i, 
    // 1-subarray: second end point j,
    // 2-subarray: matrix entry ij,
    // 3-subarray: matrix entry ji,
    // 4-subarray: matrix entry ii,
    // 5-subarray: matrix entry jj.

    // Strided cudaDMA thread to transfer edge list from integer
    // array IedgeList into shared memory s_IedgeList0 and s_IedgeList1
    cudaDMAStrided<false, sizeof(Ti), compute_threads_per_cta*sizeof(Ti),
      total_threads_per_cta, 6>dma_ind(nedge*sizeof(Ti));
#else
    // List of edges is stored as array of structures, that is, we
    // have nedge integer subarrays of length 6 which store data
    //
    // (i,j,ij,jj,ii,jj)
    
    // We need two extra buffers in shared memory
    __shared__ Ti s_Idx0[2*compute_threads_per_cta];
    __shared__ Ti s_Idx1[2*compute_threads_per_cta];

    // Sequential cudaDMA thread to transfer edge list from integer
    // array IedgeList into shared memory s_IedgeList0 and s_IedgeList1
    
    cudaDMASequential<false, sizeof(Ti), 6*compute_threads_per_cta*sizeof(Ti),
                      total_threads_per_cta>dma_ind;
#endif

    //--------------------------------------------------------------------------

    // Indirect cudaDMA thread to transfer nodal data from vec into
    // shared memory s_DataAtEdge0, we need to distinguish between vec
    // stored in interleaved format and vec stored in block format
    cudaDMAIndirect<true, true,
      MAXALIGN((isystemformat==SYSTEM_BLOCK ? 1 : NVAR2D)*sizeof(Tv)),
               (isystemformat==SYSTEM_BLOCK ? 1 : NVAR2D)*sizeof(Tv),
      dma_threads_per_ld, compute_threads_per_cta*2>
    dma_vec0(0, compute_threads_per_cta, compute_threads_per_cta);
  
    // Indirect cudaDMA thread to transfer nodal data from vec into
    // shared memory s_DataAtEdge1, we need to distinguish between vec
    // stored in interleaved format and vec stored in block format
    cudaDMAIndirect<true, true,
      MAXALIGN((isystemformat==SYSTEM_BLOCK ? 1 : NVAR2D)*sizeof(Tv)),
               (isystemformat==SYSTEM_BLOCK ? 1 : NVAR2D)*sizeof(Tv),
      dma_threads_per_ld, compute_threads_per_cta*2>
    dma_vec1(1, compute_threads_per_cta, compute_threads_per_cta+dma_threads_per_ld);

    //--------------------------------------------------------------------------

#if COEFFSATEDGE_DEVICE == SOA
    // Coefficients at edges are stored as structure of arrays, that
    // is, we have ncoeff subarrays of length nedge which store:
    //
    // 0-subarray: coefficients for x-direction, 
    // 1-subarray: coefficients for y-direction,
    // n-subarray: further coefficients not required here

    // Strided cudaDMA thread to transfer precomputed coefficients
    // CoeffsAtEdge into shared memory s_CoeffsAtEdge0 and s_CoeffsAtEdge1
    cudaDMAStrided<true, sizeof(Tc), compute_threads_per_cta*sizeof(Tc),
                   dma_threads_per_ld, HYDRO_NDIM>
      dma_coeff0(2, compute_threads_per_cta,
		 compute_threads_per_cta+2*dma_threads_per_ld,
		 nedge*sizeof(Tc));
    cudaDMAStrided<true, sizeof(Tc), compute_threads_per_cta*sizeof(Tc),
                   dma_threads_per_ld, HYDRO_NDIM>
      dma_coeff1(3, compute_threads_per_cta,
		 compute_threads_per_cta+3*dma_threads_per_ld,
		 nedge*sizeof(Tc));
#else
    // Coefficients at edges are stored as array of structure, that
    // is, we have nedge real-valued subarray of length ncoeff
    cudaDMAStrided<true, sizeof(Tc), sizeof(Tc), dma_threads_per_ld, compute_threads_per_cta>
      dma_coeff0(2, compute_threads_per_cta,
		 compute_threads_per_cta+2*dma_threads_per_ld,
		 ncoeff*sizeof(Tc), HYDRO_NDIM*sizeof(Tc));
    cudaDMAStrided<true, sizeof(Tc), sizeof(Tc), dma_threads_per_ld, compute_threads_per_cta>
      dma_coeff1(3, compute_threads_per_cta,
		 compute_threads_per_cta+3*dma_threads_per_ld,
		 ncoeff*sizeof(Tc), HYDRO_NDIM*sizeof(Tc));
#endif

    //--------------------------------------------------------------------------

    // Loop over all edge-groups to be processed by this block
    for (int ipt=0; ipt<nedge_per_thread; ipt+=2) {

      //------------------------------------------------------------------------
      // Load the indices with all threads - no warp specialisation
      //------------------------------------------------------------------------
      ptx_cudaDMA_barrier_blocking(9, total_threads_per_cta);
      
      // Buffer0
      dma_ind.execute_dma(&IedgeList[ (((ipt+0)*gridDim.x+blockIdx.x)*
				       compute_threads_per_cta+nedge_offset)*
				      (EDGELIST_DEVICE == SOA ? 1 : 6)], s_IedgeList0);
      // Buffer1
      dma_ind.execute_dma(&IedgeList[ (((ipt+1)*gridDim.x+blockIdx.x)*
				       compute_threads_per_cta+nedge_offset)*
				      (EDGELIST_DEVICE == SOA ? 1 : 6)], s_IedgeList1);
      ptx_cudaDMA_barrier_blocking(9, total_threads_per_cta);

#if EDGELIST_DEVICE == SOA    
#define s_Idx0 s_IedgeList0
#define s_Idx1 s_IedgeList1
#else
      // We need to extract the start and end points of edges from
      // s_IedgeListX into s_IdxX for use in indirect gather operation
      if (tid < compute_threads_per_cta) {
      	s_Idx0[tid] = s_IedgeList0[6*tid];
      	s_Idx0[tid+compute_threads_per_cta] = s_IedgeList0[6*tid+1];
      }
      else if (tid < 2*compute_threads_per_cta) {
      	s_Idx1[tid-compute_threads_per_cta] = s_IedgeList1[6*(tid-compute_threads_per_cta)];
      	s_Idx1[tid] = s_IedgeList1[6*(tid-compute_threads_per_cta)+1];
      }
      ptx_cudaDMA_barrier_blocking(9, total_threads_per_cta);
#endif

      //------------------------------------------------------------------------
      // Start warp specialisation
      //------------------------------------------------------------------------
      if (tid < compute_threads_per_cta) {
	
	// Get solution values at edge endpoints
	dma_vec0.start_async_dma();
	dma_vec1.start_async_dma();
	
	// Get precomputed coefficients at edges
	dma_coeff0.start_async_dma();
	dma_coeff1.start_async_dma();

	//----------------------------------------------------------------------
	// Buffer0, first pass
	//----------------------------------------------------------------------

	// Wait for solution values to be available
	dma_vec0.wait_for_dma_finish();

	// Compute velocities
	if (isystemformat==SYSTEM_BLOCK) {
	  ui = XVELOCITY3(s_DataAtEdge0,IDX3T,1,tid,NVAR2D,2,compute_threads_per_cta);
	  vi = YVELOCITY3(s_DataAtEdge0,IDX3T,1,tid,NVAR2D,2,compute_threads_per_cta);
	  
	  uj = XVELOCITY3(s_DataAtEdge0,IDX3T,2,tid,NVAR2D,2,compute_threads_per_cta);
	  vj = YVELOCITY3(s_DataAtEdge0,IDX3T,2,tid,NVAR2D,2,compute_threads_per_cta);
	}
	else {
	  ui = XVELOCITY3(s_DataAtEdge0,IDX3,1,tid,NVAR2D,2,compute_threads_per_cta);
	  vi = YVELOCITY3(s_DataAtEdge0,IDX3,1,tid,NVAR2D,2,compute_threads_per_cta);
	  
	  uj = XVELOCITY3(s_DataAtEdge0,IDX3,2,tid,NVAR2D,2,compute_threads_per_cta);
	  vj = YVELOCITY3(s_DataAtEdge0,IDX3,2,tid,NVAR2D,2,compute_threads_per_cta);
	}

	// Wait for precomputed coefficients to be available
	dma_coeff0.wait_for_dma_finish();
	
	if (idissipation == DISSIPATION_ZERO) {
	  
	  // Local matrix data at edge from local memory
	  Tm MatrixAtEdge[2*NVAR2D];
	  
	  // Compute Galerkin coefficient $K_ij$ and $K_ji$
	  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
	    calcEdgeData<false>(MatrixAtEdge,s_CoeffsAtEdge0,
				scale,ui,uj,vi,vj,tid+1,compute_threads_per_cta,2);
	  
	  // Build matrix coefficients into global operator
	  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	    scatterEdgeData<true,false,blumping>(mat,MatrixAtEdge,
	  					 s_IedgeList0,tid+1,compute_threads_per_cta,na);
	  
	} 
	else {
	  
	  // Local matrix data at edge from local memory
	  Tm MatrixAtEdge[3*NVAR2D];
	  
	  // Compute Galerkin coefficient $K_ij$ and $K_ji$
	  InviscidFluxJacobiMatrix<SYSTEM_SEGREGATED>::
	    calcEdgeData<true>(MatrixAtEdge,s_CoeffsAtEdge0,
			       scale,ui,uj,vi,vj,tid+1,compute_threads_per_cta,2);
	  
	  // Compute contribution of artificial diffusion
	  InviscidFluxDissipationMatrix<SYSTEM_SEGREGATED,idissipation>::
	    calcEdgeData(MatrixAtEdge,s_CoeffsAtEdge0,s_DataAtEdge0,
			 scale,ui,uj,vi,vj,tid+1,compute_threads_per_cta,2);
	  
	  // Build matrix coefficients into global operator
	  Matrix<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	    scatterEdgeData<true,true,blumping>(mat,MatrixAtEdge,
	  					s_IedgeList0,tid+1,compute_threads_per_cta,na);
	}
	
	
      	//----------------------------------------------------------------------
	// Buffer1, first pass
	//----------------------------------------------------------------------
	
	// Wait for solution values to be available
	dma_vec1.wait_for_dma_finish();

	// Compute velocities
	if (isystemformat==SYSTEM_BLOCK) {
	  ui = XVELOCITY3(s_DataAtEdge1,IDX3T,1,tid,NVAR2D,2,compute_threads_per_cta);
	  vi = YVELOCITY3(s_DataAtEdge1,IDX3T,1,tid,NVAR2D,2,compute_threads_per_cta);
	  
	  uj = XVELOCITY3(s_DataAtEdge1,IDX3T,2,tid,NVAR2D,2,compute_threads_per_cta);
	  vj = YVELOCITY3(s_DataAtEdge1,IDX3T,2,tid,NVAR2D,2,compute_threads_per_cta);
	}
	else {
	  ui = XVELOCITY3(s_DataAtEdge1,IDX3,1,tid,NVAR2D,2,compute_threads_per_cta);
	  vi = YVELOCITY3(s_DataAtEdge1,IDX3,1,tid,NVAR2D,2,compute_threads_per_cta);
	  
	  uj = XVELOCITY3(s_DataAtEdge1,IDX3,2,tid,NVAR2D,2,compute_threads_per_cta);
	  vj = YVELOCITY3(s_DataAtEdge1,IDX3,2,tid,NVAR2D,2,compute_threads_per_cta);
	}

	// Wait for precomputed coefficients to be available
	dma_coeff1.wait_for_dma_finish();
      }
      
      //------------------------------------------------------------------------
      // DMA warps
      //------------------------------------------------------------------------

      else if(dma_vec0.owns_this_thread()) {
      	// Indirect cudaDMA transfer of global vector into s_DataAtEdge0
      	if (isystemformat==SYSTEM_BLOCK) {
      	  // Transfer each block separately (index array is 1-based)
      	  for (int ivar=0; ivar<NVAR2D; ++ivar)
      	    dma_vec0.execute_dma(s_Idx0, &vec[ivar*neq]-1,
      				 &s_DataAtEdge0[ivar*compute_threads_per_cta]);
      	}
      	else {
      	  // Transfer all blocks simultaneously (index array is 1-based)
      	  dma_vec0.execute_dma(s_Idx0, vec-NVAR2D, s_DataAtEdge0);
      	}
      }
      
      else if(dma_vec1.owns_this_thread()) {
      	// Indirect cudaDMA transfer of global vector into s_DataAtEdge1
      	if (isystemformat==SYSTEM_BLOCK) {
      	  // Transfer each block separately (index array is 1-based)
      	  for (int ivar=0; ivar<NVAR2D; ++ivar)
      	    dma_vec1.execute_dma(s_Idx1, &vec[ivar*neq]-1,
      				 &s_DataAtEdge1[ivar*compute_threads_per_cta]);
      	}
      	else {
      	  // Transfer all blocks simultaneously (index array is 1-based)
      	  dma_vec1.execute_dma(s_Idx1, vec-NVAR2D, s_DataAtEdge1);
      	}
      }

      else if(dma_coeff0.owns_this_thread()) {
#if COEFFSATEDGE_DEVICE == SOA
	// Strided cudaDMA transfer of precomputed coefficients into s_CoeffsAtEdge0
	dma_coeff0.execute_dma(&CoeffsAtEdge[ (((ipt+0)*gridDim.x+blockIdx.x)*
					       compute_threads_per_cta+nedge_offset)],
			       s_CoeffsAtEdge0);
#else
	// Strided cudaDMA transfer of precomputed coefficients into s_CoeffsAtEdge0
	for (int idim=0; idim<HYDRO_NDIM; ++idim)
	  dma_coeff0.execute_dma(&CoeffsAtEdge[ (((ipt+0)*gridDim.x+blockIdx.x)*
						 compute_threads_per_cta+nedge_offset)+idim],
				 &s_CoeffsAtEdge0[idim]);
#endif
      }

      else if(dma_coeff1.owns_this_thread()) {
#if COEFFSATEDGE_DEVICE == SOA
	// Strided cudaDMA transfer of precomputed coefficients into s_CoeffsAtEdge1
	dma_coeff1.execute_dma(&CoeffsAtEdge[ (((ipt+1)*gridDim.x+blockIdx.x)*
					       compute_threads_per_cta+nedge_offset)],
			       s_CoeffsAtEdge1);
#else
	// Strided cudaDMA transfer of precomputed coefficients into s_CoeffsAtEdge1
	for (int idim=0; idim<HYDRO_NDIM; ++idim)
	  dma_coeff1.execute_dma(&CoeffsAtEdge[ (((ipt+1)*gridDim.x+blockIdx.x)*
						 compute_threads_per_cta+nedge_offset)+idim],
				 &s_CoeffsAtEdge1[idim]);
#endif
      }
    }
  };
  
  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the full
   * global operator and assembles the artificial dissipation tensor
   * if required (baseline implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int idissipation,
	    bool blumping>
  __global__ void hydro_calcMatrix2d_baseline(Tc *CoeffsAtEdge,
					      Ti *IedgeList,
					      Tv *vec,
					      Tm *mat,
					      Tm scale,
					      Ti neq,
					      Ti na,
					      Ti nedge,
					      Ti ncoeff,
					      Ti nedge_last,
					      Ti nedge_per_thread=1,
					      Ti nedge_offset=0)

  {
    // Loop over all items per thread
    for (int ipt=0; ipt<nedge_per_thread; ++ipt) {
      
      // Global edge ID
      Ti idx = (ipt*gridDim.x+blockIdx.x)*blockDim.x+nedge_offset+threadIdx.x;

      if (idx < nedge_last)
	{
	  // Get positions of edge endpoints (idx starts at zero)
	  Ti i = IDX2_EDGELIST(IedgeList,1,idx+1,6,nedge);
	  Ti j = IDX2_EDGELIST(IedgeList,2,idx+1,6,nedge);
	  
	  // Local solution data at edge from local memory
	  Tm DataAtEdge[2*NVAR2D];
	  
	  // Get solution values at edge endpoints
	  Vector<NVAR2D,isystemformat==SYSTEM_BLOCK>::
	    gatherEdgeData<true>(DataAtEdge,vec,i,j,neq);
	  
	  // Compute velocities
	  Tm ui = XVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	  Tm vi = YVELOCITY2(DataAtEdge,IDX2,1,NVAR2D,2);
	  
	  Tm uj = XVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	  Tm vj = YVELOCITY2(DataAtEdge,IDX2,2,NVAR2D,2);
	  
	  // Compute specific energies
	  Tm Ei = SPECIFICTOTALENERGY2(DataAtEdge,IDX2,1,NVAR2D,2);
	  Tm Ej = SPECIFICTOTALENERGY2(DataAtEdge,IDX2,2,NVAR2D,2);
	  
	  if (idissipation == DISSIPATION_ZERO) {
	    
	    // Local matrix data at edge from local memory
	    Tm MatrixAtEdge[2*NVAR2D*NVAR2D];
	    
	    // Compute Galerkin coefficient $K_ij$ and $K_ji$
	    InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
	      calcEdgeData<false>(MatrixAtEdge,CoeffsAtEdge,
				  scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
	    // // Build matrix coefficients into global operator
	    Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
	      scatterEdgeData<true,false,blumping>(mat,MatrixAtEdge,
						   IedgeList,idx+1,nedge,na);
	  } else {
	    
	    // Local matrix data at edge from local memory
	    Tm MatrixAtEdge[3*NVAR2D*NVAR2D];
	    
	    // Compute Galerkin coefficient $K_ij$ and $K_ji$
	    InviscidFluxJacobiMatrix<SYSTEM_ALLCOUPLED>::
	      calcEdgeData<true>(MatrixAtEdge,CoeffsAtEdge,
				 scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
	    // Compute contribution of artificial diffusion
	    InviscidFluxDissipationMatrix<SYSTEM_ALLCOUPLED,idissipation>::
	      calcEdgeData(MatrixAtEdge,CoeffsAtEdge,DataAtEdge,
			   scale,ui,uj,vi,vj,Ei,Ej,idx+1,nedge,ncoeff);
	    
	    // Build matrix coefficients into global operator
	    Matrix<NVAR2D*NVAR2D,isystemformat==SYSTEM_BLOCK>::
	      scatterEdgeData<true,true,blumping>(mat,MatrixAtEdge,
						  IedgeList,idx+1,nedge,na);
	  }
	}
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the off-diagonal entries of the full
   * global operator and assembles the artificial dissipation tensor
   * if required (cudaDMA implementation).
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int isystemformat,
	    int idissipation,
	    bool blumping,
	    int compute_threads_per_cta,
	    int dma_threads_per_ld>
  __global__ void hydro_calcMatrix2d_cudaDMA(Tc *CoeffsAtEdge,
					     Ti *IedgeList,
					     Tv *vec,
					     Tm *mat,
					     Tm scale,
					     Ti neq,
					     Ti na,
					     Ti nedge,
					     Ti ncoeff,
					     Ti nedge_last,
					     Ti nedge_per_thread=1,
					     Ti nedge_offset=0)
    
  {
    // Not implemented yet
    printf("Not implemented\n");
  };
  
  /*****************************************************************************
   * Internal C++ functions which invoke the CUDA kernels
   ****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti>
  inline
  int hydro_calcMatDiagMatD2d_cuda(__SIZET *d_CoeffsAtDiag,
				   __SIZET *d_IdiagList,
				   __SIZET *d_vec,
				   __SIZET *d_mat,
				   Tm scale,
				   Ti nblocks,
				   Ti neq,
				   Ti na,
				   Ti ncoeff,
				   hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();
    
    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of equations
    const int compute_threads_per_cta  = 32*0;
    const int dma_threads_per_ld       = 32*1;
    const int dma_lds                  = 1;
    const int neq_per_thread_cudaDMA   = 1;

    const int threads_per_cta_baseline = 32*1;
    const int neq_per_thread_baseline  = 1;
    
    int blocks, threads, neq_cudaDMA, neq_baseline;
    prepare_cudaDMA(devProp, neq, neq_per_thread_cudaDMA,
		    compute_threads_per_cta, dma_threads_per_ld,
		    dma_lds, &blocks, &threads, &neq_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, neq-neq_cudaDMA, neq_per_thread_baseline,
		     threads_per_cta_baseline, &blocks, &threads, &neq_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);
    
    Tv *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtDiag = (Tc*)(*d_CoeffsAtDiag);
    Ti *IdiagList = (Ti*)(*d_IdiagList);
        
    cout << "hydro_calcMatDiagMatD2d_cuda" 
	 << " nblocks=" << nblocks
	 << " neq=" << neq
	 << " CudaDMA:"
	 << " #blocks=" << grid_cudaDMA.x << ","
	 << grid_cudaDMA.y << "," << grid_cudaDMA.z 
	 << " #threads per block=" << block_cudaDMA.x 
	 << "," << block_cudaDMA.y << "," << block_cudaDMA.z 
	 << " Baseline:"
	 << " #blocks=" << grid_baseline.x << "," 
	 << grid_baseline.y << "," << grid_baseline.z 
	 << " #threads per block=" << block_baseline.x 
	 << "," << block_baseline.y << "," << block_baseline.z 
	 << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);
    
      hipEventRecord(start,stream);
      
      if (grid_cudaDMA.x>0)
      	// CudaDMA implementation
      	hydro_calcMatDiagMatD2d_cudaDMA
      	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,compute_threads_per_cta,dma_threads_per_ld>
      	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
      						       IdiagList,
      						       vec, mat, scale,
      						       neq, na, ncoeff,
						       neq_cudaDMA,
						       neq_per_thread_cudaDMA);
      if (grid_baseline.x>0)
      	// Baseline implementation
      	hydro_calcMatDiagMatD2d_baseline
      	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR>
      	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
      							 IdiagList,
      							 vec, mat, scale,
      							 neq, na, ncoeff,
							 neq, 
							 neq_per_thread_baseline,
							 neq_cudaDMA);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      __SIZET cmemPool[NVAR2D];
#pragma unroll
      for (int i=0; i<NVAR2D; i++)
	cmemPool[i] = d_mat[i*(NVAR2D+1)];
      
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), cmemPool,
      			      sizeof(__SIZET)*NVAR2D, 0,
      			      hipMemcpyHostToDevice,
      			      stream);
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");
      
      hipEventRecord(start,stream);

      if (grid_cudaDMA.x>0)
	// CudaDMA implementation
	hydro_calcMatDiagMatD2d_cudaDMA
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,compute_threads_per_cta,dma_threads_per_ld>
	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
						       IdiagList,
						       vec, mat, scale,
						       neq, na, ncoeff,
						       neq_cudaDMA,
						       neq_per_thread_cudaDMA);
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatDiagMatD2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
							 IdiagList,
							 vec, mat, scale,
							 neq, na, ncoeff,
							 neq,
							 neq_per_thread_baseline,
							 neq_cudaDMA);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    }

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Memory NEQ:   " << NVAR2D*neq*sizeof(Tv)/1000000.0f
	 << " MB" << " NEQ=" << neq << endl;
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    cout << "Bandwidth:    " << (4*NVAR2D*neq*sizeof(Tv)+
				 2*neq*sizeof(Ti))/1000000000.0f/elapsedTime*1000.0f
	 << " GB/s" << endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    coproc_checkErrors("hydro_calcMatDiagMatD2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti>
  inline
  int hydro_calcMatDiag2d_cuda(__SIZET *d_CoeffsAtDiag,
			       __SIZET *d_IdiagList,
			       __SIZET *d_vec,
			       __SIZET *d_mat,
			       Tm scale,
			       Ti nblocks,
			       Ti neq,
			       Ti na,
			       Ti ncoeff,
			       hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();

    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of equations
    const int compute_threads_per_cta  = 32*0;
    const int dma_threads_per_ld       = 32*1;
    const int dma_lds                  = 1;
    const int neq_per_thread_cudaDMA   = 1;

    const int threads_per_cta_baseline = 32*1;
    const int neq_per_thread_baseline  = 1;
    
    int blocks, threads, neq_cudaDMA, neq_baseline;
    prepare_cudaDMA(devProp, neq, neq_per_thread_cudaDMA,
		    compute_threads_per_cta, dma_threads_per_ld,
		    dma_lds, &blocks, &threads, &neq_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, neq-neq_cudaDMA, neq_per_thread_baseline,
		     threads_per_cta_baseline, &blocks, &threads, &neq_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);

    Tv *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtDiag = (Tc*)(*d_CoeffsAtDiag);
    Ti *IdiagList = (Ti*)(*d_IdiagList);
  
    cout << "hydro_calcMatDiag2d_cuda" 
	 << " nblocks=" << nblocks
	 << " neq=" << neq
	 << " CudaDMA:"
	 << " #blocks=" << grid_cudaDMA.x << ","
	 << grid_cudaDMA.y << "," << grid_cudaDMA.z 
	 << " #threads per block=" << block_cudaDMA.x 
	 << "," << block_cudaDMA.y << "," << block_cudaDMA.z 
	 << " Baseline:"
	 << " #blocks=" << grid_baseline.x << "," 
	 << grid_baseline.y << "," << grid_baseline.z 
	 << " #threads per block=" << block_baseline.x 
	 << "," << block_baseline.y << "," << block_baseline.z 
	 << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);
      
      hipEventRecord(start,stream);

      if (grid_cudaDMA.x>0)
	//CudaDMA implementation
	hydro_calcMatDiag2d_cudaDMA
	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,compute_threads_per_cta,dma_threads_per_ld>
	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
						       IdiagList,
						       vec, mat, scale,
						       neq, na, ncoeff,
						       neq_cudaDMA,
						       neq_per_thread_cudaDMA);
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatDiag2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
							 IdiagList,
							 vec, mat, scale,
							 neq, na, ncoeff,
							 neq,
							 neq_per_thread_baseline,
							 neq_cudaDMA);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    } else {
      hipEventRecord(start,stream);
      
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), d_mat,
			      sizeof(__SIZET)*NVAR2D*NVAR2D, 0,
			      hipMemcpyHostToDevice,
			      stream);
      
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

      if (grid_cudaDMA.x>0)
	// CudaDMA implementation
	hydro_calcMatDiag2d_cudaDMA
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,compute_threads_per_cta,dma_threads_per_ld>
	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtDiag,
						       IdiagList,
						       vec, mat, scale,
						       neq, na, ncoeff,
						       neq_cudaDMA,
						       neq_per_thread_cudaDMA);
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatDiag2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtDiag,
							 IdiagList,
							 vec, mat, scale,
							 neq, na, ncoeff,
							 neq,
							 neq_per_thread_baseline,
							 neq_cudaDMA);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    }
        
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Memory NEQ:   " << NVAR2D*NVAR2D*neq*sizeof(Tv)/1000000.0f
	 << " MB" << " NEQ=" << neq << endl;
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    cout << "Bandwidth:    " << (NVAR2D*neq*sizeof(Tv)+
				 3*NVAR2D*NVAR2D*neq*sizeof(Tv)+
				 2*neq*sizeof(Ti))/1000000000.0f/elapsedTime*1000.0f
	 << " GB/s" << endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    coproc_checkErrors("hydro_calcMatDiag2d_cuda");
    return 0;
  };

  /*****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int idissipationtype,
	    bool blumping>
  inline
  int hydro_calcMatrixMatD2d_cuda(__SIZET *d_CoeffsAtEdge,
				  __SIZET *d_IedgeList,
				  __SIZET *d_vec,
				  __SIZET *d_mat,
				  Tm scale,
				  Ti nblocks,
				  Ti neq,
				  Ti na,
				  Ti nedge,
				  Ti ncoeff,
				  Ti nedgeset,
				  Ti iedgeset,
				  hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();

    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta  = 32*0;
    const int dma_threads_per_ld       = 32*1;
    const int dma_lds                  = 4;
    const int nedge_per_thread_cudaDMA = 2;

    const int threads_per_cta_baseline  = 32*1;
    const int nedge_per_thread_baseline = 1;
    
    int blocks, threads, nedge_cudaDMA, nedge_baseline;
    prepare_cudaDMA(devProp, nedgeset, nedge_per_thread_cudaDMA,
		    compute_threads_per_cta, dma_threads_per_ld,
		    dma_lds, &blocks, &threads, &nedge_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, nedgeset-nedge_cudaDMA, nedge_per_thread_baseline,
		     threads_per_cta_baseline, &blocks, &threads, &nedge_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);
    
    Tv  *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);

    cout << "hydro_calcMatrixMatD2d_cuda" 
	 << " nblocks=" << nblocks
	 << " nedgeset=" << nedgeset
	 << " CudaDMA:"
	 << " #blocks=" << grid_cudaDMA.x << ","
	 << grid_cudaDMA.y << "," << grid_cudaDMA.z 
	 << " #threads per block=" << block_cudaDMA.x 
	 << "," << block_cudaDMA.y << "," << block_cudaDMA.z 
	 << " Baseline:"
	 << " #blocks=" << grid_baseline.x << "," 
	 << grid_baseline.y << "," << grid_baseline.z 
	 << " #threads per block=" << block_baseline.x 
	 << "," << block_baseline.y << "," << block_baseline.z 
	 << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);

      hipEventRecord(start,stream);

      // if (grid_cudaDMA.x>0)
      // 	// CudaDMA implementation
      // 	hydro_calcMatrixMatD2d_cudaDMA
      // 	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,
      // 	   compute_threads_per_cta,dma_threads_per_ld>
      // 	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
      // 						       IedgeList,
      // 						       vec, mat, scale,
      // 						       neq, na, nedge, ncoeff,
      // 						       nedge_cudaDMA+iedgeset-1, 
      // 						       nedge_per_thread_cudaDMA,
      // 						       iedgeset-1);
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatrixMatD2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
							 IedgeList,
							 vec, mat, scale,
							 neq, na, nedge, ncoeff,
							 nedgeset+iedgeset-1, 
							 nedge_per_thread_baseline,
							 nedge_cudaDMA+iedgeset-1);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);  
    } else {
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      __SIZET cmemPool[NVAR2D];
#pragma unroll
      for (int i=0; i<NVAR2D; i++)
	cmemPool[i] = d_mat[i*(NVAR2D+1)];
      
      hipEventRecord(start,stream);
      
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), cmemPool,
			      sizeof(__SIZET)*NVAR2D, 0,
			      hipMemcpyHostToDevice,
			      stream);
      
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

      // if (grid_cudaDMA.x>0)
      // 	// CudaDMA implementation
      // 	hydro_calcMatrixMatD2d_cudaDMA
      // 	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,
      // 	   compute_threads_per_cta,dma_threads_per_ld>
      // 	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
      // 						       IedgeList,
      // 						       vec, mat, scale,
      // 						       neq, na, nedge, ncoeff,
      // 						       nedge_cudaDMA+iedgeset-1, 
      // 						       nedge_per_thread_cudaDMA,
      // 						       iedgeset-1);
      if (grid_baseline.x>0)
	hydro_calcMatrixMatD2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
							 IedgeList,
							 vec, mat, scale,
							 neq, na, nedge, ncoeff,
							 nedgeset+iedgeset-1,
							 nedge_per_thread_baseline,
							 nedge_cudaDMA+iedgeset-1);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    }

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Memory NEDGE: " << NVAR2D*nedgeset*sizeof(Tv)/1000000.0f
	 << " MB" << " NEDGE=" << nedgeset << endl;
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    cout << "Bandwidth:    " << (6*nedgeset*sizeof(Ti)+
				 3*NVAR2D*nedgeset*sizeof(Tv))/1000000000.0f/elapsedTime*1000.0f
	 << " GB/s" << endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    coproc_checkErrors("hydro_calcMatrixMatD2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename Tv,
	    typename Tm,
	    typename Ti,
	    int idissipationtype,
	    bool blumping>
  inline
  int hydro_calcMatrix2d_cuda(__SIZET *d_CoeffsAtEdge,
			      __SIZET *d_IedgeList,
			      __SIZET *d_vec,
			      __SIZET *d_mat,
			      Tm scale,
			      Ti nblocks,
			      Ti neq,
			      Ti na,
			      Ti nedge,
			      Ti ncoeff,
			      Ti nedgeset,
			      Ti iedgeset,
			      hipStream_t stream=0)
  {
    const hipDeviceProp_t *devProp = coproc_getCurrentDeviceProp();

    // Strategy: run the largest possible number of blocks with a
    // predefined number of compute/dma threads per block and let each
    // compute thread process the minimal number of edges
    const int compute_threads_per_cta  = 32*0;
    const int dma_threads_per_ld       = 32*1;
    const int dma_lds                  = 1;
    const int nedge_per_thread_cudaDMA = 1;

    const int threads_per_cta_baseline  = 32*1;
    const int nedge_per_thread_baseline = 1;
    
    int blocks, threads, nedge_cudaDMA, nedge_baseline;
    prepare_cudaDMA(devProp, nedgeset, nedge_per_thread_cudaDMA,
		    compute_threads_per_cta, dma_threads_per_ld,
		    dma_lds, &blocks, &threads, &nedge_cudaDMA);
    dim3 grid_cudaDMA(blocks, 1, 1);
    dim3 block_cudaDMA(threads, 1, 1);

    prepare_baseline(devProp, nedgeset-nedge_cudaDMA, nedge_per_thread_baseline,
		     threads_per_cta_baseline, &blocks, &threads, &nedge_baseline);
    dim3 grid_baseline(blocks, 1, 1);
    dim3 block_baseline(threads, 1, 1);

    Tv  *vec = (Tv*)(*d_vec);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);

    cout << "hydro_calcMatrix2d_cuda" 
	 << " nblocks=" << nblocks
	 << " nedgeset=" << nedgeset
	 << " CudaDMA:"
	 << " #blocks=" << grid_cudaDMA.x << ","
	 << grid_cudaDMA.y << "," << grid_cudaDMA.z 
	 << " #threads per block=" << block_cudaDMA.x 
	 << "," << block_cudaDMA.y << "," << block_cudaDMA.z 
	 << " Baseline:"
	 << " #blocks=" << grid_baseline.x << "," 
	 << grid_baseline.y << "," << grid_baseline.z 
	 << " #threads per block=" << block_baseline.x 
	 << "," << block_baseline.y << "," << block_baseline.z 
	 << endl;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (nblocks == 1) {
      // Matrix is store in interleaved matrix so that all matrix data
      // are stored contiguously in one single device memory block
      Tm *mat = (Tm*)(*d_mat);

      hipEventRecord(start,stream);

      if (grid_cudaDMA.x>0)
	// CudaDMA implementation
	hydro_calcMatrix2d_cudaDMA
	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping,
	   compute_threads_per_cta,dma_threads_per_ld>
	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
						       IedgeList,
						       vec, mat, scale,
						       neq, na, nedge, ncoeff,
						       nedge_cudaDMA+iedgeset-1, 
						       nedge_per_thread_cudaDMA,
						       iedgeset-1);
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatrix2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_SCALAR,idissipationtype,blumping>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
							 IedgeList,
							 vec, mat, scale,
							 neq, na, nedge, ncoeff,
							 nedgeset+iedgeset-1, 
							 nedge_per_thread_baseline,
							 nedge_cudaDMA+iedgeset-1);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    } else {
      hipEventRecord(start,stream);
      
      // Matrix is stored in block format, that is, the data of each
      // scalar submatrix resides in an individual device memory
      // block; thus we transfer the starting addresses of each memory
      // block into constant device memory and pass a dummy argument
      hipMemcpyToSymbolAsync(HIP_SYMBOL("constMemPool"), d_mat,
			      sizeof(__SIZET)*NVAR2D*NVAR2D, 0,
			      hipMemcpyHostToDevice,
			      stream);
      
      Tm *mat;
      hipGetSymbolAddress(((void**)&mat), "constMemPool");

      if (grid_cudaDMA.x>0)
	// CudaDMA implementation
	hydro_calcMatrix2d_cudaDMA
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping,
	   compute_threads_per_cta,dma_threads_per_ld>
	  <<<grid_cudaDMA, block_cudaDMA, 0, stream>>>(CoeffsAtEdge,
						       IedgeList,
						       vec, mat, scale,
						       neq, na, nedge, ncoeff,
						       nedge_cudaDMA+iedgeset-1, 
						       nedge_per_thread_cudaDMA,
						       iedgeset-1);
      
      if (grid_baseline.x>0)
	// Baseline implementation
	hydro_calcMatrix2d_baseline
	  <Tc,Tv,Tm,Ti,SYSTEM_BLOCK,idissipationtype,blumping>
	  <<<grid_baseline, block_baseline, 0, stream>>>(CoeffsAtEdge,
							 IedgeList,
							 vec, mat, scale,
							 neq, na, nedge, ncoeff,
							 nedgeset+iedgeset-1, 
							 nedge_per_thread_baseline,
							 nedge_cudaDMA+iedgeset-1);
      
      hipEventRecord(stop,stream);
      hipEventSynchronize(stop);
    }
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Memory NEDGE: " << NVAR2D*NVAR2D*nedgeset*sizeof(Tv)/1000000.0f
	 << " MB" << " NEDGE=" << nedgeset << endl;
    cout << "Elapsed time: " << elapsedTime << " ms" << endl;
    cout << "Bandwidth:    " << (6*nedgeset*sizeof(Ti)+
				 2*NVAR2D*nedgeset*sizeof(Tv)+
				 2*NVAR2D*NVAR2D*nedgeset*sizeof(Tv))/1000000000.0f/elapsedTime*1000.0f
	 << " GB/s" << endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    coproc_checkErrors("hydro_calcMatrix2d_cuda");
    return 0;
  };
  
  /*****************************************************************************
   * External C functions which can be called from the Fortran code
   ****************************************************************************/

  extern "C" {
    __INT FNAME(hydro_calcmatdiagmatd2d_cuda)(__SIZET *d_CoeffsAtDiag,
					      __SIZET *d_IdiagList,
					      __SIZET *d_vec,
					      __SIZET *d_mat,
					      __DP *scale,
					      __INT *nblocks,
					      __INT *neq,
					      __INT *na,
					      __INT *ncoeff,
					      __I64 *stream)
    {
      return (__INT) hydro_calcMatDiagMatD2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtDiag, d_IdiagList, d_vec, d_mat,
			       *scale, *nblocks, *neq, *na, *ncoeff,
			       (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatdiag2d_cuda)(__SIZET *d_CoeffsAtDiag,
					  __SIZET *d_IdiagList,
					  __SIZET *d_vec,
					  __SIZET *d_mat,
					  __DP *scale,
					  __INT *nblocks,
					  __INT *neq,
					  __INT *na,
					  __INT *ncoeff,
					  __I64 *stream)
    {
      return (__INT) hydro_calcMatDiag2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtDiag, d_IdiagList, d_vec, d_mat,
			       *scale, *nblocks, *neq, *na, *ncoeff,
			       (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatgalmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
					     __SIZET *d_IedgeList,
					     __SIZET *d_vec,
					     __SIZET *d_mat,
					     __DP *scale,
					     __INT *nblocks,
					     __INT *neq,
					     __INT *na,
					     __INT *nedge,
					     __INT *ncoeff,
					     __INT *nedgeset,
					     __INT *iedgeset,
					     __INT *cconstrType,
					     __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }
    
    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatgalerkin2d_cuda)(__SIZET *d_CoeffsAtEdge,
					      __SIZET *d_IedgeList,
					      __SIZET *d_vec,
					      __SIZET *d_mat,
					      __DP *scale,
					      __INT *nblocks,
					      __INT *neq,
					      __INT *na,
					      __INT *nedge,
					      __INT *ncoeff,
					      __INT *nedgeset,
					      __INT *iedgeset,
					      __INT *cconstrType,
					      __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ZERO,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatscdissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
					     __SIZET *d_IedgeList,
					     __SIZET *d_vec,
					     __SIZET *d_mat,
					     __DP *scale,
					     __INT *nblocks,
					     __INT *neq,
					     __INT *na,
					     __INT *nedge,
					     __INT *ncoeff,
					     __INT *nedgeset,
					     __INT *iedgeset,
					     __INT *cconstrType,
					     __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatscdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					    __SIZET *d_IedgeList,
					    __SIZET *d_vec,
					    __SIZET *d_mat,
					    __DP *scale,
					    __INT *nblocks,
					    __INT *neq,
					    __INT *na,
					    __INT *nedge,
					    __INT *ncoeff,
					    __INT *nedgeset,
					    __INT *iedgeset,
					    __INT *cconstrType,
					    __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_SCALAR,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatroedissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
						 __SIZET *d_IedgeList,
						 __SIZET *d_vec,
						 __SIZET *d_mat,
						 __DP *scale,
						 __INT *nblocks,
						 __INT *neq,
						 __INT *na,
						 __INT *nedge,
						 __INT *ncoeff,
						 __INT *nedgeset,
						 __INT *iedgeset,
						 __INT *cconstrType,
						 __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatroediss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					     __SIZET *d_IedgeList,
					     __SIZET *d_vec,
					     __SIZET *d_mat,
					     __DP *scale,
					     __INT *nblocks,
					     __INT *neq,
					     __INT *na,
					     __INT *nedge,
					     __INT *ncoeff,
					     __INT *nedgeset,
					     __INT *iedgeset,
					     __INT *cconstrType,
					     __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_ROE,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatrusdissmatd2d_cuda)(__SIZET *d_CoeffsAtEdge,
						 __SIZET *d_IedgeList,
						 __SIZET *d_vec,
						 __SIZET *d_mat,
						 __DP *scale,
						 __INT *nblocks,
						 __INT *neq,
						 __INT *na,
						 __INT *nedge,
						 __INT *ncoeff,
						 __INT *nedgeset,
						 __INT *iedgeset,
						 __INT *cconstrType,
						 __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrixMatD2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcmatrusdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					     __SIZET *d_IedgeList,
					     __SIZET *d_vec,
					     __SIZET *d_mat,
					     __DP *scale,
					     __INT *nblocks,
					     __INT *neq,
					     __INT *na,
					     __INT *nedge,
					     __INT *ncoeff,
					     __INT *nedgeset,
					     __INT *iedgeset,
					     __INT *cconstrType,
					     __I64 *stream)
    {
      if (*cconstrType == 0)
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,false>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
      else
	return (__INT) hydro_calcMatrix2d_cuda
	  <__DP,__DP,__DP,__INT,DISSIPATION_RUSANOV,true>
	  (d_CoeffsAtEdge, d_IedgeList, d_vec, d_mat,
	   *scale, *nblocks, *neq, *na, *nedge,
	   *ncoeff, *nedgeset, *iedgeset,
	   (hipStream_t)(*stream));
    }
  };
}
