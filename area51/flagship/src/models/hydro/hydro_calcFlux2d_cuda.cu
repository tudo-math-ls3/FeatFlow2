#include "hip/hip_runtime.h"
/*#############################################################################
 **************************************<****************************************
 * <name> hydro_calcFlux2d_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This file provides CUDA kernels to compute the fluxes for the low-order
 * scheme in 2D using different types if artificial viscosities.
 * </purpose>
 *
 *#############################################################################
 */

#include <cmath>
#include <cfloat>
#include <iostream>
#include <coproc_core.h>
#include <coproc_storage_cuda.h>
#include "../../cudaDMA.h"
#include "../../cudaGatherScatter.h"

#define LANGUAGE LANGUAGE_C
#include "../../flagship.h"

#define HYDRO_NDIM 2
#include "hydro.h"

// Number of compute threads per cooperative thread block (CTA)
#define COMPUTE_THREADS_PER_CTA (32 * 4) // multiple of warp size

// Number of DMA threads per load/store operation
#define DMA_THREADS_PER_LD      (32 * 1) // multiple of warp size

// Define short-hand IDX-macros
#if (EDGELIST_DEVICE == AOS)
#define IDX2_EDGELIST IDX2
#else
#define IDX2_EDGELIST IDX2T
#endif

#if (COEFFSATEDGE_DEVICE == AOS)
#define IDX3_COEFFSATEDGE IDX3
#else
#define IDX3_COEFFSATEDGE IDX3T
#endif


// Delete later !!!
#ifdef ENABLE_COPROC_SHMEM
#define SHMEM_IDX idx
#define SHMEM_BLOCKSIZE blockDim.x
#else
#define SHMEM_IDX 1
#define SHMEM_BLOCKSIZE 1
#endif

namespace hydro2d_cuda
{
  
  /*****************************************************************************
   * CUDA kernels for hydrodynamic model in 2D
   ****************************************************************************/

  using namespace std;

  /*****************************************************************************
   * This CUDA kernel collects the nodal solution data at the two
   * endpoints of the given edge from the global solution vector.
   ****************************************************************************/

  template <int isystemformat>
  struct gather_DataAtEdge
  { 
  };

  /*****************************************************************************
   * Input:  solution vector Dx stored in interleaved format
   * Output: DataAtEdge vector
   ****************************************************************************/

  template <>
  struct gather_DataAtEdge<SYSTEM_SCALAR>
  {
    template <typename TdSrc,
	      typename TdDest,
	      typename Ti>
    __device__ inline
    static void eval (TdDest *DataAtEdge,
		      TdSrc *Dx,
		      Ti i,
		      Ti j,
		      Ti neq,
		      Ti idx)
    {
      // Solution vector is stored in interleaved format
      
      // Gather solution data at first end point i
      IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,1,i,NVAR2D,neq);
      IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,2,i,NVAR2D,neq);
      IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,3,i,NVAR2D,neq);
      IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,4,i,NVAR2D,neq);

      // Gather solution data at second end point j
      IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,1,j,NVAR2D,neq);
      IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,2,j,NVAR2D,neq);
      IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,3,j,NVAR2D,neq);
      IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_REVERSE(Dx,4,j,NVAR2D,neq);
    }
  };
  
  /*****************************************************************************
   * Input:  solution vector Dx stored in block format
   * Output: DataAtEdge vector
   ****************************************************************************/

  template <>
  struct gather_DataAtEdge<SYSTEM_BLOCK>
  {
    template <typename TdSrc,
	      typename TdDest,
	      typename Ti>
    __device__ inline
    static void eval (TdDest *DataAtEdge,
		      TdSrc *Dx,
		      Ti i,
		      Ti j,
		      Ti neq,
		      Ti idx)
    {
      // Solution vector is stored in block format

      // Gather solution data at first end point i
      IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,1,i,NVAR2D,neq);
      IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,2,i,NVAR2D,neq);
      IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,3,i,NVAR2D,neq);
      IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,4,i,NVAR2D,neq);
    
      // Gather solution data at second end point j
      IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,1,j,NVAR2D,neq);
      IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,2,j,NVAR2D,neq);
      IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,3,j,NVAR2D,neq);
      IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = IDX2_FORWARD(Dx,4,j,NVAR2D,neq);
    }
  };

  /*****************************************************************************
   * This CUDA kernel scatters the fluxes into the global solution vector.
   ****************************************************************************/

  template <int isystemformat>
  struct scatter_FluxesAtEdge
  { 
  };

  /*****************************************************************************
   * Input:  FluxesAtEdge
   * Output: right-hand side vector Dy stored in interleaved format
   ****************************************************************************/

  template <>
  struct scatter_FluxesAtEdge<SYSTEM_SCALAR>
  {
    template <typename Td,
	      typename Ti>
    __device__ inline
    static void eval (Td *FluxesAtEdge,
		      Td *Dy,
		      Ti i,
		      Ti j,
		      Ti neq,
		      Ti idx)
    {
      // Solution vector is stored in interleaved format

      // Scatter flux to first node i
      IDX2_REVERSE(Dy,1,i,NVAR2D,neq) += IDX3(FluxesAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,2,i,NVAR2D,neq) += IDX3(FluxesAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,3,i,NVAR2D,neq) += IDX3(FluxesAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,4,i,NVAR2D,neq) += IDX3(FluxesAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Scatter flux to first node j
      IDX2_REVERSE(Dy,1,j,NVAR2D,neq) += IDX3(FluxesAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,2,j,NVAR2D,neq) += IDX3(FluxesAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,3,j,NVAR2D,neq) += IDX3(FluxesAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_REVERSE(Dy,4,j,NVAR2D,neq) += IDX3(FluxesAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    }
  };

  /*****************************************************************************
   * Input:  FluxesAtEdge
   * Output: right-hand side vector Dy stored in block format
   ****************************************************************************/

  template <>
  struct scatter_FluxesAtEdge<SYSTEM_BLOCK>
  {
    template <typename Td,
	      typename Ti>
    __device__ inline
    static void eval (Td *FluxesAtEdge,
		      Td *Dy,
		      Ti i,
		      Ti j,
		      Ti neq,
		      Ti idx)
    {
      // Solution vector is stored in block format

      // Scatter flux to first node i
      IDX2_FORWARD(Dy,1,i,NVAR2D,neq) += IDX3(FluxesAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,2,i,NVAR2D,neq) += IDX3(FluxesAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,3,i,NVAR2D,neq) += IDX3(FluxesAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,4,i,NVAR2D,neq) += IDX3(FluxesAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Scatter flux to first node j
      IDX2_FORWARD(Dy,1,j,NVAR2D,neq) += IDX3(FluxesAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,2,j,NVAR2D,neq) += IDX3(FluxesAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,3,j,NVAR2D,neq) += IDX3(FluxesAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX2_FORWARD(Dy,4,j,NVAR2D,neq) += IDX3(FluxesAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the Galerkin fluxes at the given edge.
   ****************************************************************************/

  struct calc_GalerkinFluxAtEdge
  {
    template <typename Td,
	      typename Ti>
#ifdef HYDRO_USE_IBP
    __device__ inline
    static void eval(Td *Fxi,
		     Td *Fxj,
		     Td *Fyi,
		     Td *Fyj,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti idx)
    {
      // Compute the Galerkin fluxes for x-direction
      IDX1(Fxi,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi);
      IDX1(Fxi,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi);
      IDX1(Fxi,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi);
      IDX1(Fxi,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi);
    
      IDX1(Fxj,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fxj,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fxj,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fxj,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
    
      // Compute Galerkin fluxes for y-direction
      IDX1(Fyi,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi);
      IDX1(Fyi,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi);
      IDX1(Fyi,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi);
      IDX1(Fyi,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi);
    
      IDX1(Fyj,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fyj,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fyj,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fyj,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
    }
#else
    __device__ inline
    static void eval(Td *Fx_ij,
		     Td *Fy_ij,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti idx)
    {
      // Compute Galerkin flux difference for x-direction
      IDX1(Fx_ij,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi)-
	              INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fx_ij,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi)-
      	              INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fx_ij,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi)-
	              INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      IDX1(Fx_ij,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,ui,pi)-
	              INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,uj,pj);
      
      // Compute Galerkin flux difference for y-direction
      IDX1(Fy_ij,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi)-
	              INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fy_ij,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi)-
	              INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fy_ij,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi)-
	              INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
      IDX1(Fy_ij,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vi,pi)-
	              INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE,vj,pj);
    }
#endif
  };

  /*****************************************************************************
   * This CUDA kernel calculates the artificial dissipation at the given edge.
   ****************************************************************************/

  template <int idissipationtype>
  struct calc_DissipationAtEdge
  {
  };

  /*****************************************************************************
   * Zero artificial dissipation, aka standard Galerkin approach
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_ZERO>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff,
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti iedge,
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      Diff[0] = 0.0;
      Diff[1] = 0.0;
      Diff[2] = 0.0;
      Diff[3] = 0.0;
    }
  };

  /*****************************************************************************
   * Scalar artificial dissipation proportional to the spectral radius
   * (largest eigenvector) of the cumulative Roe matrix.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_SCALAR>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff,
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti iedge,
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
    
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
    
      // Compute scalar dissipation
      Td d_ij = abs(vel_ij) + anorm*c_ij;
    
      // Multiply the solution difference by the scalar dissipation
      Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
    }
  };

  /*****************************************************************************
   * Scalar artificial dissipation proportional to the spectral radius
   * (largest eigenvector) of the dimensional-split Roe matrix.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_SCALAR_DSPLIT>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff,
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti iedge,
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pj)/rj;
    
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
    
      // Compute auxiliary variables
      Td q_ij = RCONST(0.5) *(u_ij * u_ij + v_ij * v_ij);
    
      // Compute the speed of sound
      Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON));
    
      // Compute scalar dissipation
      Td d_ij = ( abs(a[0]*u_ij) + abs(a[0])*c_ij +
		  abs(a[1]*v_ij) + abs(a[1])*c_ij );
    
      // Multiply the solution difference by the scalar dissipation
      Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
    }
  };

  /*****************************************************************************
   * Tensorial artificial dissipation of Roe-type.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_ROE>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff, 
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi, 
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti iedge, 
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {

      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      if (anorm > DBL_EPSILON) {
      
	// Normalise the skew-symmetric coefficient
	a[0] = a[0]/anorm;
	a[1] = a[1]/anorm;
      
	// Compute densities
	Td ri = DENSITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	Td rj = DENSITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute enthalpies
	Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pi)/ri;
	Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pj)/rj;
      
	// Compute Roe mean values
	Td aux  = ROE_MEAN_RATIO(ri,rj);
	Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
	Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
	Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
	// Compute auxiliary variables
	Td vel_ij = u_ij * a[0] + v_ij * a[1];
	Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
      
	// Compute the speed of sound
	Td c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON);
	Td c_ij  = sqrt(c2_ij);
      
	// Compute eigenvalues
	Td l1 = abs(vel_ij-c_ij);
	Td l2 = abs(vel_ij);
	Td l3 = abs(vel_ij+c_ij);
	Td l4 = abs(vel_ij);
      
	// Compute solution difference U_j-U_i
	Diff[0] = IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	         -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	Diff[1] = IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		 -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	Diff[2] = IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		 -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	Diff[3] = IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		 -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute auxiliary quantities for characteristic variables
	Td aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*Diff[0]
					      -u_ij*Diff[1]
					      -v_ij*Diff[2]
					           +Diff[3])/RCONST(2.0)/c2_ij;
	Td aux2 = (vel_ij*Diff[0]
		    -a[0]*Diff[1]
		    -a[1]*Diff[2])/RCONST(2.0)/c_ij;
      
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	Td w1 = l1 * (aux1 + aux2);
	Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*Diff[0]
		      +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*Diff[1]
						   +v_ij*Diff[2]
						        -Diff[3])/c2_ij);
	Td w3 = l3 * (aux1 - aux2);
	Td w4 = l4 * ((a[0]*v_ij-a[1]*u_ij)*Diff[0]
		                      +a[1]*Diff[1]
		                      -a[0]*Diff[2]);

	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] = anorm * ( w1 + w2 + w3 );
	Diff[1] = anorm * ( (u_ij-c_ij*a[0])*w1 + u_ij*w2 +
			    (u_ij+c_ij*a[0])*w3 + a[1]*w4 );
	Diff[2] = anorm * ( (v_ij-c_ij*a[1])*w1 + v_ij*w2 +
			    (v_ij+c_ij*a[1])*w3 - a[0]*w4 );
	Diff[3] = anorm * ( (H_ij-c_ij*vel_ij)*w1 + q_ij*w2 +
			    (H_ij+c_ij*vel_ij)*w3 + (u_ij*a[1]-v_ij*a[0])*w4 );
      
      } else {
	Diff[0] = 0.0;
	Diff[1] = 0.0;
	Diff[2] = 0.0;
	Diff[3] = 0.0;
      }
    }
  };

  /*****************************************************************************
   * Tensorial artificial dissipation of Roe-type using dimensional splitting.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_ROE_DSPLIT>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff,
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi,
		     Td pj,
		     Ti iedge, 
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      // Compute skew-symmetric coefficient
      Td a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
      Td anorm = sqrt(a[0] * a[0] + a[1] * a[1]);
    
      Td DiffAux[NVAR2D];
      if (anorm > DBL_EPSILON) {
      
	// Compute the absolute value
	a[0] = abs(a[0]);
	a[1] = abs(a[1]);
      
	// Compute densities
	Td ri = DENSITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	Td rj = DENSITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute enthalpies
	Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pi)/ri;
	Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)+pj)/rj;
      
	// Compute Roe mean values
	Td aux  = ROE_MEAN_RATIO(ri,rj);
	Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
	Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
	Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
	// Compute auxiliary variable
	Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
      
	// Compute the speed of sound
	Td c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), DBL_EPSILON);
	Td c_ij  = sqrt(c2_ij);
      
	//----------------------------------------------------------------------
	// Dimensional splitting: x-direction
	//----------------------------------------------------------------------
      
	// Compute eigenvalues
	Td l1 = abs(u_ij-c_ij);
	Td l2 = abs(u_ij);
	Td l3 = abs(u_ij+c_ij);
	Td l4 = abs(u_ij);
      
	// Compute solution difference U_j-U_i
	DiffAux[0] = IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[1] = IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[2] = IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[3] = IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute auxiliary quantities for characteristic variables
	Td aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffAux[0]
					      -u_ij*DiffAux[1]
					      -v_ij*DiffAux[2]
					           +DiffAux[3])/RCONST(2.0)/c2_ij;
	Td aux2 = (u_ij*DiffAux[0]
		   -DiffAux[1])/RCONST(2.0)/c_ij;
      
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	Td w1 = l1 * (aux1 + aux2);
	Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffAux[0]
		      +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffAux[1]
						   +v_ij*DiffAux[2]
						        -DiffAux[3])/c2_ij);
	Td w3 = l3 * (aux1 - aux2);
	Td w4 = l4 * (v_ij*DiffAux[0]
		      -DiffAux[2]);
        
	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] = a[0] * ( w1 + w2 + w3 );
	Diff[1] = a[0] * ( (u_ij-c_ij)*w1 + u_ij*w2 + (u_ij+c_ij)*w3 );
	Diff[2] = a[0] * (        v_ij*w1 + v_ij*w2 +        v_ij*w3 - w4 );
	Diff[3] = a[0] * ( (H_ij-c_ij*u_ij)*w1 + q_ij*w2 +
			   (H_ij+c_ij*u_ij)*w3 - v_ij*w4 );
      
	//----------------------------------------------------------------------
	// Dimensional splitting: y-direction
	//----------------------------------------------------------------------
      
	// Compute eigenvalues
	l1 = abs(v_ij-c_ij);
	l2 = abs(v_ij);
	l3 = abs(v_ij+c_ij);
	l4 = abs(v_ij);
      
	// Compute solution difference U_j-U_i
	DiffAux[0] = IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[1] = IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[2] = IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	DiffAux[3] = IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
	            -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute auxiliary quantities for characteristic variables
	aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffAux[0]
					   -u_ij*DiffAux[1]
					   -v_ij*DiffAux[2]
					        +DiffAux[3])/RCONST(2.0)/c2_ij;
	aux2 = (v_ij*DiffAux[0]
		-DiffAux[2])/RCONST(2.0)/c_ij;
      
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	w1 = l1 * (aux1 + aux2);
	w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffAux[0]
		   +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffAux[1]
						+v_ij*DiffAux[2]
						     -DiffAux[3])/c2_ij);
	w3 = l3 * (aux1 - aux2);
	w4 = l4 * (-u_ij*DiffAux[0]
		   +DiffAux[1]);
      
	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] += a[1] * ( w1 + w2 + w3 );
	Diff[1] += a[1] * (        u_ij*w1 + u_ij*w2 +        u_ij*w3 + w4 );
	Diff[2] += a[1] * ( (v_ij-c_ij)*w1 + v_ij*w2 + (v_ij+c_ij)*w3 );
	Diff[3] += a[1] * ( (H_ij-c_ij*v_ij)*w1 + q_ij*w2 +
			    (H_ij+c_ij*v_ij)*w3 + u_ij*w4 );
      
      } else {
	Diff[0] = 0.0;
	Diff[1] = 0.0;
	Diff[2] = 0.0;
	Diff[3] = 0.0;
      } 
    }
  };

  /*****************************************************************************
   * Scalar artificial dissipation of Rusanov-type.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_RUSANOV>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff,
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui, 
		     Td uj,
		     Td vi,
		     Td vj, 
		     Td pi,
		     Td pj,
		     Ti iedge, 
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      // Compute specific energies
      Td Ei = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      Td Ej = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Compute the speed of sound
      Td ci = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*
		       (HYDRO_GAMMA)*(Ei-RCONST(0.5)*(ui*ui+vi*vi)), DBL_EPSILON));
      Td cj = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*
		       (HYDRO_GAMMA)*(Ej-RCONST(0.5)*(uj*uj+vj*vj)), DBL_EPSILON));
    
#ifdef HYDRO_USE_IBP
      // Compute scalar dissipation based on the skew-symmetric part
      // which does not include the symmetric boundary contribution
      Td d_ij = max( abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*uj+
			 RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*vj)+
		     RCONST(0.5)*sqrt(POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
					  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
				      POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
					  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),2))*cj,
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*ui+
			 RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*vi)+
		     RCONST(0.5)*sqrt(POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
					  IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
				      POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
					  IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),2))*ci );
#else
      // Compute scalar dissipation
      Td d_ij = max( abs(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*uj+
			 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*vj)+
		     sqrt(POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
			  POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),2))*cj,
		     abs(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*ui+
			 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*vi)+
		     sqrt(POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
			  POW(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),2))*ci );
#endif
    
      // Multiply the solution difference by the scalar dissipation
      Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
    }
  };

  /*****************************************************************************
   * Scalar artificial dissipation of Rusanov-type using dimensional splitting.
   ****************************************************************************/

  template <>
  struct calc_DissipationAtEdge<DISSIPATION_RUSANOV_DSPLIT>
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
    __device__ inline
    static void eval(Td *Diff, 
		     Tc *CoeffsAtEdge,
		     Td *DataAtEdge,
		     Td ui,
		     Td uj,
		     Td vi,
		     Td vj,
		     Td pi, 
		     Td pj,
		     Ti iedge, 
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      // Compute specific energies
      Td Ei = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      Td Ej = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    
      // Compute the speed of sound
      Td ci = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*
		       (HYDRO_GAMMA)*(Ei-RCONST(0.5)*(ui*ui+vi*vi)), DBL_EPSILON));
      Td cj = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*
		       (HYDRO_GAMMA)*(Ej-RCONST(0.5)*(uj*uj+vj*vj)), DBL_EPSILON));
    
#ifdef HYDRO_USE_IBP
      // Compute scalar dissipation with dimensional splitting based on
      // the skew-symmetric part which does not include the symmetric
      // boundary contribution
      Td d_ij = max( abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*uj)+
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)))*cj,
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*ui)+
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)))*ci )
 	      + max( abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*vj)+
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)))*cj,
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*vi)+
		     abs(RCONST(0.5)*(IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				      IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)))*ci );
#else
      // Compute scalar dissipation with dimensional splitting
      Td d_ij = max( abs(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*uj)+
		     abs(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*cj,
		     abs(IDX3(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*ui)+
		     abs(IDX3(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*ci )
	      + max( abs(IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*vj)+
		     abs(IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*cj,
		     abs(IDX3(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*vi)+
		     abs(IDX3(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*ci );
#endif
    
      // Multiply the solution difference by the scalar dissipation
      Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
      Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE)
		     -IDX3(DataAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE));
    }
  };

  /*****************************************************************************
   * This CUDA kernel calculates the fluxes at the given edge.
   ****************************************************************************/

  struct calc_FluxesAtEdge
  {
    template <typename Tc,
	      typename Td,
	      typename Ti>
#ifdef HYDRO_USE_IBP
    __device__ inline
    static void eval(Td *FluxesAtEdge,
		     Tc *CoeffsAtEdge,
		     Td *Fxi,
		     Td *Fxj, 
		     Td *Fyi, 
		     Td *Fyj,
		     Td *Diff,
		     Td scale,
		     Ti iedge, 
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      IDX3(FluxesAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[0]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[0]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[0]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[0] + Diff[0]);
    
      IDX3(FluxesAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[1]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[1]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[1]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[1] + Diff[1]);
    
      IDX3(FluxesAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[2]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[2]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[2]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[2] + Diff[2]);
    
      IDX3(FluxesAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[3]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[3]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[3]-
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[3] + Diff[3]);
    
    
      IDX3(FluxesAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) =
     -IDX3(FluxesAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX3(FluxesAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) =
     -IDX3(FluxesAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX3(FluxesAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) =
     -IDX3(FluxesAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      IDX3(FluxesAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) =
     -IDX3(FluxesAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
    }
#else
    __device__ inline
    static void eval(Td *FluxesAtEdge,
		     Tc *CoeffsAtEdge,
		     Td *Fx_ij,
		     Td *Fy_ij,
		     Td *Diff,
		     Td scale,
		     Ti iedge,
		     Ti nedge,
		     Ti ncoeff,
		     Ti idx)
    {
      IDX3(FluxesAtEdge,1,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[0]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[0] + Diff[0]);
    
      IDX3(FluxesAtEdge,2,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[1]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[1] + Diff[1]);
    
      IDX3(FluxesAtEdge,3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[2]+
	 IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[2] + Diff[2]);
    
      IDX3(FluxesAtEdge,4,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = scale *
	(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[3]+
	 IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[3] + Diff[3]);
    
    
      IDX3(FluxesAtEdge,1,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = -scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[0]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[0] + Diff[0]);
    
      IDX3(FluxesAtEdge,2,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = -scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[1]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[1] + Diff[1]);
    
      IDX3(FluxesAtEdge,3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = -scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[2]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[2] + Diff[2]);
    
      IDX3(FluxesAtEdge,4,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE) = -scale *
	(IDX3_COEFFSATEDGE(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[3]+
	 IDX3_COEFFSATEDGE(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[3] + Diff[3]);
    }
#endif
  };

  /*****************************************************************************
   * This CUDA kernel calculates the inviscid fluxes.
   ****************************************************************************/
  
  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti,
	    int isystemformat,
	    int idissipationtype>
  __global__ void hydro_calcFlux2d_knl(Tc *CoeffsAtEdge,
				       Ti *IedgeList,
				       TdSrc *Dx,
				       TdDest *Dy,
				       TdDest scale,
				       Ti neq,
				       Ti nedge,
				       Ti ncoeff,
				       Ti nedges,
				       Ti iedgeset)
  {
#ifdef ENABLE_COPROC_SHMEM
    // Use shared memory
    extern __shared__ TdDest shmemData[];
#endif

    Ti idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    if (idx<nedges)
      {
	// Get positions of edge endpoints (idx starts at zero)
	Ti i = IDX2_EDGELIST(IedgeList,1,iedgeset+idx,6,nedge);
	Ti j = IDX2_EDGELIST(IedgeList,2,iedgeset+idx,6,nedge);
      
#ifdef ENABLE_COPROC_SHMEM
	// Local data at edge from shared memory
	TdDest *DataAtEdge = shmemData;
#else
	// Local data at edge from local memory
	TdDest DataAtEdge[2*NVAR2D];
#endif
      
	// Get solution values at edge endpoints
	gather_DataAtEdge<isystemformat>::
	  eval(DataAtEdge,Dx,i,j,neq,idx);
      
	// Compute velocities
	TdDest ui = XVELOCITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	TdDest vi = YVELOCITY3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	TdDest uj = XVELOCITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	TdDest vj = YVELOCITY3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
      
	// Compute pressures
	TdDest pi = PRESSURE3(DataAtEdge,IDX3,1,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);
	TdDest pj = PRESSURE3(DataAtEdge,IDX3,2,SHMEM_IDX,NVAR2D,2,SHMEM_BLOCKSIZE);

#ifdef HYDRO_USE_IBP
	TdDest Fxi[NVAR2D];
	TdDest Fxj[NVAR2D];
	TdDest Fyi[NVAR2D];
	TdDest Fyj[NVAR2D];
      
	// Compute the Galerkin fluxes
	calc_GalerkinFluxAtEdge::
	  eval(Fxi,Fxj,Fyi,Fyj,DataAtEdge,ui,uj,vi,vj,pi,pj,idx);
#else
	TdDest Fx_ij[NVAR2D];
	TdDest Fy_ij[NVAR2D];

	// Compute the Galerkin fluxes
	calc_GalerkinFluxAtEdge::
	  eval(Fx_ij,Fy_ij,DataAtEdge,ui,uj,vi,vj,pi,pj,idx);
#endif

	TdDest Diff[NVAR2D];
	// Compute the artificial viscosities
	calc_DissipationAtEdge<idissipationtype>::
	  eval(Diff,CoeffsAtEdge,DataAtEdge,
	       ui,uj,vi,vj,pi,pj,iedgeset+idx,nedge,ncoeff,idx);
      
	// Build both contributions into the fluxes
#ifdef HYDRO_USE_IBP
	calc_FluxesAtEdge::
	  eval(DataAtEdge,CoeffsAtEdge,Fxi,Fxj,Fyi,Fyj,Diff,
	       scale,iedgeset+idx,nedge,ncoeff,idx);
#else
	calc_FluxesAtEdge::
	  eval(DataAtEdge,CoeffsAtEdge,Fx_ij,Fy_ij,Diff,
	       scale,iedgeset+idx,nedge,ncoeff,idx);
#endif

	// Build fluxes into nodal vector
	scatter_FluxesAtEdge<isystemformat>::
	  eval(DataAtEdge,Dy,i,j,neq,idx);
      }
  };

  /*****************************************************************************
   * Internal C++ functions which invoke the CUDA kernels
   ****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxGalerkin2d_cuda(__SIZET *d_CoeffsAtEdge,
				    __SIZET *d_IedgeList,
				    __SIZET *d_Dx,
				    __SIZET *d_Dy,
				    TdDest scale,
				    Ti nblocks,
				    Ti neq,
				    Ti nedge, 
				    Ti ncoeff,
				    Ti nedges,
				    Ti iedgeset,
				    hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_ZERO>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale,
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_ZERO>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxGalerkin2d_cuda");
    return 0;
  }; 

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxScDiss2d_cuda(__SIZET *d_CoeffsAtEdge,
				  __SIZET *d_IedgeList,
				  __SIZET *d_Dx,
				  __SIZET *d_Dy,
				  TdDest scale,
				  Ti nblocks,
				  Ti neq,
				  Ti nedge, 
				  Ti ncoeff,
				  Ti nedges,
				  Ti iedgeset,
				  hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_SCALAR>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale,
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_SCALAR>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxScDiss2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxScDissDiSp2d_cuda(__SIZET *d_CoeffsAtEdge,
				      __SIZET *d_IedgeList,
				      __SIZET *d_Dx,
				      __SIZET *d_Dy,
				      TdDest scale,
				      Ti nblocks,
				      Ti neq, 
				      Ti nedge,
				      Ti ncoeff,
				      Ti nedges,
				      Ti iedgeset,
				      hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_SCALAR_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_SCALAR_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxScDissDiSp2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxRoeDiss2d_cuda(__SIZET *d_CoeffsAtEdge,
				   __SIZET *d_IedgeList,
				   __SIZET *d_Dx,
				   __SIZET *d_Dy,
				   TdDest scale,
				   Ti nblocks, 
				   Ti neq, 
				   Ti nedge,
				   Ti ncoeff,
				   Ti nedges, 
				   Ti iedgeset,
				   hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_ROE>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_ROE>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList,
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxRoeDiss2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxRoeDissDiSp2d_cuda(__SIZET *d_CoeffsAtEdge,
				       __SIZET *d_IedgeList,
				       __SIZET *d_Dx,
				       __SIZET *d_Dy,
				       TdDest scale,
				       Ti nblocks, 
				       Ti neq, 
				       Ti nedge,
				       Ti ncoeff,
				       Ti nedges, 
				       Ti iedgeset,
				       hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_ROE_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge, 
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff, 
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_ROE_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge, 
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxRoeDissDiSp2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxRusDiss2d_cuda(__SIZET *d_CoeffsAtEdge,
				   __SIZET *d_IedgeList,
				   __SIZET *d_Dx,
				   __SIZET *d_Dy,
				   TdDest scale,
				   Ti nblocks, 
				   Ti neq, 
				   Ti nedge, 
				   Ti ncoeff,
				   Ti nedges, 
				   Ti iedgeset,
				   hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_RUSANOV>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge,
				     IedgeList, 
				     Dx, Dy, scale,
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_RUSANOV>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge, 
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff,
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxRusDiss2d_cuda");
    return 0;
  };

  /****************************************************************************/

  template <typename Tc,
	    typename TdSrc,
	    typename TdDest,
	    typename Ti>
  inline
  int hydro_calcFluxRusDissDiSp2d_cuda(__SIZET *d_CoeffsAtEdge,
				       __SIZET *d_IedgeList,
				       __SIZET *d_Dx,
				       __SIZET *d_Dy,
				       TdDest scale,
				       Ti nblocks, 
				       Ti neq,
				       Ti nedge, 
				       Ti ncoeff,
				       Ti nedges, 
				       Ti iedgeset,
				       hipStream_t stream=0)
  {
    TdSrc  *Dx = (TdSrc*)(*d_Dx);
    TdDest *Dy = (TdDest*)(*d_Dy);
    Tc *CoeffsAtEdge = (Tc*)(*d_CoeffsAtEdge);
    Ti *IedgeList = (Ti*)(*d_IedgeList);
  
    // Define number of threads per block
    int blocksize = 128;
    dim3 grid;
    dim3 block;
    block.x = blocksize;
    grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
    if (nblocks == 1) {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_SCALAR,DISSIPATION_RUSANOV_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge, 
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff, 
				     nedges, iedgeset);
    } else {
      hydro_calcFlux2d_knl
	<Tc,TdSrc,TdDest,Ti,SYSTEM_BLOCK,DISSIPATION_RUSANOV_DSPLIT>
	<<<grid, block, 0, stream>>>(CoeffsAtEdge, 
				     IedgeList, 
				     Dx, Dy, scale, 
				     neq, nedge, ncoeff, 
				     nedges, iedgeset);
    }
    coproc_checkErrors("hydro_calcFluxRusDissDiSp2d_cuda");
    return 0;
  };

  /*****************************************************************************
   * External C functions which can be called from the Fortran code
   ****************************************************************************/

  extern "C"
  {
    __INT FNAME(hydro_calcfluxgalerkin2d_cuda)(__SIZET *d_CoeffsAtEdge,
					       __SIZET *d_IedgeList,
					       __SIZET *d_Dx,
					       __SIZET *d_Dy,
					       __DP *scale,
					       __INT *nblocks,
					       __INT *neq,
					       __INT *nedge,
					       __INT *ncoeff,
					       __INT *nedges,
					       __INT *iedgeset,
					       __I64 *stream)
    {
      return (__INT) hydro_calcFluxGalerkin2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset, 
			       (hipStream_t)(*stream));
    }

    /**************************************************************************/
    
    __INT FNAME(hydro_calcfluxscdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					     __SIZET *d_IedgeList,
					     __SIZET *d_Dx,
					     __SIZET *d_Dy,
					     __DP *scale,
					     __INT *nblocks,
					     __INT *neq,
					     __INT *nedge,
					     __INT *ncoeff,
					     __INT *nedges,
					     __INT *iedgeset,
					     __I64 *stream)
    {
      return (__INT) hydro_calcFluxScDiss2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset, 
			       (hipStream_t)(*stream));
    }

    /**************************************************************************/

    __INT FNAME(hydro_calcfluxscdissdisp2d_cuda)(__SIZET *d_CoeffsAtEdge,
						 __SIZET *d_IedgeList,
						 __SIZET *d_Dx,
						 __SIZET *d_Dy,
						 __DP *scale,
						 __INT *nblocks, 
						 __INT *neq, 
						 __INT *nedge, 
						 __INT *ncoeff,
						 __INT *nedges, 
						 __INT *iedgeset,
						 __I64 *stream)
    {
      return (__INT) hydro_calcFluxScDissDiSp2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset,
			       (hipStream_t)(*stream));
    }

    /**************************************************************************/

    __INT FNAME(hydro_calcfluxroediss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					      __SIZET *d_IedgeList,
					      __SIZET *d_Dx,
					      __SIZET *d_Dy,
					      __DP *scale,
					      __INT *nblocks, 
					      __INT *neq, 
					      __INT *nedge, 
					      __INT *ncoeff,
					      __INT *nedges, 
					      __INT *iedgeset,
					      __I64 *stream)
    {
      return (__INT) hydro_calcFluxRoeDiss2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset,
			       (hipStream_t)(*stream));
    }

  /***************************************************************************/

    __INT FNAME(hydro_calcfluxroedissdisp2d_cuda)(__SIZET *d_CoeffsAtEdge,
						  __SIZET *d_IedgeList,
						  __SIZET *d_Dx,
						  __SIZET *d_Dy,
						  __DP *scale,
						  __INT *nblocks, 
						  __INT *neq, 
						  __INT *nedge, 
						  __INT *ncoeff,
						  __INT *nedges, 
						  __INT *iedgeset,
						  __I64 *stream)
    {
      return (__INT) hydro_calcFluxRoeDissDiSp2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset,
			       (hipStream_t)*stream);
    }

    /**************************************************************************/

    __INT FNAME(hydro_calcfluxrusdiss2d_cuda)(__SIZET *d_CoeffsAtEdge,
					      __SIZET *d_IedgeList,
					      __SIZET *d_Dx,
					      __SIZET *d_Dy,
					      __DP *scale,
					      __INT *nblocks, 
					      __INT *neq, 
					      __INT *nedge, 
					      __INT *ncoeff,
					      __INT *nedges, 
					      __INT *iedgeset,
					      __I64 *stream)
    {
      return (__INT)hydro_calcFluxRusDiss2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset,
			       (hipStream_t)*stream);
    }
    
    /**************************************************************************/

    __INT FNAME(hydro_calcfluxrusdissdisp2d_cuda)(__SIZET *d_CoeffsAtEdge,
						  __SIZET *d_IedgeList,
						  __SIZET *d_Dx,
						  __SIZET *d_Dy,
						  __DP *scale,
						  __INT *nblocks, 
						  __INT *neq, 
						  __INT *nedge, 
						  __INT *ncoeff,
						  __INT *nedges, 
						  __INT *iedgeset,
						  __I64 *stream)
    {
      return (__INT) hydro_calcFluxRusDissDiSp2d_cuda
	<__DP,__DP,__DP,__INT>(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
			       *scale, *nblocks, *neq, *nedge,
			       *ncoeff, *nedges, *iedgeset,
			       (hipStream_t)*stream);
    }
  };

}
