#include "hip/hip_runtime.h"
/*#############################################################################
 ******************************************************************************
 * <name> hydro_calcFluxScDiss2d_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This CUDA kernel computes the fluxes for the low-order scheme in 2D
 * using tensorial artificial viscosities of Roe-type, whereby dimensional
 * splitting is employed.
 * </purpose>
 *
 *#############################################################################/
 */

#include <stdio.h>
#include <math.h>
#include <iostream>
#include "coproc_core.h"
#include "coproc_storage_cuda.h"

#define LANGUAGE LANGUAGE_C
#include "../../../../../kernel/System/idxmanager.h"

#define HYDRO_NDIM 2
#include "hydro.h"

extern "C"
{
  int hydro_calcFluxRoeDissDiSp2d_cuda(unsigned long * h_DcoeffsAtEdge,
				       unsigned long * h_IedgeList,
				       unsigned long * h_Dx,
				       unsigned long * h_Dy,
				       double * dscale,
				       int * nblocks,
				       int * neq,
				       int * nvar,
				       int * nedge,
				       int * nmatcoeff,
				       int * nedges,
				       int * iedgeset);
  int FNAME(hydro_calcfluxroedissdisp2d_cuda)(unsigned long * h_DcoeffsAtEdge,
					      unsigned long * h_IedgeList,
					      unsigned long * h_Dx,
					      unsigned long * h_Dy,
					      double * dscale,
					      int * nblocks,
					      int * neq,
					      int * nvar,
					      int * nedge,
					      int * nmatcoeff,
					      int * nedges,
					      int * iedgeset);
}

/*******************************************************************************/
template <int isystemformat>
__global__ void hydro_calcFluxRoeDissDiSp2d_knl(double * DcoeffsAtEdge,
						int * IedgeList,
						double * Dx,
						double * Dy,
						double dscale,
						int neq,
						int nvar,
						int nedge,
						int nmatcoeff,
						int nedges,
						int iedgeset)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx<nedges)
  {
    // Get positions of edge endpoints (idx starts at zero)
    int i = IDX2T(IedgeList,1,iedgeset+idx,6,nedge);
    int j = IDX2T(IedgeList,2,iedgeset+idx,6,nedge);

    // Get solution values at edge endpoints
    double DdataAtEdge[2*NVAR2D];

    if (isystemformat == 0) {
      // Solution vector is stored in interleaved format
      IDX2(DdataAtEdge,1,1,NVAR2D,2) = IDX2_REVERSE(Dx,1,i,NVAR2D,neq);
      IDX2(DdataAtEdge,2,1,NVAR2D,2) = IDX2_REVERSE(Dx,2,i,NVAR2D,neq);
      IDX2(DdataAtEdge,3,1,NVAR2D,2) = IDX2_REVERSE(Dx,3,i,NVAR2D,neq);
      IDX2(DdataAtEdge,4,1,NVAR2D,2) = IDX2_REVERSE(Dx,4,i,NVAR2D,neq);
      
      IDX2(DdataAtEdge,1,2,NVAR2D,2) = IDX2_REVERSE(Dx,1,j,NVAR2D,neq);
      IDX2(DdataAtEdge,2,2,NVAR2D,2) = IDX2_REVERSE(Dx,2,j,NVAR2D,neq);
      IDX2(DdataAtEdge,3,2,NVAR2D,2) = IDX2_REVERSE(Dx,3,j,NVAR2D,neq);
      IDX2(DdataAtEdge,4,2,NVAR2D,2) = IDX2_REVERSE(Dx,4,j,NVAR2D,neq);

    } else {
      // Solution vector is stored in block format
      IDX2(DdataAtEdge,1,1,NVAR2D,2) = IDX2_FORWARD(Dx,1,i,NVAR2D,neq);
      IDX2(DdataAtEdge,2,1,NVAR2D,2) = IDX2_FORWARD(Dx,2,i,NVAR2D,neq);
      IDX2(DdataAtEdge,3,1,NVAR2D,2) = IDX2_FORWARD(Dx,3,i,NVAR2D,neq);
      IDX2(DdataAtEdge,4,1,NVAR2D,2) = IDX2_FORWARD(Dx,4,i,NVAR2D,neq);

      IDX2(DdataAtEdge,1,2,NVAR2D,2) = IDX2_FORWARD(Dx,1,j,NVAR2D,neq);
      IDX2(DdataAtEdge,2,2,NVAR2D,2) = IDX2_FORWARD(Dx,2,j,NVAR2D,neq);
      IDX2(DdataAtEdge,3,2,NVAR2D,2) = IDX2_FORWARD(Dx,3,j,NVAR2D,neq);
      IDX2(DdataAtEdge,4,2,NVAR2D,2) = IDX2_FORWARD(Dx,4,j,NVAR2D,neq);
    }

    //--------------------------------------------------------------------------
    // Evaluate the Galerkin fluxes
    //--------------------------------------------------------------------------

    // Compute velocities
    double ui = XVELOCITY3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1);
    double vi = YVELOCITY3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1);

    double uj = XVELOCITY3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1);
    double vj = YVELOCITY3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1);

    // Compute pressures
    double pi = PRESSURE3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1);
    double pj = PRESSURE3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1);

#ifdef HYDRO_USE_IBP
    double Fxi[NVAR2D];
    double Fxj[NVAR2D];

    // Compute fluxes for x-direction
    IDX1(Fxi,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi);
    IDX1(Fxi,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi);
    IDX1(Fxi,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi);
    IDX1(Fxi,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi);

    IDX1(Fxj,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fxj,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fxj,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fxj,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);

    double Fyi[NVAR2D];
    double Fyj[NVAR2D];

    // Compute fluxes for x-direction
    IDX1(Fyi,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi);
    IDX1(Fyi,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi);
    IDX1(Fyi,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi);
    IDX1(Fyi,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi);
      
    IDX1(Fyj,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fyj,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fyj,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fyj,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
#else
    double Fx_ij[NVAR2D];
    
    // Compute flux difference for x-direction
    IDX1(Fx_ij,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi)-
                    INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fx_ij,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi)-
                    INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fx_ij,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi)-
                    INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);
    IDX1(Fx_ij,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,ui,pi)-
                    INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,uj,pj);

    double Fy_ij[NVAR2D];

    // Compute flux difference for y-direction
    IDX1(Fy_ij,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi)-
                    INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fy_ij,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi)-
                    INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fy_ij,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi)-
                    INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
    IDX1(Fy_ij,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1,vi,pi)-
                    INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1,vj,pj);
#endif

    //--------------------------------------------------------------------------
    // Evaluate the scalar dissipation tensor of Roe-type
    //--------------------------------------------------------------------------

    // Compute skew-symmetric coefficient
    double a[HYDRO_NDIM];
    a[0] = RCONST(0.5)*(IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)-
			IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge));
    a[1] = RCONST(0.5)*(IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)-
			IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge));
    double anorm = sqrt(a[0] * a[0] + a[1] * a[1]);

    double DiffX[NVAR2D];
    double DiffY[NVAR2D];
    if (anorm > 1e-14) {

      // Compute the absolute value
      a[0] = abs(a[0]);
      a[1] = abs(a[1]);
      
      // Compute densities
      double ri = DENSITY3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1);
      double rj = DENSITY3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1);
      
      // Compute enthalpies
      double hi = (TOTALENERGY3(DdataAtEdge,IDX3,1,1,NVAR2D,2,1)+pi)/ri;
      double hj = (TOTALENERGY3(DdataAtEdge,IDX3,2,1,NVAR2D,2,1)+pj)/rj;
      
      //! Compute Roe mean values
      double aux  = ROE_MEAN_RATIO(ri,rj);
      double u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      double v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      double H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variable
      double q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij);
      
      // Compute the speed of sound
      //TODO echtes double epsilon einbauen
      double c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14);
      double c_ij  = sqrt(c2_ij);
      
      //------------------------------------------------------------------------
      // Dimensional splitting: x-direction
      //------------------------------------------------------------------------
        
      // Compute eigenvalues
      double l1 = abs(u_ij-c_ij);
      double l2 = abs(u_ij);
      double l3 = abs(u_ij+c_ij);
      double l4 = abs(u_ij);
        
      // Compute solution difference U_j-U_i
      DiffX[0] = IDX3(DdataAtEdge,1,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,1,1,1,NVAR2D,2,1);
      DiffX[1] = IDX3(DdataAtEdge,2,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,2,1,1,NVAR2D,2,1);
      DiffX[2] = IDX3(DdataAtEdge,3,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,3,1,1,NVAR2D,2,1);
      DiffX[3] = IDX3(DdataAtEdge,4,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,4,1,1,NVAR2D,2,1);

      // Compute auxiliary quantities for characteristic variables
      double aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffX[0]
						-u_ij*DiffX[1]
						-v_ij*DiffX[2]
						     +DiffX[3])/RCONST(2.0)/c2_ij;
      double aux2 = (u_ij*DiffX[0]
		         -DiffX[1])/RCONST(2.0)/c_ij;
        
      // Compute characteristic variables multiplied by the corresponding eigenvalue
      double w1 = l1 * (aux1 + aux2);
      double w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffX[0]
			                  +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffX[1]
								       +v_ij*DiffX[2]
									     -DiffX[3])/c2_ij);
      double w3 = l3 * (aux1 - aux2);
      double w4 = l4 * (v_ij*DiffX[0]
			    -DiffX[2]);
        
      // Compute "R_ij * |Lbd_ij| * L_ij * dU"
      DiffX[0] = a[0] * ( w1 + w2 + w3 );
      DiffX[1] = a[0] * ( (u_ij-c_ij)*w1 + u_ij*w2 + (u_ij+c_ij)*w3 );
      DiffX[2] = a[0] * (        v_ij*w1 + v_ij*w2 +        v_ij*w3 - w4 );
      DiffX[3] = a[0] * ( (H_ij-c_ij*u_ij)*w1 + q_ij*w2 +
			  (H_ij+c_ij*u_ij)*w3 - v_ij*w4 );

      //------------------------------------------------------------------------
      // Dimensional splitting: y-direction
      //------------------------------------------------------------------------

      // Compute eigenvalues
      l1 = abs(v_ij-c_ij);
      l2 = abs(v_ij);
      l3 = abs(v_ij+c_ij);
      l4 = abs(v_ij);
        
      // Compute solution difference U_j-U_i
      DiffY[0] = IDX3(DdataAtEdge,1,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,1,1,1,NVAR2D,2,1);
      DiffY[1] = IDX3(DdataAtEdge,2,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,2,1,1,NVAR2D,2,1);
      DiffY[2] = IDX3(DdataAtEdge,3,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,3,1,1,NVAR2D,2,1);
      DiffY[3] = IDX3(DdataAtEdge,4,2,1,NVAR2D,2,1)-IDX3(DdataAtEdge,4,1,1,NVAR2D,2,1);
      
      // Compute auxiliary quantities for characteristic variables
      aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffY[0]
					 -u_ij*DiffY[1]
					 -v_ij*DiffY[2]
					      +DiffY[3])/RCONST(2.0)/c2_ij;
      aux2 = (v_ij*DiffY[0]
	          -DiffY[2])/RCONST(2.0)/c_ij;

      // Compute characteristic variables multiplied by the corresponding eigenvalue
      w1 = l1 * (aux1 + aux2);
      w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffY[0]
		                   +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffY[1]
								+v_ij*DiffY[2]
								     -DiffY[3])/c2_ij);
      w3 = l3 * (aux1 - aux2);
      w4 = l4 * (-u_ij*DiffY[0]
		      +DiffY[1]);
        
      // Compute "R_ij * |Lbd_ij| * L_ij * dU"
      DiffY[0] = a[1] * ( w1 + w2 + w3 );
      DiffY[1] = a[1] * (        u_ij*w1 + u_ij*w2 +        u_ij*w3 + w4 );
      DiffY[2] = a[1] * ( (v_ij-c_ij)*w1 + v_ij*w2 + (v_ij+c_ij)*w3 );
      DiffY[3] = a[1] * ( (H_ij-c_ij*v_ij)*w1 + q_ij*w2 +
			  (H_ij+c_ij*v_ij)*w3 + u_ij*w4 );

    } else {
      DiffX[0] = 0.0;
      DiffX[1] = 0.0;
      DiffX[2] = 0.0;
      DiffX[3] = 0.0;

      DiffY[0] = 0.0;
      DiffY[1] = 0.0;
      DiffY[2] = 0.0;
      DiffY[3] = 0.0;
    }
   
    //--------------------------------------------------------------------------
    // Build both contributions into the fluxes
    //--------------------------------------------------------------------------

#ifdef HYDRO_USE_IBP
    double DfluxesAtEdge[2*NVAR2D];
    IDX3(DfluxesAtEdge,1,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[0]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[0]-
       IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[0]-
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[0]+
       DiffX[0]+DiffY[0]);
    
    IDX3(DfluxesAtEdge,2,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[1]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[1]-
       IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[1]-
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[1]+
       DiffX[1]+DiffY[1]);
    
    IDX3(DfluxesAtEdge,3,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[2]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[2]-
       IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[2]-
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[2]+
       DiffX[2]+DiffY[2]);
    
    IDX3(DfluxesAtEdge,4,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[3]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[3]-
       IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[3]-
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[3]+
       DiffX[3]+DiffY[3]);
    

    IDX3(DfluxesAtEdge,1,2,1,NVAR2D,2,1) = -IDX3(DfluxesAtEdge,1,1,1,NVAR2D,2,1);
    IDX3(DfluxesAtEdge,2,2,1,NVAR2D,2,1) = -IDX3(DfluxesAtEdge,2,1,1,NVAR2D,2,1);
    IDX3(DfluxesAtEdge,3,2,1,NVAR2D,2,1) = -IDX3(DfluxesAtEdge,3,1,1,NVAR2D,2,1);
    IDX3(DfluxesAtEdge,4,2,1,NVAR2D,2,1) = -IDX3(DfluxesAtEdge,4,1,1,NVAR2D,2,1);
#else
    double DfluxesAtEdge[2*NVAR2D];    
    IDX3(DfluxesAtEdge,1,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[0]+
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[0]+
       DiffX[0]+DiffY[0]);

    IDX3(DfluxesAtEdge,2,1,1,NVAR2D,2,1) = dscale *
      (IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[1]+
       IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[1]+
       DiffX[1]+DiffY[1]);
    
    IDX3(DfluxesAtEdge,3,1,1,NVAR2D,2,1) = dscale *
      (IDX3(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[2]+
       IDX3(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[2]+
       DiffX[2]+DiffY[2]);
    
    IDX3(DfluxesAtEdge,4,1,1,NVAR2D,2,1) = dscale *
      (IDX3(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[3]+
       IDX3(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[3]+
       DiffX[3]+DiffY[3]);

    
    IDX3(DfluxesAtEdge,1,2,1,NVAR2D,2,1) = -dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[0]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[0]+
       DiffX[0]+DiffY[0]);

    IDX3(DfluxesAtEdge,2,2,1,NVAR2D,2,1) = -dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[1]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[1]+
       DiffX[1]+DiffY[1]);
    
    IDX3(DfluxesAtEdge,3,2,1,NVAR2D,2,1) = -dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[2]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[2]+
       DiffX[2]+DiffY[2]);
    
    IDX3(DfluxesAtEdge,4,2,1,NVAR2D,2,1) = -dscale *
      (IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[3]+
       IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[3]+
       DiffX[3]+DiffY[3]);
#endif
    
    //--------------------------------------------------------------------------
    // Build fluxes into nodal vector
    //--------------------------------------------------------------------------

    if (isystemformat == 0) {
      // Solution vector is stored in interleaved format
      IDX2_REVERSE(Dy,1,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,1,1,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,2,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,2,1,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,3,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,3,1,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,4,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,4,1,1,NVAR2D,2,1);
      
      IDX2_REVERSE(Dy,1,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,1,2,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,2,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,2,2,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,3,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,3,2,1,NVAR2D,2,1);
      IDX2_REVERSE(Dy,4,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,4,2,1,NVAR2D,2,1);

    } else {
      // Solution vector is stored in block format
      IDX2_FORWARD(Dy,1,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,1,1,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,2,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,2,1,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,3,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,3,1,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,4,i,NVAR2D,neq) += IDX3(DfluxesAtEdge,4,1,1,NVAR2D,2,1);

      IDX2_FORWARD(Dy,1,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,1,2,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,2,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,2,2,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,3,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,3,2,1,NVAR2D,2,1);
      IDX2_FORWARD(Dy,4,j,NVAR2D,neq) += IDX3(DfluxesAtEdge,4,2,1,NVAR2D,2,1);
    }
  }
}

/*******************************************************************************/

int hydro_calcFluxRoeDissDiSp2d_cuda(unsigned long * h_DcoeffsAtEdge,
				     unsigned long * h_IedgeList,
				     unsigned long * h_Dx,
				     unsigned long * h_Dy,
				     double * dscale,
				     int * nblocks, int * neq, int * nvar,
				     int * nedge, int * nmatcoeff,
				     int * nedges, int * iedgeset)
{
  double * d_Dx = (double*)(*h_Dx);
  double * d_Dy = (double*)(*h_Dy);
  double * d_DcoeffsAtEdge = (double*)(*h_DcoeffsAtEdge);
  int * d_IedgeList = (int*)(*h_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*nedges)/(double)(block.x));

  if (*nblocks == 1) {
    hydro_calcFluxRoeDissDiSp2d_knl<0><<<grid, block>>>(d_DcoeffsAtEdge,
							d_IedgeList,
							d_Dx, d_Dy, (*dscale), 
							(*neq), (*nvar),
							(*nedge), (*nmatcoeff),
							(*nedges), (*iedgeset));
  } else {
    hydro_calcFluxRoeDissDiSp2d_knl<1><<<grid, block>>>(d_DcoeffsAtEdge,
							d_IedgeList,
							d_Dx, d_Dy, (*dscale), 
							(*neq), (*nvar),
							(*nedge), (*nmatcoeff),
							(*nedges), (*iedgeset));
  }
  coproc_checkErrors("hydro_calcFluxRoeDissDiSp2d_cuda");
  return 0;
}

int FNAME(hydro_calcfluxroedissdisp2d_cuda)(unsigned long * h_DcoeffsAtEdge,
					    unsigned long * h_IedgeList,
					    unsigned long * h_Dx,
					    unsigned long * h_Dy,
					    double * dscale,
					    int * nblocks, int * neq, int * nvar,
					    int * nedge,   int * nmatcoeff,
					    int * nedges,  int * iedgeset)
{
  return hydro_calcFluxRoeDissDiSp2d_cuda(h_DcoeffsAtEdge, h_IedgeList,
					  h_Dx, h_Dy, dscale, nblocks, neq, nvar,
					  nedge, nmatcoeff, nedges, iedgeset);
}
