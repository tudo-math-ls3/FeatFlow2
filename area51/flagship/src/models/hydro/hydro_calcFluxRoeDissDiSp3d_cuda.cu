#include "hip/hip_runtime.h"
/*#############################################################################
 ******************************************************************************
 * <name>  hydro_calcFluxRoeDissDiSp3d_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This CUDA kernel computes the fluxes for the low-order scheme in 3D
 * using tensorial artificial viscosities of Roe-type, whereby dimensional
 * splitting is employed.
 * </purpose>
 *
 *#############################################################################/
 */

#include <stdio.h>
#include <math.h>
#include <iostream>
#include "coproc_core.h"
#include "coproc_storage_cuda.h"

#define LANGUAGE LANGUAGE_C
#include "../../../../../kernel/System/idxmanager.h"

#define HYDRO_NDIM 3
#include "hydro.h"

extern "C"
{
  int hydro_calcFluxRoeDissDiSp3d_cuda(unsigned long * h_DcoeffsAtEdge,
				       unsigned long * h_IedgeList,
				       unsigned long * h_Dx,
				       unsigned long * h_Dy,
				       double * dscale,
				       int * nblocks,
				       int * neq,
				       int * nvar,
				       int * nedge,
				       int * nmatcoeff,
				       int * nedges,
				       int * iedgeset);
  int FNAME(hydro_calcfluxroedissdisp3d_cuda)(unsigned long * h_DcoeffsAtEdge,
					      unsigned long * h_IedgeList,
					      unsigned long * h_Dx,
					      unsigned long * h_Dy,
					      double * dscale,
					      int * nblocks,
					      int * neq,
					      int * nvar,
					      int * nedge,
					      int * nmatcoeff,
					      int * nedges,
					      int * iedgeset);
}

/*******************************************************************************/
template <int isystemformat>
__global__ void hydro_calcFluxRoeDissDiSp3d_knl(double * DcoeffsAtEdge,
						int * IedgeList,
						double * Dx,
						double * Dy,
						double dscale,
						int neq,
						int nvar,
						int nedge,
						int nmatcoeff,
						int nedges,
						int iedgeset)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx<nedges)
    {
      // Get positions of edge endpoints (idx starts at zero)
      int i = IDX2T(IedgeList,1,iedgeset+idx,6,nedge);
      int j = IDX2T(IedgeList,2,iedgeset+idx,6,nedge);
      
      // Get solution values at edge endpoints
      double DdataAtEdge[2*NVAR3D];
      
      if (isystemformat == 0) {
	// Solution vector is stored in interleaved format
	IDX2(DdataAtEdge,1,1,NVAR3D,2) = IDX2_REVERSE(Dx,1,i,NVAR3D,neq);
	IDX2(DdataAtEdge,2,1,NVAR3D,2) = IDX2_REVERSE(Dx,2,i,NVAR3D,neq);
	IDX2(DdataAtEdge,3,1,NVAR3D,2) = IDX2_REVERSE(Dx,3,i,NVAR3D,neq);
	IDX2(DdataAtEdge,4,1,NVAR3D,2) = IDX2_REVERSE(Dx,4,i,NVAR3D,neq);
	IDX2(DdataAtEdge,5,1,NVAR3D,2) = IDX2_REVERSE(Dx,5,i,NVAR3D,neq);
	
	IDX2(DdataAtEdge,1,2,NVAR3D,2) = IDX2_REVERSE(Dx,1,j,NVAR3D,neq);
	IDX2(DdataAtEdge,2,2,NVAR3D,2) = IDX2_REVERSE(Dx,2,j,NVAR3D,neq);
	IDX2(DdataAtEdge,3,2,NVAR3D,2) = IDX2_REVERSE(Dx,3,j,NVAR3D,neq);
	IDX2(DdataAtEdge,4,2,NVAR3D,2) = IDX2_REVERSE(Dx,4,j,NVAR3D,neq);
	IDX2(DdataAtEdge,5,2,NVAR3D,2) = IDX2_REVERSE(Dx,5,j,NVAR3D,neq);
	
      } else {
	// Solution vector is stored in block format
	IDX2(DdataAtEdge,1,1,NVAR3D,2) = IDX2_FORWARD(Dx,1,i,NVAR3D,neq);
	IDX2(DdataAtEdge,2,1,NVAR3D,2) = IDX2_FORWARD(Dx,2,i,NVAR3D,neq);
	IDX2(DdataAtEdge,3,1,NVAR3D,2) = IDX2_FORWARD(Dx,3,i,NVAR3D,neq);
	IDX2(DdataAtEdge,4,1,NVAR3D,2) = IDX2_FORWARD(Dx,4,i,NVAR3D,neq);
	IDX2(DdataAtEdge,5,1,NVAR3D,2) = IDX2_FORWARD(Dx,5,i,NVAR3D,neq);
	
	IDX2(DdataAtEdge,1,2,NVAR3D,2) = IDX2_FORWARD(Dx,1,j,NVAR3D,neq);
	IDX2(DdataAtEdge,2,2,NVAR3D,2) = IDX2_FORWARD(Dx,2,j,NVAR3D,neq);
	IDX2(DdataAtEdge,3,2,NVAR3D,2) = IDX2_FORWARD(Dx,3,j,NVAR3D,neq);
	IDX2(DdataAtEdge,4,2,NVAR3D,2) = IDX2_FORWARD(Dx,4,j,NVAR3D,neq);
	IDX2(DdataAtEdge,5,2,NVAR3D,2) = IDX2_FORWARD(Dx,5,j,NVAR3D,neq);
      }

      //------------------------------------------------------------------------
      // Evaluate the Galerkin fluxes
      //------------------------------------------------------------------------
      
      // Compute velocities
      double ui = XVELOCITY3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1);
      double vi = YVELOCITY3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1);
      double wi = ZVELOCITY3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1);
      
      double uj = XVELOCITY3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1);
      double vj = YVELOCITY3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1);
      double wj = ZVELOCITY3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1);
      
      // Compute pressures
      double pi = PRESSURE3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1);
      double pj = PRESSURE3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1);
      
#ifdef HYDRO_USE_IBP
      double Fxi[NVAR3D];
      double Fxj[NVAR3D];
      
      // Compute fluxes for x-direction
      IDX1(Fxi,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,5) = INVISCIDFLUX5_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      
      IDX1(Fxj,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,5) = INVISCIDFLUX5_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      
      double Fyi[NVAR3D];
      double Fyj[NVAR3D];
      
      // Compute fluxes for y-direction
      IDX1(Fyi,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,5) = INVISCIDFLUX5_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      
      IDX1(Fyj,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,5) = INVISCIDFLUX5_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      
      double Fzi[NVAR3D];
      double Fzj[NVAR3D];
      
      // Compute fluxes for z-direction
      IDX1(Fzi,1) = INVISCIDFLUX1_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,2) = INVISCIDFLUX2_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,3) = INVISCIDFLUX3_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,4) = INVISCIDFLUX4_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,5) = INVISCIDFLUX5_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      
      IDX1(Fzj,1) = INVISCIDFLUX1_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,2) = INVISCIDFLUX2_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,3) = INVISCIDFLUX3_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,4) = INVISCIDFLUX4_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,5) = INVISCIDFLUX5_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
#else
      double Fx_ij[NVAR3D];
      
      // Compute flux difference for x-direction
      IDX1(Fx_ij,1) = INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                      INVISCIDFLUX1_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fx_ij,2) = INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                      INVISCIDFLUX2_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fx_ij,3) = INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                      INVISCIDFLUX3_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fx_ij,4) = INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                      INVISCIDFLUX4_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fx_ij,5) = INVISCIDFLUX5_XDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                      INVISCIDFLUX5_XDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      
      double Fy_ij[NVAR3D];
      
      // Compute flux difference for y-direction
      IDX1(Fy_ij,1) = INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX1_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fy_ij,2) = INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX2_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fy_ij,3) = INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX3_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fy_ij,4) = INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX4_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fy_ij,5) = INVISCIDFLUX5_YDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX5_YDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);

      double Fz_ij[NVAR3D];
      
      // Compute flux difference for y-direction
      IDX1(Fz_ij,1) = INVISCIDFLUX1_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX1_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fz_ij,2) = INVISCIDFLUX2_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX2_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fz_ij,3) = INVISCIDFLUX3_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX3_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fz_ij,4) = INVISCIDFLUX4_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
	              INVISCIDFLUX4_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fz_ij,5) = INVISCIDFLUX5_ZDIR3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                      INVISCIDFLUX5_ZDIR3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
#endif

      //------------------------------------------------------------------------
      // Evaluate the dissipation tensor of Roe-type
      //------------------------------------------------------------------------
      
      // Compute skew-symmetric coefficient
      double a[HYDRO_NDIM];
      a[0] = RCONST(0.5)*(IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)-
			  IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge));
      a[1] = RCONST(0.5)*(IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)-
			  IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge));
      a[2] = RCONST(0.5)*(IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)-
			  IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge));
      double anorm = sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
      
      double DiffX[NVAR3D];
      double DiffY[NVAR3D];
      double DiffZ[NVAR3D];
      if (anorm > 1e-14) {
	
	// Compute the absolute value
	a[0] = abs(a[0]);
	a[1] = abs(a[1]);
	a[2] = abs(a[2]);
	
	// Compute densities
	double ri = DENSITY3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1);
	double rj = DENSITY3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1);
	
	// Compute enthalpies
	double hi = (TOTALENERGY3(DdataAtEdge,IDX3,1,1,NVAR3D,2,1)+pi)/ri;
	double hj = (TOTALENERGY3(DdataAtEdge,IDX3,2,1,NVAR3D,2,1)+pj)/rj;
	
	// Compute Roe mean values
	double aux  = ROE_MEAN_RATIO(ri,rj);
	double u_ij = ROE_MEAN_VALUE(ui,uj,aux);
	double v_ij = ROE_MEAN_VALUE(vi,vj,aux);
	double w_ij = ROE_MEAN_VALUE(wi,wj,aux);
	double H_ij = ROE_MEAN_VALUE(hi,hj,aux);
	
	// Compute auxiliary variable
	double q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij + w_ij * w_ij);
	
	// Compute the speed of sound
	//TODO echtes double epsilon einbauen
	double c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14);
	double c_ij  = sqrt(c2_ij);
	
	//----------------------------------------------------------------------
	// Dimensional splitting: x-direction
	//----------------------------------------------------------------------
        
	// Compute eigenvalues
	double l1 = abs(u_ij-c_ij);
	double l2 = abs(u_ij);
	double l3 = abs(u_ij+c_ij);
	double l4 = abs(u_ij);
	double l5 = abs(u_ij);
        
	// Compute solution difference U_j-U_i
	DiffX[0] = IDX3(DdataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,1,1,1,NVAR3D,2,1);
	DiffX[1] = IDX3(DdataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,2,1,1,NVAR3D,2,1);
	DiffX[2] = IDX3(DdataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,3,1,1,NVAR3D,2,1);
	DiffX[3] = IDX3(DdataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,4,1,1,NVAR3D,2,1);
	DiffX[4] = IDX3(DdataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,5,1,1,NVAR3D,2,1);

	// Compute auxiliary quantities for characteristic variables
        double aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffX[0]
					          -u_ij*DiffX[1]
						  -v_ij*DiffX[2]
						  -w_ij*DiffX[3]
					               +DiffX[4])/RCONST(2.0)/c2_ij;
        double aux2 = (u_ij*DiffX[0]-DiffX[1])/RCONST(2.0)/c_ij;

        // Compute characteristic variables multiplied by the corresponding eigenvalue
        double w1 = l1 * (aux1 + aux2);
        double w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffX[0]
                                           +((HYDRO_GAMMA)-RCONST(1.0))*( u_ij*DiffX[1]
                                                                         +v_ij*DiffX[2]
                                                                         +w_ij*DiffX[3]
									      -DiffX[4])/c2_ij);
        double w3 = l3 * (aux1 - aux2);
        double w4 = l4 * ( v_ij*DiffX[0]-DiffX[2]);
        double w5 = l5 * (-w_ij*DiffX[0]+DiffX[3]);

        // Compute "R_ij * |Lbd_ij| * L_ij * dU"
        DiffX[0] = a[0] * ( w1 + w2 + w3 );
        DiffX[1] = a[0] * ( (u_ij-c_ij)*w1 + u_ij*w2 + (u_ij+c_ij)*w3 );
        DiffX[2] = a[0] * ( v_ij*(w1 + w2 + w3) - w4 );
        DiffX[3] = a[0] * ( w_ij*(w1 + w2 + w3) + w5 );
        DiffX[4] = a[0] * ( (H_ij-c_ij*u_ij)*w1 + q_ij*w2 + (H_ij+c_ij*u_ij)*w3
			     -v_ij*w4 + w_ij*w5 );
	
	//----------------------------------------------------------------------
	// Dimensional splitting: y-direction
	//----------------------------------------------------------------------

	// Compute eigenvalues
	l1 = abs(v_ij-c_ij);
	l2 = abs(v_ij);
	l3 = abs(v_ij+c_ij);
	l4 = abs(v_ij);
	l5 = abs(v_ij);
        
	// Compute solution difference U_j-U_i
	DiffY[0] = IDX3(DdataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,1,1,1,NVAR3D,2,1);
	DiffY[1] = IDX3(DdataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,2,1,1,NVAR3D,2,1);
	DiffY[2] = IDX3(DdataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,3,1,1,NVAR3D,2,1);
	DiffY[3] = IDX3(DdataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,4,1,1,NVAR3D,2,1);
	DiffY[4] = IDX3(DdataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,5,1,1,NVAR3D,2,1);
	
	// Compute auxiliary quantities for characteristic variables
        aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffY[0]
                                           -u_ij*DiffY[1]
                                           -v_ij*DiffY[2]
                                           -w_ij*DiffY[3]
					        +DiffY[4])/RCONST(2.0)/c2_ij;
        aux2 = (v_ij*DiffY[0]-DiffY[2])/RCONST(2.0)/c_ij;

        // Compute characteristic variables multiplied by the corresponding eigenvalue
        w1 = l1 * (aux1 + aux2);
        w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffY[0]
                                     +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffY[1]
                                                                  +v_ij*DiffY[2]
                                                                  +w_ij*DiffY[3]
								       -DiffY[4])/c2_ij);
        w3 = l3 * (aux1 - aux2);
        w4 = l4 * (-u_ij*DiffY[0]+DiffY[1]);
        w5 = l5 * ( w_ij*DiffY[0]-DiffY[3]);

        // Compute "R_ij * |Lbd_ij| * L_ij * dU"
        DiffY[0] = a[1] * ( w1 + w2 + w3 );
        DiffY[1] = a[1] * ( u_ij*(w1 + w2 + w3) + w4 );
        DiffY[2] = a[1] * ( (v_ij-c_ij)*w1 + v_ij*w2 + (v_ij+c_ij)*w3 );
        DiffY[3] = a[1] * ( w_ij*(w1 + w2 + w3) - w5 );
        DiffY[4] = a[1] * ( (H_ij-c_ij*v_ij)*w1 + q_ij*w2 + (H_ij+c_ij*v_ij)*w3
                            + u_ij*w4 -w_ij*w5 );

	//----------------------------------------------------------------------
	// Dimensional splitting: z-direction
	//----------------------------------------------------------------------

	// Compute eigenvalues
	l1 = abs(w_ij-c_ij);
	l2 = abs(w_ij);
	l3 = abs(w_ij+c_ij);
	l4 = abs(w_ij);
	l5 = abs(w_ij);
        
	// Compute solution difference U_j-U_i
	DiffZ[0] = IDX3(DdataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,1,1,1,NVAR3D,2,1);
	DiffZ[1] = IDX3(DdataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,2,1,1,NVAR3D,2,1);
	DiffZ[2] = IDX3(DdataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,3,1,1,NVAR3D,2,1);
	DiffZ[3] = IDX3(DdataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,4,1,1,NVAR3D,2,1);
	DiffZ[4] = IDX3(DdataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DdataAtEdge,5,1,1,NVAR3D,2,1);
	
	// Compute auxiliary quantities for characteristic variables
        aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffZ[0]
                                           -u_ij*DiffZ[1]
                                           -v_ij*DiffZ[2]
                                           -w_ij*DiffZ[3]
					        +DiffZ[4])/RCONST(2.0)/c2_ij;
        aux2 = (w_ij*DiffZ[0]-DiffZ[2])/RCONST(2.0)/c_ij;

        // Compute characteristic variables multiplied by the corresponding eigenvalue
        w1 = l1 * (aux1 + aux2);
        w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffZ[0]
                                     +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffZ[1]
                                                                  +v_ij*DiffZ[2]
                                                                  +w_ij*DiffZ[3]
								       -DiffZ[4])/c2_ij);
        w3 = l3 * (aux1 - aux2);
        w4 = l4 * ( u_ij*DiffZ[0]-DiffZ[1]);
        w5 = l5 * (-v_ij*DiffZ[0]+DiffZ[2]);
        
        // Compute "R_ij * |Lbd_ij| * L_ij * dU"
        DiffZ[0] = a[2] * ( w1 + w2 + w3 );
        DiffZ[1] = a[2] * ( u_ij*(w1 + w2 + w3) - w4 );
        DiffZ[2] = a[2] * ( v_ij*(w1 + w2 + w3) + w5 );
        DiffZ[3] = a[2] * ( (w_ij-c_ij)*w1 + w_ij*w2 + (w_ij+c_ij)*w3 );
        DiffZ[4] = a[2] * ( (H_ij-c_ij*w_ij)*w1 + q_ij*w2 + (H_ij+c_ij*w_ij)*w3
                            -u_ij*w4 + v_ij*w5 );
      } else {
	DiffX[0] = 0.0;
	DiffX[1] = 0.0;
	DiffX[2] = 0.0;
	DiffX[3] = 0.0;
	DiffX[4] = 0.0;
	
	DiffY[0] = 0.0;
	DiffY[1] = 0.0;
	DiffY[2] = 0.0;
	DiffY[3] = 0.0;
	DiffY[4] = 0.0;

	DiffZ[0] = 0.0;
	DiffZ[1] = 0.0;
	DiffZ[2] = 0.0;
	DiffZ[3] = 0.0;
	DiffZ[4] = 0.0;
      }
      
      //------------------------------------------------------------------------
      // Build both contributions into the fluxes
      //------------------------------------------------------------------------
      
#ifdef HYDRO_USE_IBP
      double DfluxesAtEdge[2*NVAR3D];
      IDX3(DfluxesAtEdge,1,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[0]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[0]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzj[0]-
	 IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[0]-
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[0]-
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzi[0]+
	 DiffX[0]+DiffY[0]+DiffZ[0]);
      
      IDX3(DfluxesAtEdge,2,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[1]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[1]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzj[1]-
	 IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[1]-
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[1]-
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzi[1]+
	 DiffX[1]+DiffY[1]+DiffZ[1]);
      
      IDX3(DfluxesAtEdge,3,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[2]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[2]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzj[2]-
	 IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[2]-
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[2]-
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzi[2]+
	 DiffX[2]+DiffY[2]+DiffZ[2]);
      
      IDX3(DfluxesAtEdge,4,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[3]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[3]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzj[3]-
	 IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[3]-
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[3]-
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzi[3]+
	 DiffX[3]+DiffY[3]+DiffZ[3]);

      IDX3(DfluxesAtEdge,5,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxj[4]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyj[4]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzj[4]-
	 IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fxi[4]-
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fyi[4]-
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fzi[4]+
	 DiffX[4]+DiffY[4]+DiffZ[4]);
      
      
      IDX3(DfluxesAtEdge,1,2,1,NVAR3D,2,1) = -IDX3(DfluxesAtEdge,1,1,1,NVAR3D,2,1);
      IDX3(DfluxesAtEdge,2,2,1,NVAR3D,2,1) = -IDX3(DfluxesAtEdge,2,1,1,NVAR3D,2,1);
      IDX3(DfluxesAtEdge,3,2,1,NVAR3D,2,1) = -IDX3(DfluxesAtEdge,3,1,1,NVAR3D,2,1);
      IDX3(DfluxesAtEdge,4,2,1,NVAR3D,2,1) = -IDX3(DfluxesAtEdge,4,1,1,NVAR3D,2,1);
      IDX3(DfluxesAtEdge,5,2,1,NVAR3D,2,1) = -IDX3(DfluxesAtEdge,5,1,1,NVAR3D,2,1);
#else
      double DfluxesAtEdge[2*NVAR3D];    
      IDX3(DfluxesAtEdge,1,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[0]+
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[0]+
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[0]+
	 DiffX[0]+DiffY[0]+DiffZ[0]);
      
      IDX3(DfluxesAtEdge,2,1,1,NVAR3D,2,1) = dscale *
	(IDX3T(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[1]+
	 IDX3T(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[1]+
	 IDX3T(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[1]+
	 DiffX[1]+DiffY[1]+DiffZ[1]);
      
      IDX3(DfluxesAtEdge,3,1,1,NVAR3D,2,1) = dscale *
	(IDX3(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[2]+
	 IDX3(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[2]+
	 IDX3(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[2]+
	 DiffX[2]+DiffY[2]+DiffZ[2]);
      
      IDX3(DfluxesAtEdge,4,1,1,NVAR3D,2,1) = dscale *
	(IDX3(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[3]+
	 IDX3(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[3]+
	 IDX3(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[3]+
	 DiffX[3]+DiffY[3]+DiffZ[3]);

      IDX3(DfluxesAtEdge,5,1,1,NVAR3D,2,1) = dscale *
	(IDX3(DcoeffsAtEdge,1,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[4]+
	 IDX3(DcoeffsAtEdge,2,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[4]+
	 IDX3(DcoeffsAtEdge,3,1,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[4]+
	 DiffX[4]+DiffY[4]+DiffZ[4]);
      
      
      IDX3(DfluxesAtEdge,1,2,1,NVAR3D,2,1) = -dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[0]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[0]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[0]+
	 DiffX[0]+DiffY[0]+DiffZ[0]);
      
      IDX3(DfluxesAtEdge,2,2,1,NVAR3D,2,1) = -dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[1]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[1]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[1]+
	 DiffX[1]+DiffY[1]+DiffZ[1]);
      
      IDX3(DfluxesAtEdge,3,2,1,NVAR3D,2,1) = -dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[2]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[2]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[2]+
	 DiffX[2]+DiffY[2]+DiffZ[2]);
      
      IDX3(DfluxesAtEdge,4,2,1,NVAR3D,2,1) = -dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[3]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[3]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[3]+
	 DiffX[3]+DiffY[3]+DiffZ[3]);
	 
      IDX3(DfluxesAtEdge,5,2,1,NVAR3D,2,1) = -dscale *
	(IDX3T(DcoeffsAtEdge,1,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fx_ij[4]+
	 IDX3T(DcoeffsAtEdge,2,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fy_ij[4]+
	 IDX3T(DcoeffsAtEdge,3,2,iedgeset+idx,HYDRO_NDIM,nmatcoeff,nedge)*Fz_ij[4]+
	 DiffX[4]+DiffY[4]+DiffZ[4]);
#endif
      
      //--------------------------------------------------------------------------
      // Build fluxes into nodal vector
      //--------------------------------------------------------------------------
      
       if (isystemformat == 0) {
	// Solution vector is stored in interleaved format
	IDX2_REVERSE(Dy,1,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,1,1,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,2,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,2,1,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,3,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,3,1,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,4,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,4,1,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,5,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,5,1,1,NVAR3D,2,1);
	
	IDX2_REVERSE(Dy,1,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,1,2,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,2,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,2,2,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,3,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,3,2,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,4,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,4,2,1,NVAR3D,2,1);
	IDX2_REVERSE(Dy,5,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,5,2,1,NVAR3D,2,1);
      } else {
	// Solution vector is stored in block format
	IDX2_FORWARD(Dy,1,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,1,1,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,2,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,2,1,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,3,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,3,1,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,4,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,4,1,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,5,i,NVAR3D,neq) += IDX3(DfluxesAtEdge,5,1,1,NVAR3D,2,1);
	
	IDX2_FORWARD(Dy,1,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,1,2,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,2,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,2,2,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,3,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,3,2,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,4,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,4,2,1,NVAR3D,2,1);
	IDX2_FORWARD(Dy,5,j,NVAR3D,neq) += IDX3(DfluxesAtEdge,5,2,1,NVAR3D,2,1);
      }
    }
}

/*******************************************************************************/

int hydro_calcFluxRoeDissDiSp3d_cuda(unsigned long * h_DcoeffsAtEdge,
				     unsigned long * h_IedgeList,
				     unsigned long * h_Dx,
				     unsigned long * h_Dy,
				     double * dscale,
				     int * nblocks, int * neq, int * nvar,
				     int * nedge, int * nmatcoeff,
				     int * nedges, int * iedgeset)
{
  double * d_Dx = (double*)(*h_Dx);
  double * d_Dy = (double*)(*h_Dy);
  double * d_DcoeffsAtEdge = (double*)(*h_DcoeffsAtEdge);
  int * d_IedgeList = (int*)(*h_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*nedges)/(double)(block.x));
  
  if (*nblocks == 1) {
    hydro_calcFluxRoeDissDiSp3d_knl<0><<<grid, block>>>(d_DcoeffsAtEdge,
							d_IedgeList,
							d_Dx, d_Dy, (*dscale), 
							(*neq), (*nvar),
							(*nedge), (*nmatcoeff),
							(*nedges), (*iedgeset));
  } else {
    hydro_calcFluxRoeDissDiSp3d_knl<1><<<grid, block>>>(d_DcoeffsAtEdge,
							d_IedgeList,
							d_Dx, d_Dy, (*dscale), 
							(*neq), (*nvar),
							(*nedge), (*nmatcoeff),
							(*nedges), (*iedgeset));
  }
  coproc_checkErrors("hydro_calcFluxRoeDissDiSp3d_cuda");
  return 0;
}

int FNAME(hydro_calcfluxroedissdisp3d_cuda)(unsigned long * h_DcoeffsAtEdge,
					    unsigned long * h_IedgeList,
					    unsigned long * h_Dx,
					    unsigned long * h_Dy,
					    double * dscale,
					    int * nblocks, int * neq, int * nvar,
					    int * nedge,   int * nmatcoeff,
					    int * nedges,  int * iedgeset)
{
  return hydro_calcFluxRoeDissDiSp3d_cuda(h_DcoeffsAtEdge, h_IedgeList,
					  h_Dx, h_Dy, dscale, nblocks, neq, nvar,
					  nedge, nmatcoeff, nedges, iedgeset);
}
