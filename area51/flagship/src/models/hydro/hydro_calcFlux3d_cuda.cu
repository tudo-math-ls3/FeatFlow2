#include "hip/hip_runtime.h"
/*#############################################################################
 ******************************************************************************
 * <name> hydro_calcFlux3d_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This file provides CUDA kernels to compute the fluxes for the low-order
 * scheme in 3D using different types if artificial viscosities.
 * </purpose>
 *
 *#############################################################################/
 */

#include <stdio.h>
#include <math.h>
#include <iostream>
#include "coproc_core.h"
#include "coproc_storage_cuda.h"

#define LANGUAGE LANGUAGE_C
#include "../../../../../kernel/System/idxmanager.h"

#define HYDRO_NDIM 3
#include "hydro.h"

/*******************************************************************************
 * CUDA kernels
 *******************************************************************************/

template <int isystemformat>
struct gather_DataAtEdge
{ 
};

/*******************************************************************************/

template <>
struct gather_DataAtEdge<SYSTEM_SEGREGATED>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval (Td *DataAtEdge,
		    Td *Dx,
		    Ti i,
		    Ti j,
		    Ti neq)
  {
    // Solution vector is stored in interleaved format
    IDX2(DataAtEdge,1,1,NVAR3D,2) = IDX2_REVERSE(Dx,1,i,NVAR3D,neq);
    IDX2(DataAtEdge,2,1,NVAR3D,2) = IDX2_REVERSE(Dx,2,i,NVAR3D,neq);
    IDX2(DataAtEdge,3,1,NVAR3D,2) = IDX2_REVERSE(Dx,3,i,NVAR3D,neq);
    IDX2(DataAtEdge,4,1,NVAR3D,2) = IDX2_REVERSE(Dx,4,i,NVAR3D,neq);
    IDX2(DataAtEdge,5,1,NVAR3D,2) = IDX2_REVERSE(Dx,5,i,NVAR3D,neq);
    
    IDX2(DataAtEdge,1,2,NVAR3D,2) = IDX2_REVERSE(Dx,1,j,NVAR3D,neq);
    IDX2(DataAtEdge,2,2,NVAR3D,2) = IDX2_REVERSE(Dx,2,j,NVAR3D,neq);
    IDX2(DataAtEdge,3,2,NVAR3D,2) = IDX2_REVERSE(Dx,3,j,NVAR3D,neq);
    IDX2(DataAtEdge,4,2,NVAR3D,2) = IDX2_REVERSE(Dx,4,j,NVAR3D,neq);
    IDX2(DataAtEdge,5,2,NVAR3D,2) = IDX2_REVERSE(Dx,5,j,NVAR3D,neq);
  }
};

/*******************************************************************************/

template <>
struct gather_DataAtEdge<SYSTEM_ALLCOUPLED>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval (Td *DataAtEdge,
		    Td *Dx,
		    Ti i,
		    Ti j,
		    Ti neq)
  {
    // Solution vector is stored in block format
    IDX2(DataAtEdge,1,1,NVAR3D,2) = IDX2_FORWARD(Dx,1,i,NVAR3D,neq);
    IDX2(DataAtEdge,2,1,NVAR3D,2) = IDX2_FORWARD(Dx,2,i,NVAR3D,neq);
    IDX2(DataAtEdge,3,1,NVAR3D,2) = IDX2_FORWARD(Dx,3,i,NVAR3D,neq);
    IDX2(DataAtEdge,4,1,NVAR3D,2) = IDX2_FORWARD(Dx,4,i,NVAR3D,neq);
    IDX2(DataAtEdge,5,1,NVAR3D,2) = IDX2_FORWARD(Dx,5,i,NVAR3D,neq);
    
    IDX2(DataAtEdge,1,2,NVAR3D,2) = IDX2_FORWARD(Dx,1,j,NVAR3D,neq);
    IDX2(DataAtEdge,2,2,NVAR3D,2) = IDX2_FORWARD(Dx,2,j,NVAR3D,neq);
    IDX2(DataAtEdge,3,2,NVAR3D,2) = IDX2_FORWARD(Dx,3,j,NVAR3D,neq);
    IDX2(DataAtEdge,4,2,NVAR3D,2) = IDX2_FORWARD(Dx,4,j,NVAR3D,neq);
    IDX2(DataAtEdge,5,2,NVAR3D,2) = IDX2_FORWARD(Dx,5,j,NVAR3D,neq);
  }
};

/*******************************************************************************/

struct calc_GalerkinFlux
{
  template <typename Td>
#ifdef HYDRO_USE_IBP
  __device__ inline
  static void eval(Td *Fxi,
		   Td *Fxj,
		   Td *Fyi,
		   Td *Fyj,
		   Td *Fzi,
		   Td *Fzj,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj)
  {
    // Compute fluxes for x-direction
      IDX1(Fxi,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      IDX1(Fxi,5) = INVISCIDFLUX5_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi);
      
      IDX1(Fxj,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
      IDX1(Fxj,5) = INVISCIDFLUX5_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);

      // Compute fluxes for y-direction
      IDX1(Fyi,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fyi,5) = INVISCIDFLUX5_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      
      IDX1(Fyj,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fyj,5) = INVISCIDFLUX5_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);

      // Compute fluxes for z-direction
      IDX1(Fzi,1) = INVISCIDFLUX1_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,2) = INVISCIDFLUX2_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,3) = INVISCIDFLUX3_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,4) = INVISCIDFLUX4_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      IDX1(Fzi,5) = INVISCIDFLUX5_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi);
      
      IDX1(Fzj,1) = INVISCIDFLUX1_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,2) = INVISCIDFLUX2_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,3) = INVISCIDFLUX3_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,4) = INVISCIDFLUX4_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      IDX1(Fzj,5) = INVISCIDFLUX5_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
  }
#else
  __device__ inline
  static void eval(Td *Fx_ij,
		   Td *Fy_ij,
		   Td *Fz_ij,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj)
  {
    // Compute flux difference for x-direction
    IDX1(Fx_ij,1) = INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                    INVISCIDFLUX1_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
    IDX1(Fx_ij,2) = INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                    INVISCIDFLUX2_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
    IDX1(Fx_ij,3) = INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                    INVISCIDFLUX3_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
    IDX1(Fx_ij,4) = INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                    INVISCIDFLUX4_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
    IDX1(Fx_ij,5) = INVISCIDFLUX5_XDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,ui,pi)-
                    INVISCIDFLUX5_XDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,uj,pj);
    
    // Compute flux difference for y-direction
    IDX1(Fy_ij,1) = INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX1_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fy_ij,2) = INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX2_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fy_ij,3) = INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX3_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fy_ij,4) = INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX4_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fy_ij,5) = INVISCIDFLUX5_YDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX5_YDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
      
    // Compute flux difference for z-direction
    IDX1(Fz_ij,1) = INVISCIDFLUX1_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX1_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fz_ij,2) = INVISCIDFLUX2_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX2_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fz_ij,3) = INVISCIDFLUX3_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX3_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fz_ij,4) = INVISCIDFLUX4_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX4_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
    IDX1(Fz_ij,5) = INVISCIDFLUX5_ZDIR3(DataAtEdge,IDX3,1,1,NVAR3D,2,1,vi,pi)-
                    INVISCIDFLUX5_ZDIR3(DataAtEdge,IDX3,2,1,NVAR3D,2,1,vj,pj);
  }
#endif
};

/*******************************************************************************/

template <int idissipationtype>
struct calc_Dissipation
{
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_ZERO>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff,
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge,
		   Ti nedge,
		   Ti ncoeff)
  {
    Diff[0] = 0.0;
    Diff[1] = 0.0;
    Diff[2] = 0.0;
    Diff[3] = 0.0;
    Diff[4] = 0.0;
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_SCALAR>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff,
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge,
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the scalar dissipation proportional to the spectral
    // radius (largest eigenvalue) of the Roe-matrix
    //------------------------------------------------------------------------
    
    // Compute skew-symmetric coefficient
    Td a[HYDRO_NDIM];
    a[0] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[1] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[2] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    Td anorm = sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
    
    // Compute densities
    Td ri = DENSITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
    Td rj = DENSITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
    
    // Compute enthalpies
    Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1)+pi)/ri;
    Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1)+pj)/rj;
    
    // Compute Roe mean values
    Td aux  = ROE_MEAN_RATIO(ri,rj);
    Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
    Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
    Td w_ij = ROE_MEAN_VALUE(wi,wj,aux);
    Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
    
    // Compute auxiliary variables
    Td vel_ij = u_ij * a[0] + v_ij * a[1] + w_ij * a[2];
    Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij + w_ij * w_ij);
    
    // Compute the speed of sound
    //TODO echtes double epsilon einbauen
    Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14));
    
    // Compute scalar dissipation
    Td d_ij = abs(vel_ij) + anorm*c_ij;
    
    // Multiply the solution difference by the scalar dissipation
    Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1));
    Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1));
    Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1));
    Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1));
    Diff[4] = d_ij*(IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1));
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_SCALAR_DSPLIT>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff,
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge,
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the scalar dissipation proportional to the spectral
    // radius (largest eigenvalue) of the dimensional-split Roe-matrix
    //------------------------------------------------------------------------
    
    // Compute skew-symmetric coefficient
    Td a[HYDRO_NDIM];
    a[0] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[1] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[2] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    
    // Compute densities
    Td ri = DENSITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
    Td rj = DENSITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
    
    // Compute enthalpies
    Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1)+pi)/ri;
    Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1)+pj)/rj;
    
    // Compute Roe mean values
    Td aux  = ROE_MEAN_RATIO(ri,rj);
    Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
    Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
    Td w_ij = ROE_MEAN_VALUE(wi,wj,aux);
    Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
    
    // Compute auxiliary variables
    Td q_ij = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij + w_ij * w_ij);
    
    // Compute the speed of sound
    //TODO echtes double epsilon einbauen
    Td c_ij = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14));
    
    // Compute scalar dissipation
    Td d_ij = ( abs(a[0]*u_ij) + abs(a[0])*c_ij +
 	        abs(a[1]*v_ij) + abs(a[1])*c_ij +
	        abs(a[2]*w_ij) + abs(a[2])*c_ij );
    
    // Multiply the solution difference by the scalar dissipation
    Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1));
    Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1));
    Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1));
    Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1));
    Diff[4] = d_ij*(IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1));
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_ROE>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff, 
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi, 
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge, 
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the dissipation tensor of Roe-type
    //------------------------------------------------------------------------
    
    // Compute skew-symmetric coefficient
    Td a[HYDRO_NDIM];
    a[0] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[1] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[2] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    Td anorm = sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
    
    if (anorm > 1e-14) {
      
      // Normalise the skew-symmetric coefficient
      a[0] = a[0]/anorm;
      a[1] = a[1]/anorm;
      a[2] = a[2]/anorm;
      
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
      
      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1)+pj)/rj;
      
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td w_ij = ROE_MEAN_VALUE(wi,wj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variables
      Td vel_ij = u_ij * a[0] + v_ij * a[1] + w_ij * a[2];
      Td q_ij   = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij + w_ij * w_ij);
      
      // Compute the speed of sound
      //TODO echtes double epsilon einbauen
      Td c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14);
      Td c_ij  = sqrt(c2_ij);
      
      // Compute eigenvalues
      Td l1 = abs(vel_ij-c_ij);
      Td l2 = abs(vel_ij);
      Td l3 = abs(vel_ij+c_ij);
      Td l4 = abs(vel_ij);
      Td l5 = abs(vel_ij);
      
      // Compute solution difference U_j-U_i
      Diff[0] = IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1);
      Diff[1] = IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1);
      Diff[2] = IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1);
      Diff[3] = IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1);
      Diff[4] = IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1);
      
      // Compute auxiliary quantities for characteristic variables
      Td aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*Diff[0]
					    -u_ij*Diff[1]
					    -v_ij*Diff[2]
					    -w_ij*Diff[3]
					         +Diff[4])/RCONST(2.0)/c2_ij;
      Td aux2 = (vel_ij*Diff[0]
		  -a[0]*Diff[1]
	          -a[1]*Diff[2]
	          -a[2]*Diff[3])/RCONST(2.0)/c_ij;
      
      // Get the dimension with largest coefficient
      if (a[0] >= a[1] && a[0] >= a[2]) {
	
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	Td w1 = l1 * (aux1 + aux2);
	Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*Diff[0]
                                       +((HYDRO_GAMMA)-RCONST(1.0))*( u_ij*Diff[1]
								     +v_ij*Diff[2]
                                                                     +w_ij*Diff[3]
								          -Diff[4])/c2_ij);
	Td w3 = l3 * (aux1 - aux2);
	Td w4 = l4 * ( (v_ij-vel_ij*a[1])/a[0]*Diff[0]
			                 +a[1]*Diff[1]
		 +(a[1]*a[1]-RCONST(1.0))/a[0]*Diff[2]
		               +a[1]*a[2]/a[0]*Diff[3]);
	Td w5 = l5 * ( (vel_ij*a[2]-w_ij)/a[0]*Diff[0]
                                         -a[2]*Diff[1]
                               -a[1]*a[2]/a[0]*Diff[2]
	         +(RCONST(1.0)-a[2]*a[2])/a[0]*Diff[3]);
	
	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] = anorm * ( w1 + w2 + w3 );
	Diff[1] = anorm * ( (u_ij-c_ij*a[0])*w1 + u_ij*w2 +
			    (u_ij+c_ij*a[0])*w3 + a[1]*w4 - a[2]*w5 );
	Diff[2] = anorm * ( (v_ij-c_ij*a[1])*w1 + v_ij*w2 +
			    (v_ij+c_ij*a[1])*w3 - a[0]*w4 );
	Diff[3] = anorm * ( (w_ij-c_ij*a[2])*w1 + w_ij*w2 +
			    (w_ij+c_ij*a[2])*w3 + a[0]*w5 );
	Diff[4] = anorm * ( (H_ij-c_ij*vel_ij)*w1 + q_ij*w2 + (H_ij+c_ij*vel_ij)*w3
			    + (u_ij*a[1]-v_ij*a[0])*w4 + (w_ij*a[0]-u_ij*a[2])*w5 );
	
      } else if (a[1] >= a[0] && a[1] >= a[2]) {
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	Td w1 = l1 * (aux1 + aux2);
	Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*Diff[0]
		                        +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*Diff[1]
								     +v_ij*Diff[2]
                                                                     +w_ij*Diff[3]
								          -Diff[4])/c2_ij);
	Td w3 = l3 * (aux1 - aux2);
	Td w4 = l4 * ( (vel_ij*a[0]-u_ij)/a[1]*Diff[0]
	 	 +(RCONST(1.0)-a[0]*a[0])/a[1]*Diff[1]
		                         -a[0]*Diff[2]
		               -a[0]*a[2]/a[1]*Diff[3]);
	Td w5 = l5 * ( (w_ij-vel_ij*a[2])/a[1]*Diff[0]
		               +a[0]*a[2]/a[1]*Diff[1]
		                         +a[2]*Diff[2]
		 +(a[2]*a[2]-RCONST(1.0))/a[1]*Diff[3]);
	  
	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] = anorm * ( w1 + w2 + w3 );
	Diff[1] = anorm * ( (u_ij-c_ij*a[0])*w1 + u_ij*w2 +
			    (u_ij+c_ij*a[0])*w3 + a[1]*w4 );
	Diff[2] = anorm * ( (v_ij-c_ij*a[1])*w1 + v_ij*w2 +
			    (v_ij+c_ij*a[1])*w3 - a[0]*w4 + a[2]*w5 );
	Diff[3] = anorm * ( (w_ij-c_ij*a[2])*w1 + w_ij*w2 +
			    (w_ij+c_ij*a[2])*w3 - a[1]*w5 );
	Diff[4] = anorm * ( (H_ij-c_ij*vel_ij)*w1 + q_ij*w2 + (H_ij+c_ij*vel_ij)*w3
			    + (u_ij*a[1]-v_ij*a[0])*w4 + (v_ij*a[2]-w_ij*a[1])*w5 );
	
      } else {
	// Compute characteristic variables multiplied by the corresponding eigenvalue
	Td w1 = l1 * (aux1 + aux2);
	Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*Diff[0]
		                        +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*Diff[1]
							 	     +v_ij*Diff[2]
								     +w_ij*Diff[3]
								          -Diff[4])/c2_ij);
	Td w3 = l3 * (aux1 - aux2);
	Td w4 = l4 * ( (u_ij-vel_ij*a[0])/a[2]*Diff[0]
	 	 +(a[0]*a[0]-RCONST(1.0))/a[2]*Diff[1]
		               +a[0]*a[1]/a[2]*Diff[2]
		                         +a[0]*Diff[3]);
	Td w5 = l5 * ( (vel_ij*a[1]-v_ij)/a[2]*Diff[0]
		               -a[0]*a[1]/a[2]*Diff[1]
		 +(RCONST(1.0)-a[1]*a[1])/a[2]*Diff[2]
		                         -a[1]*Diff[3]);
	  
	// Compute "R_ij * |Lbd_ij| * L_ij * dU"
	Diff[0] = anorm * ( w1 + w2 + w3 );
	Diff[1] = anorm * ( (u_ij-c_ij*a[0])*w1 + u_ij*w2 +
			    (u_ij+c_ij*a[0])*w3 - a[2]*w4 );
	Diff[2] = anorm * ( (v_ij-c_ij*a[1])*w1 + v_ij*w2 +
			    (v_ij+c_ij*a[1])*w3 + a[2]*w5 );
	Diff[3] = anorm * ( (w_ij-c_ij*a[2])*w1 + w_ij*w2 +
			    (w_ij+c_ij*a[2])*w3 + a[0]*w4 - a[1]*w5);
	Diff[4] = anorm * ( (H_ij-c_ij*vel_ij)*w1 + q_ij*w2 + (H_ij+c_ij*vel_ij)*w3
			    + (w_ij*a[0]-u_ij*a[2])*w4 + (v_ij*a[2]-w_ij*a[1])*w5 );
      }
    } else {
      Diff[0] = 0.0;
      Diff[1] = 0.0;
      Diff[2] = 0.0;
      Diff[3] = 0.0;
      Diff[4] = 0.0;
    }
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_ROE_DSPLIT>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff,
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge, 
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the dissipation tensor of Roe-type with dimensional splitting
    //------------------------------------------------------------------------
    
  // Compute skew-symmetric coefficient
    Td a[HYDRO_NDIM];
    a[0] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[1] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    a[2] = RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge));
    Td anorm = sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
    
    Td DiffAux[NVAR3D];
    if (anorm > 1e-14) {
      
      // Compute the absolute value
      a[0] = abs(a[0]);
      a[1] = abs(a[1]);
      a[2] = abs(a[2]);
      
      // Compute densities
      Td ri = DENSITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      Td rj = DENSITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
      
      // Compute enthalpies
      Td hi = (TOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1)+pi)/ri;
      Td hj = (TOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1)+pj)/rj;
      
      // Compute Roe mean values
      Td aux  = ROE_MEAN_RATIO(ri,rj);
      Td u_ij = ROE_MEAN_VALUE(ui,uj,aux);
      Td v_ij = ROE_MEAN_VALUE(vi,vj,aux);
      Td w_ij = ROE_MEAN_VALUE(wi,wj,aux);
      Td H_ij = ROE_MEAN_VALUE(hi,hj,aux);
      
      // Compute auxiliary variable
      Td q_ij = RCONST(0.5) * (u_ij * u_ij + v_ij * v_ij + w_ij * w_ij);
      
      // Compute the speed of sound
      //TODO echtes double epsilon einbauen
      Td c2_ij = max(((HYDRO_GAMMA)-RCONST(1.0))*(H_ij-q_ij), 1e-14);
      Td c_ij  = sqrt(c2_ij);
      
      //----------------------------------------------------------------------
      // Dimensional splitting: x-direction
      //----------------------------------------------------------------------
      
      // Compute eigenvalues
      Td l1 = abs(u_ij-c_ij);
      Td l2 = abs(u_ij);
      Td l3 = abs(u_ij+c_ij);
      Td l4 = abs(u_ij);
      Td l5 = abs(u_ij);
      
      // Compute solution difference U_j-U_i
      DiffAux[0] = IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1);
      DiffAux[1] = IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1);
      DiffAux[2] = IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1);
      DiffAux[3] = IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1);
      DiffAux[4] = IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1);
      
      // Compute auxiliary quantities for characteristic variables
      Td aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffAux[0]
					    -u_ij*DiffAux[1]
					    -v_ij*DiffAux[2]
					    -w_ij*DiffAux[3]
					         +DiffAux[4])/RCONST(2.0)/c2_ij;
      Td aux2 = (u_ij*DiffAux[0]-DiffAux[1])/RCONST(2.0)/c_ij;
      
      // Compute characteristic variables multiplied by the corresponding eigenvalue
      Td w1 = l1 * (aux1 + aux2);
      Td w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffAux[0]
		                     +((HYDRO_GAMMA)-RCONST(1.0))*( u_ij*DiffAux[1]
								   +v_ij*DiffAux[2]
                                                                   +w_ij*DiffAux[3]
								        -DiffAux[4])/c2_ij);
      Td w3 = l3 * (aux1 - aux2);
      Td w4 = l4 * ( v_ij*DiffAux[0]-DiffAux[2]);
      Td w5 = l5 * (-w_ij*DiffAux[0]+DiffAux[3]);
      
      // Compute "R_ij * |Lbd_ij| * L_ij * dU"
      Diff[0] = a[0] * ( w1 + w2 + w3 );
      Diff[1] = a[0] * ( (u_ij-c_ij)*w1 + u_ij*w2 + (u_ij+c_ij)*w3 );
      Diff[2] = a[0] * ( v_ij*(w1 + w2 + w3) - w4 );
      Diff[3] = a[0] * ( w_ij*(w1 + w2 + w3) + w5 );
      Diff[4] = a[0] * ( (H_ij-c_ij*u_ij)*w1 + q_ij*w2 + (H_ij+c_ij*u_ij)*w3
			 -v_ij*w4 + w_ij*w5 );
      
      //----------------------------------------------------------------------
      // Dimensional splitting: y-direction
      //----------------------------------------------------------------------
      
      // Compute eigenvalues
      l1 = abs(v_ij-c_ij);
      l2 = abs(v_ij);
      l3 = abs(v_ij+c_ij);
      l4 = abs(v_ij);
      l5 = abs(v_ij);
      
      // Compute solution difference U_j-U_i
      DiffAux[0] = IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1);
      DiffAux[1] = IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1);
      DiffAux[2] = IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1);
      DiffAux[3] = IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1);
      DiffAux[4] = IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1);
      
      // Compute auxiliary quantities for characteristic variables
      aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffAux[0]
					 -u_ij*DiffAux[1]
					 -v_ij*DiffAux[2]
					 -w_ij*DiffAux[3]
					      +DiffAux[4])/RCONST(2.0)/c2_ij;
      aux2 = (v_ij*DiffAux[0]-DiffAux[2])/RCONST(2.0)/c_ij;
      
      // Compute characteristic variables multiplied by the corresponding eigenvalue
      w1 = l1 * (aux1 + aux2);
      w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffAux[0]
		                   +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffAux[1]
								+v_ij*DiffAux[2]
								+w_ij*DiffAux[3]
								     -DiffAux[4])/c2_ij);
      w3 = l3 * (aux1 - aux2);
      w4 = l4 * (-u_ij*DiffAux[0]+DiffAux[1]);
      w5 = l5 * ( w_ij*DiffAux[0]-DiffAux[3]);
      
      // Compute "R_ij * |Lbd_ij| * L_ij * dU"
      Diff[0] += a[1] * ( w1 + w2 + w3 );
      Diff[1] += a[1] * ( u_ij*(w1 + w2 + w3) + w4 );
      Diff[2] += a[1] * ( (v_ij-c_ij)*w1 + v_ij*w2 + (v_ij+c_ij)*w3 );
      Diff[3] += a[1] * ( w_ij*(w1 + w2 + w3) - w5 );
      Diff[4] += a[1] * ( (H_ij-c_ij*v_ij)*w1 + q_ij*w2 + (H_ij+c_ij*v_ij)*w3
			  + u_ij*w4 -w_ij*w5 );
      
      //----------------------------------------------------------------------
      // Dimensional splitting: z-direction
      //----------------------------------------------------------------------
      
      // Compute eigenvalues
      l1 = abs(w_ij-c_ij);
      l2 = abs(w_ij);
      l3 = abs(w_ij+c_ij);
      l4 = abs(w_ij);
      l5 = abs(w_ij);
      
      // Compute solution difference U_j-U_i
      DiffAux[0] = IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1);
      DiffAux[1] = IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1);
      DiffAux[2] = IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1);
      DiffAux[3] = IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1);
      DiffAux[4] = IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1);
      
      // Compute auxiliary quantities for characteristic variables
      aux1 = ((HYDRO_GAMMA)-RCONST(1.0))*(q_ij*DiffAux[0]
					 -u_ij*DiffAux[1]
					 -v_ij*DiffAux[2]
					 -w_ij*DiffAux[3]
					      +DiffAux[4])/RCONST(2.0)/c2_ij;
      aux2 = (w_ij*DiffAux[0]-DiffAux[2])/RCONST(2.0)/c_ij;
      
      // Compute characteristic variables multiplied by the corresponding eigenvalue
      w1 = l1 * (aux1 + aux2);
      w2 = l2 * ((RCONST(1.0)-((HYDRO_GAMMA)-RCONST(1.0))*q_ij/c2_ij)*DiffAux[0]
		                   +((HYDRO_GAMMA)-RCONST(1.0))*(u_ij*DiffAux[1]
								+v_ij*DiffAux[2]
								+w_ij*DiffAux[3]
								     -DiffAux[4])/c2_ij);
      w3 = l3 * (aux1 - aux2);
      w4 = l4 * ( u_ij*DiffAux[0]-DiffAux[1]);
      w5 = l5 * (-v_ij*DiffAux[0]+DiffAux[2]);
      
      // Compute "R_ij * |Lbd_ij| * L_ij * dU"
      Diff[0] += a[2] * ( w1 + w2 + w3 );
      Diff[1] += a[2] * ( u_ij*(w1 + w2 + w3) - w4 );
      Diff[2] += a[2] * ( v_ij*(w1 + w2 + w3) + w5 );
      Diff[3] += a[2] * ( (w_ij-c_ij)*w1 + w_ij*w2 + (w_ij+c_ij)*w3 );
      Diff[4] += a[2] * ( (H_ij-c_ij*w_ij)*w1 + q_ij*w2 + (H_ij+c_ij*w_ij)*w3
			  -u_ij*w4 + v_ij*w5 );
    } else {
      Diff[0] = 0.0;
      Diff[1] = 0.0;
      Diff[2] = 0.0;
      Diff[3] = 0.0;
      Diff[4] = 0.0;
      }
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_RUSANOV>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff,
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui, 
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi,
		   Td pj,
		   Ti iedge, 
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the scalar dissipation of Rusanov-type
    //------------------------------------------------------------------------
    
    // Compute specific energies
    Td Ei = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
    Td Ej = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
    
    // Compute the speed of sound
    Td ci = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(HYDRO_GAMMA)*(Ei-RCONST(0.5)*(ui*ui+vi*vi+wi*wi)), 1e-14));
    Td cj = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(HYDRO_GAMMA)*(Ej-RCONST(0.5)*(uj*uj+vj*vj+wj*wj)), 1e-14));
    
#ifdef HYDRO_USE_IBP
    // Compute scalar dissipation based on the skew-symmetric part
    // which does not include the symmetric boundary contribution
    Td d_ij = max( abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*uj+
		       RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
			 	    IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*vj-
		       RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge))*wj)+
	       RCONST(0.5)*sqrt(POW(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
			        POW(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
		 	        POW(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge),2))*cj,
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*ui+
		       RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*vi+
	 	       RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge))*wi)+
	       RCONST(0.5)*sqrt(POW(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
			        POW(IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
			        POW(IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge),2))*ci );
#else
    // Compute scalar dissipation
    Td d_ij = max( abs(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*uj+
		       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*vj+
		       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*j)+
	      sqrt(POW(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
	 	   POW(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
		   POW(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge),2))*cj,
		   abs(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*ui+
		       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*vi+
		       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*wi)+
	      sqrt(POW(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
	 	   POW(IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge),2)+
		   POW(IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge),2))*ci );
#endif
    
    // Multiply the solution difference by the scalar dissipation
    Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1));
    Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1));
    Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1));
    Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1));
    Diff[4] = d_ij*(IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1));
  }
};

/*******************************************************************************/

template <>
struct calc_Dissipation<DISSIPATION_RUSANOV_DSPLIT>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval(Td *Diff, 
		   Td *CoeffsAtEdge,
		   Td *DataAtEdge,
		   Td ui,
		   Td uj,
		   Td vi,
		   Td vj,
		   Td wi,
		   Td wj,
		   Td pi, 
		   Td pj,
		   Ti iedge, 
		   Ti nedge,
		   Ti ncoeff)
  {
    //------------------------------------------------------------------------
    // Evaluate the scalar dissipation of Rusanov-type with dimensional splitting
    //------------------------------------------------------------------------
    
    // Compute specific energies
    Td Ei = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
    Td Ej = SPECIFICTOTALENERGY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
    
    // Compute the speed of sound
    Td ci = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(HYDRO_GAMMA)*(Ei-RCONST(0.5)*(ui*ui+vi*vi+wi*wi)), 1e-14));
    Td cj = sqrt(max(((HYDRO_GAMMA)-RCONST(1.0))*(HYDRO_GAMMA)*(Ej-RCONST(0.5)*(uj*uj+vj*vj+wj*wj)), 1e-14));
    
#ifdef HYDRO_USE_IBP
    // Compute scalar dissipation with dimensional splitting based on
    // the skew-symmetric part which does not include the symmetric
    // boundary contribution
    Td d_ij = max( abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*uj)+
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)))*cj,
	   	   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
		 	  	    IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*ui)+
	 	   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)))*ci )
            + max( abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*vj)+
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)))*cj,
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*vi)+
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)))*ci )
            + max( abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge))*wj)+
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)))*cj,
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge))*wi)+
		   abs(RCONST(0.5)*(IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)-
				    IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)))*ci );
#else
    // Compute scalar dissipation with dimensional splitting
    Td d_ij = max( abs(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*uj)+
		   abs(IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge))*cj,
		   abs(IDX3(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*ui)+
		   abs(IDX3(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge))*ci )
            + max( abs(IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*vj)+
		   abs(IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge))*cj,
		   abs(IDX3(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*vi)+
		   abs(IDX3(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge))*ci )
            + max( abs(IDX3(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*wj)+
		   abs(IDX3(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge))*cj,
		   abs(IDX3(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*wi)+
		   abs(IDX3(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge))*ci );
#endif
    
    // Multiply the solution difference by the scalar dissipation
    Diff[0] = d_ij*(IDX3(DataAtEdge,1,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,1,1,1,NVAR3D,2,1));
    Diff[1] = d_ij*(IDX3(DataAtEdge,2,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,2,1,1,NVAR3D,2,1));
    Diff[2] = d_ij*(IDX3(DataAtEdge,3,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,3,1,1,NVAR3D,2,1));
    Diff[3] = d_ij*(IDX3(DataAtEdge,4,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,4,1,1,NVAR3D,2,1));
    Diff[4] = d_ij*(IDX3(DataAtEdge,5,2,1,NVAR3D,2,1)-IDX3(DataAtEdge,5,1,1,NVAR3D,2,1));
  }
};

template <int isystemformat>
struct scatter_FluxesAtEdge
{ 
};

/*******************************************************************************/

struct calc_FluxesAtEdge
{
  template <typename Td, typename Ti>
#ifdef HYDRO_USE_IBP
  __device__ inline
  static void eval(Td *FluxesAtEdge,
		   Td *CoeffsAtEdge,
		   Td *Fxi,
		   Td *Fxj, 
		   Td *Fyi, 
		   Td *Fyj,
		   Td *Fzi,
		   Td *Fzj, 
		   Td *Diff,
		   Td scale,
		   Ti iedge, 
		   Ti ncoeff,
		   Ti nedge)
  {
    IDX3(FluxesAtEdge,1,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[0]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[0]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzj[0]-
       IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[0]-
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[0]-
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzi[0] + Diff[0]);
    
    IDX3(FluxesAtEdge,2,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[1]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[1]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzj[1]-
       IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[1]-
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[1]-
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzi[1] + Diff[1]);
    
    IDX3(FluxesAtEdge,3,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[2]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[2]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzj[2]-
       IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[2]-
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[2]-
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzi[2] + Diff[2]);
    
    IDX3(FluxesAtEdge,4,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[3]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[3]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzj[3]-
       IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[3]-
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[3]-
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzi[3] + Diff[3]);
    
    IDX3(FluxesAtEdge,5,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxj[4]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyj[4]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzj[4]-
       IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fxi[4]-
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fyi[4]-
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fzi[4] + Diff[4]);
    
    
    IDX3(FluxesAtEdge,1,2,1,NVAR3D,2,1) = -IDX3(FluxesAtEdge,1,1,1,NVAR3D,2,1);
    IDX3(FluxesAtEdge,2,2,1,NVAR3D,2,1) = -IDX3(FluxesAtEdge,2,1,1,NVAR3D,2,1);
    IDX3(FluxesAtEdge,3,2,1,NVAR3D,2,1) = -IDX3(FluxesAtEdge,3,1,1,NVAR3D,2,1);
    IDX3(FluxesAtEdge,4,2,1,NVAR3D,2,1) = -IDX3(FluxesAtEdge,4,1,1,NVAR3D,2,1);
    IDX3(FluxesAtEdge,5,2,1,NVAR3D,2,1) = -IDX3(FluxesAtEdge,5,1,1,NVAR3D,2,1);
  }
#else
  __device__ inline
  static void eval(Td *FluxesAtEdge,
		   Td *CoeffsAtEdge,
		   Td *Fx_ij,
		   Td *Fy_ij,
		   Td *Fz_ij,
		   Td *Diff,
		   Td scale,
		   Ti iedge,
		   Ti ncoeff,
		   Ti nedge)
  {
    IDX3(FluxesAtEdge,1,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[0]+
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[0]+
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[0] + Diff[0]);
    
    IDX3(FluxesAtEdge,2,1,1,NVAR3D,2,1) = scale *
      (IDX3T(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[1]+
       IDX3T(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[1]+
       IDX3T(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[1] + Diff[1]);
    
    IDX3(FluxesAtEdge,3,1,1,NVAR3D,2,1) = scale *
      (IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[2]+
       IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[2]+
       IDX3(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[2] + Diff[2]);
    
    IDX3(FluxesAtEdge,4,1,1,NVAR3D,2,1) = scale *
      (IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[3]+
       IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[3]+
       IDX3(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[3] + Diff[3]);
    
    IDX3(FluxesAtEdge,5,1,1,NVAR3D,2,1) = scale *
      (IDX3(CoeffsAtEdge,1,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[4]+
       IDX3(CoeffsAtEdge,2,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[4]+
       IDX3(CoeffsAtEdge,3,1,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[4] + Diff[4]);
    
    
    IDX3(FluxesAtEdge,1,2,1,NVAR3D,2,1) = -scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[0]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[0]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[0] + Diff[0]);
    
    IDX3(FluxesAtEdge,2,2,1,NVAR3D,2,1) = -scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[1]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[1]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[1] + Diff[1]);
    
    IDX3(FluxesAtEdge,3,2,1,NVAR3D,2,1) = -scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[2]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[2]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[2] + Diff[2]);
    
    IDX3(FluxesAtEdge,4,2,1,NVAR3D,2,1) = -scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[3]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[3]+
       IDX3T(CoeffsAtEdge,3,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[3] + Diff[3]);
    
    IDX3(FluxesAtEdge,5,2,1,NVAR3D,2,1) = -scale *
      (IDX3T(CoeffsAtEdge,1,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fx_ij[4]+
       IDX3T(CoeffsAtEdge,2,2,iedge,HYDRO_NDIM,ncoeff,nedge)*Fy_ij[4]+
       IDX3T(CoeffsAtEdge,3,2,iedge+idx,HYDRO_NDIM,ncoeff,nedge)*Fz_ij[4] + Diff[4]);
  }
#endif
};

/*******************************************************************************/

template <>
struct scatter_FluxesAtEdge<SYSTEM_SEGREGATED>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval (Td *FluxesAtEdge,
		    Td *Dy,
		    Ti i,
		    Ti j,
		    Ti neq)
  {
    // Solution vector is stored in interleaved format
    IDX2_REVERSE(Dy,1,i,NVAR3D,neq) += IDX3(FluxesAtEdge,1,1,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,2,i,NVAR3D,neq) += IDX3(FluxesAtEdge,2,1,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,3,i,NVAR3D,neq) += IDX3(FluxesAtEdge,3,1,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,4,i,NVAR3D,neq) += IDX3(FluxesAtEdge,4,1,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,5,i,NVAR3D,neq) += IDX3(FluxesAtEdge,5,1,1,NVAR3D,2,1);
    
    IDX2_REVERSE(Dy,1,j,NVAR3D,neq) += IDX3(FluxesAtEdge,1,2,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,2,j,NVAR3D,neq) += IDX3(FluxesAtEdge,2,2,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,3,j,NVAR3D,neq) += IDX3(FluxesAtEdge,3,2,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,4,j,NVAR3D,neq) += IDX3(FluxesAtEdge,4,2,1,NVAR3D,2,1);
    IDX2_REVERSE(Dy,5,j,NVAR3D,neq) += IDX3(FluxesAtEdge,5,2,1,NVAR3D,2,1);
  }
};

/*******************************************************************************/

template <>
struct scatter_FluxesAtEdge<SYSTEM_ALLCOUPLED>
{
  template <typename Td, typename Ti>
  __device__ inline
  static void eval (Td *FluxesAtEdge,
		    Td *Dy,
		    Ti i,
		    Ti j,
		    Ti neq)
  {
    // Solution vector is stored in block format
    IDX2_FORWARD(Dy,1,i,NVAR3D,neq) += IDX3(FluxesAtEdge,1,1,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,2,i,NVAR3D,neq) += IDX3(FluxesAtEdge,2,1,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,3,i,NVAR3D,neq) += IDX3(FluxesAtEdge,3,1,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,4,i,NVAR3D,neq) += IDX3(FluxesAtEdge,4,1,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,5,i,NVAR3D,neq) += IDX3(FluxesAtEdge,5,1,1,NVAR3D,2,1);
    
    IDX2_FORWARD(Dy,1,j,NVAR3D,neq) += IDX3(FluxesAtEdge,1,2,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,2,j,NVAR3D,neq) += IDX3(FluxesAtEdge,2,2,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,3,j,NVAR3D,neq) += IDX3(FluxesAtEdge,3,2,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,4,j,NVAR3D,neq) += IDX3(FluxesAtEdge,4,2,1,NVAR3D,2,1);
    IDX2_FORWARD(Dy,5,j,NVAR3D,neq) += IDX3(FluxesAtEdge,5,2,1,NVAR3D,2,1);
  }
};

/*******************************************************************************/
  
template <typename Td, typename Ti, int isystemformat, int idissipationtype>
__global__ void hydro_calcFlux3d_knl(Td *CoeffsAtEdge,
				     Ti *IedgeList,
				     Td *Dx,
				     Td *Dy,
				     Td scale,
				     Ti neq,
				     Ti nvar,
				     Ti nedge,
				     Ti ncoeff,
				     Ti nedges,
				     Ti iedgeset)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx<nedges)
    {
      // Get positions of edge endpoints (idx starts at zero)
      Ti i = IDX2(IedgeList,1,iedgeset+idx,6,nedge);
      Ti j = IDX2(IedgeList,2,iedgeset+idx,6,nedge);
      
      // Local variables
      Td DataAtEdge[2*NVAR3D];
      Td Diff[NVAR3D];
      
      // Get solution values at edge endpoints
      gather_DataAtEdge<isystemformat>::
	eval(DataAtEdge,Dx,i,j,neq);
      
      // Compute velocities
      Td ui = XVELOCITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      Td vi = YVELOCITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      Td wi = ZVELOCITY3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      
      Td uj = XVELOCITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
      Td vj = YVELOCITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
      Td wj = ZVELOCITY3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);
      
      // Compute pressures
      Td pi = PRESSURE3(DataAtEdge,IDX3,1,1,NVAR3D,2,1);
      Td pj = PRESSURE3(DataAtEdge,IDX3,2,1,NVAR3D,2,1);

#ifdef HYDRO_USE_IBP
      Td Fxi[NVAR3D];
      Td Fxj[NVAR3D];
      Td Fyi[NVAR3D];
      Td Fyj[NVAR3D];
      Td Fzi[NVAR3D];
      Td Fzj[NVAR3D];
      
      // Compute the Galerkin fluxes
      calc_GalerkinFlux::
	eval(Fxi,Fxj,Fyi,Fyj,Fzi,Fzj,DataAtEdge,ui,uj,vi,vj,wi,wj,pi,pj);
#else
      Td Fx_ij[NVAR3D];
      Td Fy_ij[NVAR3D];
      Td Fz_ij[NVAR3D];

      // Compute the Galerkin fluxes
      calc_GalerkinFlux::
	calc_eval(Fx_ij,Fy_ij,Fz_ij,DataAtEdge,ui,uj,vi,vj,wi,wj,pi,pj);
#endif

      // Compute the artificial viscosities
      calc_Dissipation<idissipationtype>::
	eval(Diff,CoeffsAtEdge,DataAtEdge,ui,uj,vi,vj,wi,wj,pi,pj,iedgeset+idx,nedge,ncoeff);
      
      Td FluxesAtEdge[2*NVAR3D];
      // Build both contributions into the fluxes
#ifdef HYDRO_USE_IBP
      calc_FluxesAtEdge::
	eval(FluxesAtEdge,CoeffsAtEdge,Fxi,Fxj,Fyi,Fyj,Fzi,Fzj,Diff,scale,iedgeset+idx,ncoeff,nedge);
#else
      calc_FluxesAtEdge::
	eval(FluxesAtEdge,CoeffsAtEdge,Fx_ij,Fy_ij,Fz_ij,Diff,scale,iedgeset+idx,ncoeff,nedge);
#endif
        
      // Build fluxes into nodal vector
      scatter_FluxesAtEdge<isystemformat>::
	eval(FluxesAtEdge,Dy,i,j,neq);
    }
}

/*******************************************************************************
 * External C functions which can be called from the Fortran code
 *******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxScDiss3d_cuda(__I64 *d_CoeffsAtEdge,
				__I64 *d_IedgeList,
				__I64 *d_Dx,
				__I64 *d_Dy,
				Td scale,
				Ti nblocks,
				Ti neq,
				Ti nvar,
				Ti nedge, 
				Ti ncoeff,
				Ti nedges,
				Ti iedgeset,
				hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_SCALAR>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale,
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_SCALAR>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxScDiss3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxscdiss3d_cuda)(__I64 *d_CoeffsAtEdge,
					   __I64 *d_IedgeList,
					   __I64 *d_Dx,
					   __I64 *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
					   __DP *scale,
#else
					   __SP *scale,
#endif
					   __INT *nblocks,
					   __INT *neq,
					   __INT *nvar,
					   __INT *nedge,
					   __INT *ncoeff,
					   __INT *nedges,
					   __INT *iedgeset,
					   __I64 *stream)
  {
    return (__INT) hydro_calcFluxScDiss3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
					       *scale, *nblocks, *neq, *nvar, *nedge,
					       *ncoeff, *nedges, *iedgeset,
					       (hipStream_t)*stream);
  }
}

/*******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxScDissDiSp3d_cuda(__I64 *d_CoeffsAtEdge,
				    __I64 *d_IedgeList,
				    __I64 *d_Dx,
				    __I64 *d_Dy,
				    Td scale,
				    Ti nblocks,
				    Ti neq, 
				    Ti nvar,
				    Ti nedge,
				    Ti ncoeff,
				    Ti nedges,
				    Ti iedgeset,
				    hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_SCALAR_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_SCALAR_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxScDissDiSp3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxscdissdisp3d_cuda)(__SIZET *d_CoeffsAtEdge,
					       __SIZET *d_IedgeList,
					       __SIZET *d_Dx,
					       __SIZET *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
					       __DP *scale,
#else
					       __SP *scale,
#endif
					       __INT *nblocks, 
					       __INT *neq, 
					       __INT *nvar,
					       __INT *nedge, 
					       __INT *ncoeff,
					       __INT *nedges, 
					       __INT *iedgeset,
					       __I64 *stream=0)
  {
    return (__INT) hydro_calcFluxScDissDiSp3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
						   *scale, *nblocks, *neq, *nvar, *nedge,
						   *ncoeff, *nedges, *iedgeset,
						   (hipStream_t)(*stream));
  }
}

/*******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxRoeDiss3d_cuda(__I64 *d_CoeffsAtEdge,
				 __I64 *d_IedgeList,
				 __I64 *d_Dx,
				 __I64 *d_Dy,
				 Td scale,
				 Ti nblocks, 
				 Ti neq, 
				 Ti nvar,
				 Ti nedge,
				 Ti ncoeff,
				 Ti nedges, 
				 Ti iedgeset,
				 hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_ROE>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_ROE>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxRoeDiss3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxroediss3d_cuda)(__SIZET *d_CoeffsAtEdge,
					    __SIZET *d_IedgeList,
					    __SIZET *d_Dx,
					    __SIZET *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
					    __DP *scale,
#else
					    __SP *scale,
#endif
					    __INT *nblocks, 
					    __INT *neq, 
					    __INT *nvar,
					    __INT *nedge, 
					    __INT *ncoeff,
					    __INT *nedges, 
					    __INT *iedgeset,
					    __I64 *stream=0)
  {
    return (__INT) hydro_calcFluxRoeDiss3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
						*scale, *nblocks, *neq, *nvar, *nedge,
						*ncoeff, *nedges, *iedgeset,
						(hipStream_t)(*stream));
  }
}

/*******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxRoeDissDiSp3d_cuda(__I64 *d_CoeffsAtEdge,
				     __I64 *d_IedgeList,
				     __I64 *d_Dx,
				     __I64 *d_Dy,
				     Td scale,
				     Ti nblocks, 
				     Ti neq, 
				     Ti nvar,
				     Ti nedge,
				     Ti ncoeff,
				     Ti nedges, 
				     Ti iedgeset,
				     hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_ROE_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_ROE_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxRoeDissDiSp3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxroedissdisp3d_cuda)(__SIZET *d_CoeffsAtEdge,
						__SIZET *d_IedgeList,
						__SIZET *d_Dx,
						__SIZET *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
						__DP *scale,
#else
						__SP *scale,
#endif
						__INT *nblocks, 
						__INT *neq, 
						__INT *nvar,
						__INT *nedge, 
						__INT *ncoeff,
						__INT *nedges, 
						__INT *iedgeset,
						__I64 *stream=0)
  {
    return (__INT) hydro_calcFluxRoeDissDiSp3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
						    *scale, *nblocks, *neq, *nvar, *nedge,
						    *ncoeff, *nedges, *iedgeset,
						    (hipStream_t)*stream);
  }
}

/*******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxRusDiss3d_cuda(__I64 *d_CoeffsAtEdge,
				 __I64 *d_IedgeList,
				 __I64 *d_Dx,
				 __I64 *d_Dy,
				 Td scale,
				 Ti nblocks, 
				 Ti neq, 
				 Ti nvar,
				 Ti nedge, 
				 Ti ncoeff,
				 Ti nedges, 
				 Ti iedgeset,
				 hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_RUSANOV>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_RUSANOV>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxRusDiss3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxrusdiss3d_cuda)(__SIZET *d_CoeffsAtEdge,
					    __SIZET *d_IedgeList,
					    __SIZET *d_Dx,
					    __SIZET *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
					    __DP *scale,
#else
					    __SP *scale,
#endif
					    __INT *nblocks, 
					    __INT *neq, 
					    __INT *nvar,
					    __INT *nedge, 
					    __INT *ncoeff,
					    __INT *nedges, 
					    __INT *iedgeset,
					    __I64 *stream=0)
  {
    return (__INT)hydro_calcFluxRusDiss3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
					       *scale, *nblocks, *neq, *nvar, *nedge,
					       *ncoeff, *nedges, *iedgeset,
					       (hipStream_t)*stream);
  }
}

/*******************************************************************************/

template <typename Td, typename Ti>
inline
int hydro_calcFluxRusDissDiSp3d_cuda(__I64 *d_CoeffsAtEdge,
				     __I64 *d_IedgeList,
				     __I64 *d_Dx,
				     __I64 *d_Dy,
				     Td scale,
				     Ti nblocks, 
				     Ti neq,
				     Ti nvar,
				     Ti nedge, 
				     Ti ncoeff,
				     Ti nedges, 
				     Ti iedgeset,
				     hipStream_t stream)
{
  Td *Dx = (Td*)(*d_Dx);
  Td *Dy = (Td*)(*d_Dy);
  Td *CoeffsAtEdge = (Td*)(*d_CoeffsAtEdge);
  Ti *IedgeList = (Ti*)(*d_IedgeList);
  
  // Define number of threads per block
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((nedges)/(double)(block.x));
  
  if (nblocks == 1) {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_SEGREGATED,DISSIPATION_RUSANOV_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  } else {
    hydro_calcFlux3d_knl
      <Td,Ti,SYSTEM_ALLCOUPLED,DISSIPATION_RUSANOV_DSPLIT>
      <<<grid, block, 0, stream>>>(CoeffsAtEdge,
				   IedgeList,
				   Dx, Dy, scale, 
				   neq, nvar,
				   nedge, ncoeff,
				   nedges, iedgeset);
  }
  coproc_checkErrors("hydro_calcFluxRusDissDiSp3d_cuda");
  return 0;
}

/*******************************************************************************/
extern "C" {
  __INT FNAME(hydro_calcfluxrusdissdisp3d_cuda)(__SIZET *d_CoeffsAtEdge,
						__SIZET *d_IedgeList,
						__SIZET *d_Dx,
						__SIZET *d_Dy,
#ifdef HAS_CUDADOUBLEPREC
						__DP *scale,
#else
						__SP *scale,
#endif
						__INT *nblocks, 
						__INT *neq, 
						__INT *nvar,
						__INT *nedge, 
						__INT *ncoeff,
						__INT *nedges, 
						__INT *iedgeset,
						__I64 *stream)
  {
    return (__INT) hydro_calcFluxRusDissDiSp3d_cuda(d_CoeffsAtEdge, d_IedgeList, d_Dx, d_Dy,
						    *scale, *nblocks, *neq, *nvar, *nedge,
						    *ncoeff, *nedges, *iedgeset,
						    (hipStream_t)*stream);
  }
}
