#include "hip/hip_runtime.h"
/*#############################################################################
 ******************************************************************************
 * <name> coproc_storage_cuda </name>
 ******************************************************************************
 *
 * <purpose>
 * This file provides the basic routines for handling coprocessor
 * support in CUDA. Routines for creating and deleting storage on the
 * device and transfering data between host and device memory are
 * provided.
 * </purpose>
 *
 *#############################################################################
 */

#include <stdio.h>
#include <math.h>
#include <iostream>
#include "coproc_core.h"
#include "coproc_storage_cuda.h"


/*******************************************************************************
 * Allocate new memory on host
 *******************************************************************************
 */
int coproc_newMemoryOnHost(unsigned long * p_MemoryBlock,
			   unsigned long * imemBytes)
{
  void * d_MemoryBlock = 0;

  hipHostAlloc((void**)&d_MemoryBlock, *imemBytes, hipHostMallocDefault);
  *p_MemoryBlock = (unsigned long) d_MemoryBlock;
  coproc_checkErrors("coproc_newMemoryOnHost");
  return 0;
}

int FNAME(coproc_newmemoryonhost)(unsigned long * p_MemoryBlock,
				  unsigned long * imemBytes)
{
  return coproc_newMemoryOnHost(p_MemoryBlock, imemBytes);
}

/*******************************************************************************
 * Free existing memory on host
 *******************************************************************************
 */
int coproc_freeMemoryOnHost(unsigned long * p_MemoryBlock)
{
  void * d_MemoryBlock = (void*)(*p_MemoryBlock);

  hipHostFree(d_MemoryBlock);
  *p_MemoryBlock = 0;
  coproc_checkErrors("coproc_freeMemoryOnHost");
  return 0;
}

int FNAME(coproc_freememoryonhost)(unsigned long * p_MemoryBlock)
{
  return coproc_freeMemoryOnHost(p_MemoryBlock);
}

/*******************************************************************************
 * Allocate new memory on device
 *******************************************************************************
 */
int coproc_newMemoryOnDevice(unsigned long * p_MemoryBlock,
			     unsigned long * imemBytes)
{
  void * d_MemoryBlock = 0;

  hipMalloc((void**)&d_MemoryBlock, *imemBytes);
  *p_MemoryBlock = (unsigned long) d_MemoryBlock;
  coproc_checkErrors("coproc_newMemoryOnDevice");
  return 0;
}

int FNAME(coproc_newmemoryondevice)(unsigned long * p_MemoryBlock,
				    unsigned long * imemBytes)
{
  return coproc_newMemoryOnDevice(p_MemoryBlock, imemBytes);
}


/*******************************************************************************
 * Free existing memory on device
 *******************************************************************************
 */
int coproc_freeMemoryOnDevice(unsigned long * p_MemoryBlock)
{
  void * d_MemoryBlock = (void*)(*p_MemoryBlock);

  hipFree(d_MemoryBlock);
  *p_MemoryBlock = 0;
  coproc_checkErrors("coproc_freeMemoryOnDevice");
  return 0;
}

int FNAME(coproc_freememoryondevice)(unsigned long * p_MemoryBlock)
{
  return coproc_freeMemoryOnDevice(p_MemoryBlock);
}


/*******************************************************************************
 * Clear memory on device
 *******************************************************************************
 */
int coproc_clearMemoryOnDevice(unsigned long * p_MemoryBlock,
			       unsigned long * imemBytes)
{
  hipMemset((void*)(*p_MemoryBlock), 0, *imemBytes);
  coproc_checkErrors("coproc_clearMemoryOnDevice");
  return 0;
}

int FNAME(coproc_clearmemoryondevice)(unsigned long * p_MemoryBlock,
				      unsigned long * imemBytes)
{
  return coproc_clearMemoryOnDevice(p_MemoryBlock, imemBytes);
}


/*******************************************************************************
 * Copy host memory to device memory
 *******************************************************************************
 */
int coproc_copyMemoryHostToDevice(unsigned long * p_MemoryBlockOnHost, 
				  unsigned long * p_MemoryBlockOnDevice,
				  unsigned long * imemBytes)
{
  void * d_MemoryBlockOnDevice = (void*)(*p_MemoryBlockOnDevice);

  hipMemcpy(d_MemoryBlockOnDevice, p_MemoryBlockOnHost,
	     *imemBytes, hipMemcpyHostToDevice);
  coproc_checkErrors("coproc_copyMemoryHostToDev");
  return 0;
}

int FNAME(coproc_copymemoryhosttodevice)(unsigned long * p_MemoryBlockOnHost,
					 unsigned long * p_MemoryBlockOnDevice,
					 unsigned long * imemBytes)
{
  return coproc_copyMemoryHostToDevice(p_MemoryBlockOnHost,
				       p_MemoryBlockOnDevice, imemBytes);
}


/*******************************************************************************
 * Copy device memory to host memory
 *******************************************************************************
 */
int coproc_copyMemoryDeviceToHost(unsigned long * p_MemoryBlockOnDevice,
				  unsigned long * p_MemoryBlockOnHost,
				  unsigned long * imemBytes)
{
  void * d_MemoryBlockOnDevice = (void*)(*p_MemoryBlockOnDevice);

  hipMemcpy(p_MemoryBlockOnHost, d_MemoryBlockOnDevice,
	     *imemBytes, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  coproc_checkErrors("coproc_copyMemoryDeviceToHost");
  return 0;
}

int FNAME(coproc_copymemorydevicetohost)(unsigned long * p_MemoryBlockOnDevice,
					 unsigned long * p_MemoryBlockOnHost,
					 unsigned long * imemBytes)
{
  return coproc_copyMemoryDeviceToHost(p_MemoryBlockOnDevice,
				       p_MemoryBlockOnHost, imemBytes);
}


/*******************************************************************************
 * Copy device memory data to device memory
 *******************************************************************************
 */
int coproc_copyMemoryDeviceToDevice(unsigned long * p_MemoryBlockSrc,
				    unsigned long * p_MemoryBlockDest,
				    unsigned long * imemBytes)
{
  void * d_MemoryBlockSrc  = (void*)(*p_MemoryBlockSrc);
  void * d_MemoryBlockDest = (void*)(*p_MemoryBlockDest);
  
  if (d_MemoryBlockDest != d_MemoryBlockSrc) {
    hipMemcpy(d_MemoryBlockDest, d_MemoryBlockSrc,
	       *imemBytes, hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    coproc_checkErrors("coproc_copyMemoryDeviceToDevice");
  }
  return 0;
}

int FNAME(coproc_copymemorydevicetodevice)(unsigned long * p_MemoryBlockSrc,
					   unsigned long * p_MemoryBlockDest,
					   unsigned long * imemBytes)
{
  return coproc_copyMemoryDeviceToDevice(p_MemoryBlockSrc,
					 p_MemoryBlockDest, imemBytes);
}

/*******************************************************************************
 * Add two real memory blocks in device memory
 *******************************************************************************
 */

__global__ void addSingleOnDevice_knl(float * d_MemoryBlock1,
				      float * d_MemoryBlock2,
				      float * d_MemoryBlockDest,
				      int imemSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx<imemSize)
  {
    d_MemoryBlockDest[idx] = d_MemoryBlock1[idx] + d_MemoryBlock2[idx];
  }
}

int coproc_addSingleOnDevice(unsigned long * p_MemoryBlock1,
			     unsigned long * p_MemoryBlock2,
			     unsigned long * p_MemoryBlockDest,
			     unsigned long * imemSize)
{
  float * d_MemoryBlock1    = (float*)(*p_MemoryBlock1);
  float * d_MemoryBlock2    = (float*)(*p_MemoryBlock2);
  float * d_MemoryBlockDest = (float*)(*p_MemoryBlockDest);

  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*imemSize)/(float)(block.x));
  addSingleOnDevice_knl<<<grid, block>>>(d_MemoryBlock1, d_MemoryBlock2,
					 d_MemoryBlockDest, *imemSize);
  return 0;
}

int FNAME(coproc_addsingleondevice)(unsigned long * p_MemoryBlock1,
				    unsigned long * p_MemoryBlock2,
				    unsigned long * p_MemoryBlockDest,
				    unsigned long * imemSize)
{
  return coproc_addSingleOnDevice(p_MemoryBlock1, p_MemoryBlock2,
				  p_MemoryBlockDest, imemSize);
}

/*******************************************************************************
 * Add two double memory blocks in device memory
 *******************************************************************************
 */

__global__ void addDoubleOnDevice_knl(double * d_MemoryBlock1,
				      double * d_MemoryBlock2,
				      double * d_MemoryBlockDest,
				      int imemSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx<imemSize)
  {
    d_MemoryBlockDest[idx] = d_MemoryBlock1[idx] + d_MemoryBlock2[idx];
  }
}

int coproc_addDoubleOnDevice(unsigned long * p_MemoryBlock1,
			     unsigned long * p_MemoryBlock2,
			     unsigned long * p_MemoryBlockDest,
			     unsigned long * imemSize)
{
  double * d_MemoryBlock1    = (double*)(*p_MemoryBlock1);
  double * d_MemoryBlock2    = (double*)(*p_MemoryBlock2);
  double * d_MemoryBlockDest = (double*)(*p_MemoryBlockDest);
  
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*imemSize)/(double)(block.x));
  addDoubleOnDevice_knl<<<grid, block>>>(d_MemoryBlock1, d_MemoryBlock2,
					 d_MemoryBlockDest, *imemSize); 
  return 0;
}

int FNAME(coproc_adddoubleondevice)(unsigned long * p_MemoryBlock1,
				    unsigned long * p_MemoryBlock2,
				    unsigned long * p_MemoryBlockDest,
				    unsigned long * imemSize)
{
  return coproc_addDoubleOnDevice(p_MemoryBlock1, p_MemoryBlock2,
				  p_MemoryBlockDest, imemSize);
}

/*******************************************************************************
 * Add two integer memory blocks in device memory
 *******************************************************************************
 */

__global__ void addIntegerOnDevice_knl(int * d_MemoryBlock1,
				       int * d_MemoryBlock2,
				       int * d_MemoryBlockDest,
				       int imemSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx<imemSize)
  {
    d_MemoryBlockDest[idx] = d_MemoryBlock1[idx] + d_MemoryBlock2[idx];
  }
}

int coproc_addIntegerOnDevice(unsigned long * p_MemoryBlock1,
			      unsigned long * p_MemoryBlock2,
			      unsigned long * p_MemoryBlockDest,
			      unsigned long * imemSize)
{
  int * d_MemoryBlock1    = (int*)(*p_MemoryBlock1);
  int * d_MemoryBlock2    = (int*)(*p_MemoryBlock2);
  int * d_MemoryBlockDest = (int*)(*p_MemoryBlockDest);
  
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*imemSize)/(int)(block.x));
  addIntegerOnDevice_knl<<<grid, block>>>(d_MemoryBlock1, d_MemoryBlock2,
					  d_MemoryBlockDest, *imemSize);
  return 0;
}

int FNAME(coproc_addintegerondevice)(unsigned long * p_MemoryBlock1,
				     unsigned long * p_MemoryBlock2,
				     unsigned long * p_MemoryBlockDest,
				     unsigned long * imemSize)
{
  return coproc_addIntegerOnDevice(p_MemoryBlock1, p_MemoryBlock2,
				   p_MemoryBlockDest, imemSize);
}

/*******************************************************************************
 * Add two logical memory blocks in device memory
 *******************************************************************************
 */

__global__ void addLogicalOnDevice_knl(int * d_MemoryBlock1,
				       int * d_MemoryBlock2,
				       int * d_MemoryBlockDest,
				       int imemSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx<imemSize)
  {
    d_MemoryBlockDest[idx] = d_MemoryBlock1[idx] || d_MemoryBlock2[idx];
  }
}

int coproc_addLogicalOnDevice(unsigned long * p_MemoryBlock1,
			      unsigned long * p_MemoryBlock2,
			      unsigned long * p_MemoryBlockDest,
			      unsigned long * imemSize)
{
  int * d_MemoryBlock1    = (int*)(*p_MemoryBlock1);
  int * d_MemoryBlock2    = (int*)(*p_MemoryBlock2);
  int * d_MemoryBlockDest = (int*)(*p_MemoryBlockDest);
  
  int blocksize = 128;
  dim3 grid;
  dim3 block;
  block.x = blocksize;
  grid.x = (unsigned)ceil((*imemSize)/(int)(block.x));
  addLogicalOnDevice_knl<<<grid, block>>>(d_MemoryBlock1, d_MemoryBlock2,
					  d_MemoryBlockDest, *imemSize);
  return 0;
}

int FNAME(coproc_addlogicalondevice)(unsigned long * p_MemoryBlock1,
				     unsigned long * p_MemoryBlock2,
				     unsigned long * p_MemoryBlockDest,
				     unsigned long * imemSize)
{
  return coproc_addLogicalOnDevice(p_MemoryBlock1, p_MemoryBlock2,
				   p_MemoryBlockDest, imemSize);
}
